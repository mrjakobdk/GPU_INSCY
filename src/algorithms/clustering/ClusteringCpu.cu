#include "hip/hip_runtime.h"

#include <torch/extension.h>
#include "ClusteringCpu.h"
#include "../../utils/util.h"
#include "../../utils/util_data.h"
#include "../../structures/ScyTreeNode.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#include <numeric>
//#include <windows.h>
//#include <thrust/device_vector.h>
#include <hip/hip_runtime_api.h>
#include <queue>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <set>
#include <map>

#define BLOCK_WIDTH 64

#define PI 3.14


using namespace std;


double gamma(double d);


double omega(int subspace_size) {
    return 2.0 / (subspace_size + 0.2);
}

double dist(int p_id, int q_id, at::Tensor X, int *subspace, int subspace_size) {
    float* p = X[p_id].data_ptr<float>();
    float* q = X[q_id].data_ptr<float>();
    double distance = 0;
    for (int i = 0; i < subspace_size; i++) {
        int d_i = subspace[i];
        double diff = p[d_i] - q[d_i];
        distance += diff * diff;
    }

    return sqrt(distance);
}

vector<int> neighborhood(ScyTreeNode *neighborhood_tree, int p_id, at::Tensor X,
                         float neighborhood_size, int *subspace, int subspace_size) {
    vector<int> neighbors;

    //get_possible_neighbors(neighborhood_size, X[p_id]);//todo just all points in scy_tree


    //printf("Possible neighbors size: %d\n", possible_neighbors.size());

    float* p = X[p_id].data_ptr<float>();
    //printf("neighborhood 1\n");
    vector<int> possible_neighbors = neighborhood_tree->get_possible_neighbors(p, subspace, subspace_size,
                                                                               neighborhood_size);

    int count = 0;

    for (int q_id: possible_neighbors) {
        count++;
        if (p_id == q_id) {//todo exclude or include your self?
            continue;
        }
        float distance = dist(p_id, q_id, X, subspace, subspace_size);

        if (neighborhood_size >= distance) {
            neighbors.push_back(q_id);
        }
    }
    return neighbors;
}

float phi(int point_id, vector<int> neighbors, float neighborhood_size, at::Tensor X, int *subspace,
          int subspace_size) {


    double sum = 0;
    for (int q_id : neighbors) {
        double d = dist(point_id, q_id, X, subspace, subspace_size) / neighborhood_size;
        double sq = d * d;
        sum += (1. - sq);
    }

    return sum;

}

double gamma(double n) {
    if (round(n) == 1) {//todo not nice cond n==1
        return 1.;
    } else if (n < 1) {//todo not nice cond n==1/2
        return sqrt(PI);
    }
    return (n - 1.) * gamma(n - 1.);
}

double gamma(int n) {
    if (n == 2) {
        return 1.;
    } else if (n == 1) {
        return sqrt(PI);
    }
    return (n / 2. - 1.) * gamma(n - 2);
}

double c(int subspace_size) {
    double r = pow(PI, subspace_size / 2.);
    r = r / gamma(subspace_size / 2. + 1.);
    return r;
}

float alpha(int subspace_size, float neighborhood_size, int n) {
    float v = 1.;//todo v is missing?? what is it??
    float r = 2 * n * pow(neighborhood_size, subspace_size) * c(subspace_size);
    r = r / (pow(v, subspace_size) * (subspace_size + 2));
    return r;
}

bool dense(int point_id, vector<int> neighbors, float neighborhood_size, at::Tensor X, int *subspace,
           int subspace_size,
           float F, int n, int num_obj) {
    float p = phi(point_id, neighbors, neighborhood_size, X, subspace, subspace_size);
    float a = alpha(subspace_size, neighborhood_size, n);
    float w = omega(subspace_size);
    return p >= max(F * a, num_obj * w);
}


vector<int>
INSCYClusteringImplCPU2(ScyTreeNode *scy_tree, ScyTreeNode *neighborhood_tree, at::Tensor X, int n,
                        float neighborhood_size, float F,
                        int num_obj) {
    int *subspace = scy_tree->restricted_dims;
    int subspace_size = scy_tree->number_of_restricted_dims;

//    printf("subspace: ");
//    print_array(subspace, subspace_size);
//
//    printf("point 5: ");
//    print_array(X[5].data(), X[5].size());

    vector<int> labels(n, -1);
    int clustered_count = 0;
    int prev_clustered_count = 0;
    int next_cluster_label = 1;
    for (int i : scy_tree->get_points()) {

        if (labels[i] != -1) {//already checked
            continue;
        }

        int label = next_cluster_label;
        prev_clustered_count = clustered_count;
        queue<int> q;
        q.push(i);

        int c = 0;
        while (!q.empty()) {
            c++;
            int p_id = q.front();
            q.pop();
            //todo how long is this function taking?
            //todo would it be faster to use a tree to restrict the neighborhood?
            vector<int> neighbors = neighborhood(neighborhood_tree, p_id, X, neighborhood_size, subspace,
                                                 subspace_size);


            //printf("%d neighborhood: ",p_id);
            //print_array(neighbors, neighbors.size());

            bool is_dense = dense(p_id, neighbors, neighborhood_size, X, subspace, subspace_size, F, n, num_obj);
            //printf("%d is dense: %d\n", p_id, is_dense);
            if (is_dense) {
                labels[p_id] = label;
                clustered_count++;
                for (int q_id : neighbors) {
                    if (labels[q_id] == -1) {
                        labels[q_id] = -2;
                        q.push(q_id);
                    }
                }
            }
        }
        if (clustered_count > prev_clustered_count) {
            next_cluster_label++;
        }
    }
    return labels;
}
