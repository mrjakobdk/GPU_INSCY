#include "hip/hip_runtime.h"
//
// Created by mrjakobdk on 5/4/20.
//

#include "InscyArrayGpu.h"
#include "../clustering/ClusteringGpuStreams.h"
#include "../clustering/ClusteringGpu.cuh"
#include "../clustering/ClusteringCpu.h"
#include "../../structures/ScyTreeArray.h"
#include "../../utils/util.h"

#include <math.h>
#include <map>
#include <vector>

using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

void InscyArrayGpu(ScyTreeArray *scy_tree, float *d_X, int n, int d, float neighborhood_size, float F, int num_obj, int min_size,
                   map <vector<int>, vector<int>, vec_cmp> &result,
                   int first_dim_no,
                   int total_number_of_dim, float r, int &calls) {

//    printf("call: %d, first_dim_no: %d, points: %d\n", calls, first_dim_no, scy_tree->number_of_points);
//    scy_tree->copy_to_host();
//    scy_tree->print();

//    std::vector < ScyTreeArray * > scy_tree_list;
    int dim_no = first_dim_no;
    calls++;
    while (dim_no < total_number_of_dim) {
        int cell_no = 0;

        vector<int> subspace;
        int *d_clustering; // number_of_points
        hipMalloc(&d_clustering, sizeof(int) * n);
        hipMemset(d_clustering, -1, sizeof(int) * n);

        while (cell_no < scy_tree->number_of_cells) {

            //restricted-tree := restrict(scy-tree, descriptor);

            gpuErrchk(hipPeekAtLastError());
            ScyTreeArray *restricted_scy_tree = scy_tree->restrict_gpu(dim_no, cell_no);
            gpuErrchk(hipPeekAtLastError());

            hipMemcpy(restricted_scy_tree->h_restricted_dims, restricted_scy_tree->d_restricted_dims,
                       sizeof(int) * restricted_scy_tree->number_of_restricted_dims, hipMemcpyDeviceToHost);
            subspace = vector<int>(restricted_scy_tree->h_restricted_dims,
                                   restricted_scy_tree->h_restricted_dims +
                                   restricted_scy_tree->number_of_restricted_dims);

            //restricted-tree := mergeWithNeighbors(restricted-tree);
            restricted_scy_tree = restricted_scy_tree->mergeWithNeighbors_gpu1(scy_tree, dim_no, cell_no);

            //pruneRecursion(restricted-tree); //prune sparse regions
            if (restricted_scy_tree->pruneRecursion_gpu(min_size, d_X, n, d, neighborhood_size, F, num_obj)) {

                //INSCY(restricted-tree,result); //depth-first via recursion
                map <vector<int>, vector<int>, vec_cmp> sub_result;
                InscyArrayGpu(restricted_scy_tree, d_X, n, d, neighborhood_size,
                              F, num_obj, min_size, sub_result, dim_no + 1, total_number_of_dim, r, calls);
                result.insert(sub_result.begin(), sub_result.end());

                //pruneRedundancy(restricted-tree); //in-process-removal
                if (restricted_scy_tree->pruneRedundancy_gpu(r, result)) {

                    //scy_tree_list.push_back(restricted_scy_tree);
//                    vector<int> subspace_clustering = ClusteringGPU(restricted_scy_tree, d_X, n, d, neighborhood_size,
//                                                                    F, num_obj);
                    ClusteringGPU(d_clustering, restricted_scy_tree, d_X, n, d, neighborhood_size,
                                  F, num_obj);


//                    vector<int> subspace(restricted_scy_tree->h_restricted_dims,
//                                         restricted_scy_tree->h_restricted_dims +
//                                         restricted_scy_tree->number_of_restricted_dims);

                }
            } else {
                // delete restricted_scy_tree;
            }
            cell_no++;
        }

        int *h_clustering = new int[n];
        hipMemcpy(h_clustering, d_clustering, sizeof(int) * n, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
        vector<int> subspace_clustering(h_clustering, h_clustering + n);

        join(result, subspace_clustering, subspace, min_size, r);

        dim_no++;
    }
    gpuErrchk(hipPeekAtLastError());

//
//    std::vector<std::vector<int>> new_clustering_list = ClusteringGpuStream(scy_tree_list, d_X, n, d,
//                                                                           neighborhood_size, F,
//                                                                           num_obj);

//    for (int k = 0; k < scy_tree_list.size(); k++) {
//
//        ScyTreeArray *restricted_scy_tree_gpu = scy_tree_list[k];
//        std::vector<int> new_clustering = ClusteringGPU(restricted_scy_tree_gpu, d_X, n, d, neighborhood_size, F,
//                                                        num_obj);
//        gpuErrchk(hipPeekAtLastError());
//        //result := DBClustering(restricted-tree) ∪ result;
//        int idx = restricted_scy_tree_gpu->get_dims_idx();
//
//        if (result.count(idx)) {
//            std::vector<int> clustering = result[idx];
//            int m = v_max(clustering);
//            if (m < 0) {
//                result[idx] = new_clustering;
//            } else {
//                for (int i = 0; i < n; i++) {
//                    if (new_clustering[i] == -2) {
//                        clustering[i] = new_clustering[i];
//                    } else if (new_clustering[i] >= 0) {
//                        clustering[i] = m + 1 + new_clustering[i];
//                    }
//                }
//                result[idx] = clustering;
//            }
//        } else {
//            result.insert(std::pair < int, std::vector < int >> (idx, new_clustering));
//        }

    // delete restricted_scy_tree_gpu;
//    }

    int total_inscy = pow(2, total_number_of_dim);
    printf("GPU-INSCY(%d): %d%%      \r", calls, int((result.size() * 100) / total_inscy));
}