#include "hip/hip_runtime.h"
//
// Created by mrjakobdk on 5/2/20.
//

#include "InscyCpuGpuMix.h"
#include "../clustering/ClusteringGpu.cuh"
#include "../clustering/ClusteringCpu.h"
#include "../../structures/ScyTreeNode.h"
#include "../../structures/ScyTreeArray.h"
#include "../../utils/util.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#include <map>
#include <vector>

using namespace std;

void InscyCpuGpuMix(ScyTreeNode *scy_tree, ScyTreeNode *neighborhood_tree, at::Tensor X, float *d_X, int n, int d,
                    float neighborhood_size, int *subspace,
                    int subspace_size, float F, int num_obj, int min_size,
                    map <vector<int>, vector<int>, vec_cmp> &result,
                    int first_dim_no,
                    int total_number_of_dim, int &calls) {
    int dim_no = first_dim_no;
    calls++;
    while (dim_no < total_number_of_dim) {
        int cell_no = 0;
        while (cell_no < scy_tree->get_number_of_cells()) {
            //restricted-tree := restrict(scy-tree, descriptor);
            ScyTreeNode *restricted_scy_tree = dynamic_cast<ScyTreeNode *>(scy_tree->restrict(dim_no, cell_no));

            //restricted-tree := mergeWithNeighbors(restricted-tree);
            //updates cell_no if merged with neighbors
            restricted_scy_tree->mergeWithNeighbors(scy_tree, dim_no, cell_no);

            //pruneRecursion(restricted-tree); //prune sparse regions
            if (restricted_scy_tree->pruneRecursion(min_size, neighborhood_tree, X, neighborhood_size,
                                                    restricted_scy_tree->restricted_dims,
                                                    restricted_scy_tree->number_of_restricted_dims, F, num_obj, n,
                                                    subspace_size)) {

                //INSCY(restricted-tree,result); //depth-first via recursion
                map <vector<int>, vector<int>, vec_cmp> sub_result;
                InscyCpuGpuMix(restricted_scy_tree, neighborhood_tree, X, d_X, n, d, neighborhood_size, subspace,
                               subspace_size, F,
                               num_obj, min_size, sub_result, dim_no + 1, total_number_of_dim, calls);
                result.insert(sub_result.begin(), sub_result.end());

                //pruneRedundancy(restricted-tree); //in-process-removal
                if (restricted_scy_tree->pruneRedundancy(1.1, sub_result)) {


                    //result := DBClustering(restricted-tree) ∪ result;
                    int idx = restricted_scy_tree->get_dims_idx();

                    ScyTreeArray *restricted_scy_tree_gpu = restricted_scy_tree->convert_to_ScyTreeArray();
                    restricted_scy_tree_gpu->copy_to_device();

                    vector<int> subspace_clustering = ClusteringGPU(restricted_scy_tree_gpu, d_X, n, d,
                                                                    neighborhood_size,
                                                                    F, num_obj);
                    vector<int> subspace(restricted_scy_tree_gpu->h_restricted_dims,
                                         restricted_scy_tree_gpu->h_restricted_dims +
                                         restricted_scy_tree_gpu->number_of_restricted_dims);


                    if (result.count(subspace)) {
                        vector<int> clustering = result[subspace];
                        int m = v_max(clustering);
                        for (int i = 0; i < n; i++) {
                            if (subspace_clustering[i] == -2) {
                                clustering[i] = subspace_clustering[i];
                            } else if (subspace_clustering[i] >= 0) {
                                clustering[i] = subspace_clustering[i];
                            }
                        }
                        result[subspace] = clustering;
                    } else {
                        result.insert(pair < vector < int > , vector < int >> (subspace, subspace_clustering));
                    }


//                if (result.count(idx)) {
//                    vector<int> clustering = result[idx];
//                    int m = v_max(clustering);
//                    if (m < 0) {
//                        result[idx] = new_clustering;
//                    } else {
//                        for (int i = 0; i < n; i++) {
//                            if (new_clustering[i] == -2) {
//                                clustering[i] = new_clustering[i];
//                            } else if (new_clustering[i] >= 0) {
//                                clustering[i] = m + 1 + new_clustering[i];
//                            }
//                        }
//                        result[idx] = clustering;
//                    }
//                } else {
//                    result.insert(pair < int, vector < int >> (idx, new_clustering));
//                }
                }
            }
            cell_no++;
        }

        dim_no++;
    }
    int total_inscy = pow(2, total_number_of_dim);
    printf("InscyCpuGpuMix(%d): %d%%      \r", calls, int((result.size() * 100) / total_inscy));
}