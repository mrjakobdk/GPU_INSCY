#include "hip/hip_runtime.h"
//
// Created by mrjakobdk on 6/8/20.
//

#include "InscyArrayGpuMulti2.cuh"
#include "../clustering/ClusteringGpu.cuh"
#include "../../structures/ScyTreeArray.h"
#include "../../utils/util.h"
#include "../../utils/TmpMalloc.cuh"


#include <math.h>
#include <map>
#include <vector>
#include "nvToolsExt.h"

using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

void free_tree(TmpMalloc *tmps, ScyTreeArray *&restricted_scy_tree, int *&d_new_neighborhoods,
               int *&d_new_neighborhood_end);

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

void
InscyArrayGpuMulti2(TmpMalloc *tmps, ScyTreeArray *scy_tree, float *d_X, int n, int d, float neighborhood_size, float F,
                    int num_obj,
                    int min_size, map <vector<int>, vector<int>, vec_cmp> &result, int first_dim_no,
                    int total_number_of_dim, float r, int &calls) {
    calls++;
    int total_inscy = pow(2, d);
    printf("InscyArrayGpuMulti2(%d): %d%%      \r", calls, int((calls * 100) / total_inscy));

    int number_of_dims = total_number_of_dim - first_dim_no;
    int number_of_cells = scy_tree->number_of_cells;


    nvtxRangePushA("restrict_merge_gpu_multi");
    vector <vector<ScyTreeArray *>> L_merged = scy_tree->restrict_merge_gpu_multi2(tmps, first_dim_no, number_of_dims,
                                                                                   number_of_cells);
    hipDeviceSynchronize();
    nvtxRangePop();

    int dim_no = first_dim_no;
    while (dim_no < total_number_of_dim) {

        vector<int> subspace;
        int *d_clustering;// = tmps->get_int_array(tmps->CLUSTERING, n); // number_of_points
        hipMalloc(&d_clustering, sizeof(int) * n);
        hipMemset(d_clustering, -1, sizeof(int) * n);

        int i = dim_no - first_dim_no;
        for (ScyTreeArray *restricted_scy_tree : L_merged[i]) {


            hipMemcpy(restricted_scy_tree->h_restricted_dims, restricted_scy_tree->d_restricted_dims,
                       sizeof(int) * restricted_scy_tree->number_of_restricted_dims, hipMemcpyDeviceToHost);
            subspace = vector<int>(restricted_scy_tree->h_restricted_dims,
                                   restricted_scy_tree->h_restricted_dims +
                                   restricted_scy_tree->number_of_restricted_dims);

            //pruneRecursion(restricted-tree); //prune sparse regions
            if (restricted_scy_tree->pruneRecursion_gpu(min_size, d_X, n, d, neighborhood_size, F, num_obj)) {

                //INSCY(restricted-tree,result); //depth-first via recursion
                map <vector<int>, vector<int>, vec_cmp> sub_result;
                InscyArrayGpuMulti2(tmps, restricted_scy_tree, d_X, n, d, neighborhood_size,
                                    F, num_obj, min_size, sub_result, dim_no + 1, total_number_of_dim, r, calls);
                result.insert(sub_result.begin(), sub_result.end());

                //pruneRedundancy(restricted-tree); //in-process-removal
                if (restricted_scy_tree->pruneRedundancy_gpu(r, result)) {

                    nvtxRangePushA("clustering");
                    ClusteringGPU(tmps, d_clustering, restricted_scy_tree, d_X, n, d, neighborhood_size,
                                  F, num_obj);
                    hipDeviceSynchronize();
                    nvtxRangePop();
                }
            }
            delete restricted_scy_tree;
        }


        nvtxRangePushA("joining");
        int *h_clustering = new int[n];
        hipMemcpy(h_clustering, d_clustering,
                   sizeof(int) * n, hipMemcpyDeviceToHost);
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
        vector<int> subspace_clustering(h_clustering, h_clustering + n);

        join(result, subspace_clustering, subspace, min_size, r);

        hipFree(d_clustering);

        hipDeviceSynchronize();
        nvtxRangePop();

        dim_no++;
    }
    gpuErrchk(hipPeekAtLastError());

}


void
InscyArrayGpuMulti2All(int *d_neighborhoods, int *d_neighborhood_end, TmpMalloc *tmps, ScyTreeArray *scy_tree,
                       float *d_X, int n, int d, float neighborhood_size, float F,
                       int num_obj,
                       int min_size, map <vector<int>, vector<int>, vec_cmp> &result, int first_dim_no,
                       int total_number_of_dim, float r, int &calls) {
    calls++;
    int total_inscy = pow(2, d);
    printf("InscyArrayGpuMulti2(%d): %d%%      \r", calls, int((calls * 100) / total_inscy));

    int number_of_dims = total_number_of_dim - first_dim_no;
    int number_of_cells = scy_tree->number_of_cells;


    nvtxRangePushA("restrict_merge_gpu_multi");
    vector <vector<ScyTreeArray *>> L_merged = scy_tree->restrict_merge_gpu_multi2(tmps, first_dim_no, number_of_dims,
                                                                                   number_of_cells);
    hipDeviceSynchronize();
    nvtxRangePop();

    int dim_no = first_dim_no;
    while (dim_no < total_number_of_dim) {

        vector<int> subspace;
        int *d_clustering;// = tmps->get_int_array(tmps->CLUSTERING, n); // number_of_points
        hipMalloc(&d_clustering, sizeof(int) * n);
        hipMemset(d_clustering, -1, sizeof(int) * n);

        int i = dim_no - first_dim_no;
        for (ScyTreeArray *restricted_scy_tree : L_merged[i]) {


            hipMemcpy(restricted_scy_tree->h_restricted_dims, restricted_scy_tree->d_restricted_dims,
                       sizeof(int) * restricted_scy_tree->number_of_restricted_dims, hipMemcpyDeviceToHost);
            subspace = vector<int>(restricted_scy_tree->h_restricted_dims,
                                   restricted_scy_tree->h_restricted_dims +
                                   restricted_scy_tree->number_of_restricted_dims);

            //pruneRecursion(restricted-tree); //prune sparse regions
            if (restricted_scy_tree->pruneRecursion_gpu(min_size, d_X, n, d, neighborhood_size, F, num_obj)) {

                //INSCY(restricted-tree,result); //depth-first via recursion
                map <vector<int>, vector<int>, vec_cmp> sub_result;
                InscyArrayGpuMulti2All(d_neighborhoods, d_neighborhood_end, tmps, restricted_scy_tree, d_X, n, d,
                                       neighborhood_size,
                                       F, num_obj, min_size, sub_result, dim_no + 1, total_number_of_dim, r, calls);
                result.insert(sub_result.begin(), sub_result.end());

                //pruneRedundancy(restricted-tree); //in-process-removal
                if (restricted_scy_tree->pruneRedundancy_gpu(r, result)) {

                    nvtxRangePushA("clustering");
                    ClusteringGPUAll(d_neighborhoods, d_neighborhood_end, tmps, d_clustering, restricted_scy_tree, d_X,
                                     n, d, neighborhood_size,
                                     F, num_obj);
                    hipDeviceSynchronize();
                    nvtxRangePop();
                }
            }
            delete restricted_scy_tree;
        }


        nvtxRangePushA("joining");
        int *h_clustering = new int[n];
        hipMemcpy(h_clustering, d_clustering,
                   sizeof(int) * n, hipMemcpyDeviceToHost);
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
        vector<int> subspace_clustering(h_clustering, h_clustering + n);

        join(result, subspace_clustering, subspace, min_size, r);

        hipFree(d_clustering);

        hipDeviceSynchronize();
        nvtxRangePop();

        dim_no++;
    }
    gpuErrchk(hipPeekAtLastError());

}


void
InscyArrayGpuMulti2ReAll(int *d_neighborhoods, int *d_neighborhood_end, TmpMalloc *tmps, ScyTreeArray *scy_tree,
                         float *d_X, int n, int d, float neighborhood_size, float F,
                         int num_obj,
                         int min_size, map <vector<int>, vector<int>, vec_cmp> &result, int first_dim_no,
                         int total_number_of_dim, float r, int &calls) {
    calls++;
    int total_inscy = pow(2, d);
    printf("InscyArrayGpuMulti2ReAll(%d): %d%%      \r", calls, int((calls * 100) / total_inscy));

    int number_of_dims = total_number_of_dim - first_dim_no;
    int number_of_cells = scy_tree->number_of_cells;

    gpuErrchk(hipPeekAtLastError());

    nvtxRangePushA("restrict_merge_gpu_multi");
    vector <vector<ScyTreeArray *>> L_merged = scy_tree->restrict_merge_gpu_multi2(tmps, first_dim_no, number_of_dims,
                                                                                   number_of_cells);


    gpuErrchk(hipPeekAtLastError());

    hipDeviceSynchronize();
    nvtxRangePop();

    int dim_no = first_dim_no;
    while (dim_no < total_number_of_dim) {

        vector<int> subspace;
        int *d_clustering;// = tmps->get_int_array(tmps->CLUSTERING, n); // number_of_points
        hipMalloc(&d_clustering, sizeof(int) * n);
        hipMemset(d_clustering, -1, sizeof(int) * n);

        int i = dim_no - first_dim_no;
        for (ScyTreeArray *restricted_scy_tree : L_merged[i]) {


            gpuErrchk(hipPeekAtLastError());
            hipMemcpy(restricted_scy_tree->h_restricted_dims, restricted_scy_tree->d_restricted_dims,
                       sizeof(int) * restricted_scy_tree->number_of_restricted_dims, hipMemcpyDeviceToHost);
            subspace = vector<int>(restricted_scy_tree->h_restricted_dims,
                                   restricted_scy_tree->h_restricted_dims +
                                   restricted_scy_tree->number_of_restricted_dims);


            int *d_new_neighborhoods;
            int *d_new_neighborhood_sizes;
            int *d_new_neighborhood_end;

            gpuErrchk(hipPeekAtLastError());
            find_neighborhoods_re(d_neighborhoods, d_neighborhood_end,
                                  d_new_neighborhoods, d_new_neighborhood_end, d_new_neighborhood_sizes,
                                  d_X, n, d, scy_tree, restricted_scy_tree, neighborhood_size);
            gpuErrchk(hipPeekAtLastError());

            //pruneRecursion(restricted-tree); //prune sparse regions
            if (restricted_scy_tree->pruneRecursion_gpu(min_size, d_X, n, d, neighborhood_size, F, num_obj)) {

                //INSCY(restricted-tree,result); //depth-first via recursion
                map <vector<int>, vector<int>, vec_cmp> sub_result;
                InscyArrayGpuMulti2ReAll(d_new_neighborhoods, d_new_neighborhood_end, tmps, restricted_scy_tree, d_X, n,
                                         d,
                                         neighborhood_size,
                                         F, num_obj, min_size, sub_result, dim_no + 1, total_number_of_dim, r, calls);
                result.insert(sub_result.begin(), sub_result.end());

                //pruneRedundancy(restricted-tree); //in-process-removal
                if (restricted_scy_tree->pruneRedundancy_gpu(r, result)) {

                    nvtxRangePushA("clustering");
                    gpuErrchk(hipPeekAtLastError());
                    ClusteringGPUReAll(d_new_neighborhoods, d_new_neighborhood_end, tmps, d_clustering,
                                       restricted_scy_tree,
                                       d_X, n, d, neighborhood_size,
                                       F, num_obj, false);
                    hipDeviceSynchronize();
                    gpuErrchk(hipPeekAtLastError());
                    nvtxRangePop();
                }
            }
            if (restricted_scy_tree->number_of_points > 0) {
                hipFree(d_new_neighborhoods);
                gpuErrchk(hipPeekAtLastError());
            }
            hipFree(d_new_neighborhood_sizes);
            gpuErrchk(hipPeekAtLastError());
            hipFree(d_new_neighborhood_end);
            gpuErrchk(hipPeekAtLastError());
            delete restricted_scy_tree;
            gpuErrchk(hipPeekAtLastError());
        }


        nvtxRangePushA("joining");
        int *h_clustering = new int[n];
        hipMemcpy(h_clustering, d_clustering,
                   sizeof(int) * n, hipMemcpyDeviceToHost);
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
        vector<int> subspace_clustering(h_clustering, h_clustering + n);

        join(result, subspace_clustering, subspace, min_size, r);

        hipFree(d_clustering);

        hipDeviceSynchronize();
        nvtxRangePop();

        dim_no++;
    }
    gpuErrchk(hipPeekAtLastError());

}


void
InscyArrayGpuMulti2Weak(int *d_neighborhoods, int *d_neighborhood_end, TmpMalloc *tmps, ScyTreeArray *scy_tree,
                        float *d_X, int n, int d, float neighborhood_size, float F,
                        int num_obj,
                        int min_size, map <vector<int>, vector<int>, vec_cmp> &result, int first_dim_no,
                        int total_number_of_dim, float r, int &calls) {
    calls++;
    int total_inscy = pow(2, d);
    printf("InscyArrayGpuMulti2Weak(%d): %d%%      \r", calls, int((calls * 100) / total_inscy));

    int number_of_dims = total_number_of_dim - first_dim_no;
    int number_of_cells = scy_tree->number_of_cells;

    gpuErrchk(hipPeekAtLastError());

    nvtxRangePushA("restrict_merge_gpu_multi");
    vector <vector<ScyTreeArray *>> L_merged = scy_tree->restrict_merge_gpu_multi2(tmps, first_dim_no, number_of_dims,
                                                                                   number_of_cells);


    gpuErrchk(hipPeekAtLastError());

    hipDeviceSynchronize();
    nvtxRangePop();

    int dim_no = first_dim_no;
    while (dim_no < total_number_of_dim) {

        vector<int> subspace;
        int *d_clustering;// = tmps->get_int_array(tmps->CLUSTERING, n); // number_of_points
        hipMalloc(&d_clustering, sizeof(int) * n);
        hipMemset(d_clustering, -1, sizeof(int) * n);

        int i = dim_no - first_dim_no;
//        if(L_merged[i].size()>1){
//            printf("2 We did split it! %d\n", L_merged[i].size());
//        }
        for (ScyTreeArray *restricted_scy_tree : L_merged[i]) {


            gpuErrchk(hipPeekAtLastError());
            hipMemcpy(restricted_scy_tree->h_restricted_dims, restricted_scy_tree->d_restricted_dims,
                       sizeof(int) * restricted_scy_tree->number_of_restricted_dims, hipMemcpyDeviceToHost);
            subspace = vector<int>(restricted_scy_tree->h_restricted_dims,
                                   restricted_scy_tree->h_restricted_dims +
                                   restricted_scy_tree->number_of_restricted_dims);


            int *d_new_neighborhoods;
            int *d_new_neighborhood_sizes;
            int *d_new_neighborhood_end;

            gpuErrchk(hipPeekAtLastError());
            find_neighborhoods_re(d_neighborhoods, d_neighborhood_end,
                                  d_new_neighborhoods, d_new_neighborhood_end, d_new_neighborhood_sizes,
                                  d_X, n, d, scy_tree, restricted_scy_tree, neighborhood_size);
            gpuErrchk(hipPeekAtLastError());

            //pruneRecursion(restricted-tree); //prune sparse regions
            if (restricted_scy_tree->pruneRecursionAndRemove_gpu(min_size, d_X, n, d, neighborhood_size, F, num_obj,
                                                                 d_new_neighborhoods, d_new_neighborhood_end)) {

                //INSCY(restricted-tree,result); //depth-first via recursion
                map <vector<int>, vector<int>, vec_cmp> sub_result;
                InscyArrayGpuMulti2Weak(d_new_neighborhoods, d_new_neighborhood_end, tmps, restricted_scy_tree, d_X, n,
                                        d,
                                        neighborhood_size,
                                        F, num_obj, min_size, sub_result, dim_no + 1, total_number_of_dim, r, calls);
                result.insert(sub_result.begin(), sub_result.end());

                //pruneRedundancy(restricted-tree); //in-process-removal
                if (restricted_scy_tree->pruneRedundancy_gpu(r, result)) {

                    nvtxRangePushA("clustering");
                    gpuErrchk(hipPeekAtLastError());
                    ClusteringGPUReAll(d_new_neighborhoods, d_new_neighborhood_end, tmps, d_clustering,
                                       restricted_scy_tree,
                                       d_X, n, d, neighborhood_size,
                                       F, num_obj, false);
                    hipDeviceSynchronize();
                    gpuErrchk(hipPeekAtLastError());
                    nvtxRangePop();
                }
            }
            if (restricted_scy_tree->number_of_points > 0) {
                hipFree(d_new_neighborhoods);
                gpuErrchk(hipPeekAtLastError());
            }
            hipFree(d_new_neighborhood_sizes);
            gpuErrchk(hipPeekAtLastError());
            hipFree(d_new_neighborhood_end);
            gpuErrchk(hipPeekAtLastError());
            delete restricted_scy_tree;
            gpuErrchk(hipPeekAtLastError());
        }


        nvtxRangePushA("joining");
        int *h_clustering = new int[n];
        hipMemcpy(h_clustering, d_clustering,
                   sizeof(int) * n, hipMemcpyDeviceToHost);
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
        vector<int> subspace_clustering(h_clustering, h_clustering + n);

        join(result, subspace_clustering, subspace, min_size, r);

        hipFree(d_clustering);

        hipDeviceSynchronize();
        nvtxRangePop();

        dim_no++;
    }
    gpuErrchk(hipPeekAtLastError());
}


void
InscyArrayGpuMulti3Weak(int *d_neighborhoods, int *d_neighborhood_end, TmpMalloc *tmps, ScyTreeArray *scy_tree,
                        float *d_X, int n, int d, float neighborhood_size, float F,
                        int num_obj,
                        int min_size, map <vector<int>, vector<int>, vec_cmp> &result, int first_dim_no,
                        int total_number_of_dim, float r, int &calls, bool rectangular) {
    calls++;
    int total_inscy = pow(2, d);
    printf("InscyArrayGpuMulti3Weak(%d): %d%%      \r", calls, int((calls * 100) / total_inscy));

//    printf("\nsubspace: %d\n", scy_tree->number_of_points);
//    print_array_gpu<< <1,1>>>(scy_tree->d_restricted_dims, scy_tree->number_of_restricted_dims);

    int number_of_dims = total_number_of_dim - first_dim_no;
    int number_of_cells = scy_tree->number_of_cells;

    gpuErrchk(hipPeekAtLastError());

    nvtxRangePushA("restrict_merge_gpu_multi");
    vector <vector<ScyTreeArray *>> L_merged = scy_tree->restrict_merge_gpu_multi3(tmps, first_dim_no, number_of_dims,
                                                                                   number_of_cells);


    gpuErrchk(hipPeekAtLastError());

    hipDeviceSynchronize();
    nvtxRangePop();

    int dim_no = first_dim_no;
    while (dim_no < total_number_of_dim) {

        vector<int> subspace;
        int *d_clustering;// = tmps->get_int_array(tmps->CLUSTERING, n); // number_of_points
        hipMalloc(&d_clustering, sizeof(int) * n);
        hipMemset(d_clustering, -1, sizeof(int) * n);

        int i = dim_no - first_dim_no;
//        if(L_merged[i].size()>1){
//            printf("3 We did split it! %d\n", L_merged[i].size());
//        }
        for (ScyTreeArray *restricted_scy_tree : L_merged[i]) {


            gpuErrchk(hipPeekAtLastError());
            hipMemcpy(restricted_scy_tree->h_restricted_dims, restricted_scy_tree->d_restricted_dims,
                       sizeof(int) * restricted_scy_tree->number_of_restricted_dims, hipMemcpyDeviceToHost);
            subspace = vector<int>(restricted_scy_tree->h_restricted_dims,
                                   restricted_scy_tree->h_restricted_dims +
                                   restricted_scy_tree->number_of_restricted_dims);


            int *d_new_neighborhoods;
            int *d_new_neighborhood_sizes;
            int *d_new_neighborhood_end;

            nvtxRangePushA("find_neighborhoods_re");
            gpuErrchk(hipPeekAtLastError());
            find_neighborhoods_re(d_neighborhoods, d_neighborhood_end,
                                  d_new_neighborhoods, d_new_neighborhood_end, d_new_neighborhood_sizes,
                                  d_X, n, d, scy_tree, restricted_scy_tree, neighborhood_size);
            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());
            nvtxRangePop();

            //pruneRecursion(restricted-tree); //prune sparse regions
            nvtxRangePushA("pruneRecursion");
            bool pruneRecursion = restricted_scy_tree->pruneRecursionAndRemove_gpu3(min_size, d_X, n, d,
                                                                                    neighborhood_size, F, num_obj,
                                                                                    d_new_neighborhoods,
                                                                                    d_new_neighborhood_end,
                                                                                    rectangular);
            nvtxRangePop();
            if (pruneRecursion) {

                //INSCY(restricted-tree,result); //depth-first via recursion
                InscyArrayGpuMulti3Weak(d_new_neighborhoods, d_new_neighborhood_end, tmps, restricted_scy_tree, d_X, n,
                                        d,
                                        neighborhood_size,
                                        F, num_obj, min_size, result, dim_no + 1, total_number_of_dim, r, calls,
                                        rectangular);

                //pruneRedundancy(restricted-tree); //in-process-removal
                nvtxRangePushA("pruneRedundancy");
//                bool pruneRedundancy = restricted_scy_tree->pruneRedundancy_gpu1(r, result, n);
                bool pruneRedundancy = restricted_scy_tree->pruneRedundancy_gpu(r, result);
//                pruneRedundancy = true;
                nvtxRangePop();
                if (pruneRedundancy) {

                    nvtxRangePushA("clustering");
                    gpuErrchk(hipPeekAtLastError());
                    ClusteringGPUReAll(d_new_neighborhoods, d_new_neighborhood_end, tmps, d_clustering,
                                       restricted_scy_tree,
                                       d_X, n, d, neighborhood_size,
                                       F, num_obj, rectangular);
                    hipDeviceSynchronize();
                    gpuErrchk(hipPeekAtLastError());
                    nvtxRangePop();
                } else {
                    printf("pruned due to prune Redundancy_gpu\n");
                }
            }
            if (restricted_scy_tree->number_of_points > 0) {
                hipFree(d_new_neighborhoods);
                gpuErrchk(hipPeekAtLastError());
            }
            hipFree(d_new_neighborhood_sizes);
            gpuErrchk(hipPeekAtLastError());
            hipFree(d_new_neighborhood_end);
            gpuErrchk(hipPeekAtLastError());
            delete restricted_scy_tree;
            gpuErrchk(hipPeekAtLastError());
        }


        nvtxRangePushA("joining");
        int *h_clustering = new int[n];
        hipMemcpy(h_clustering, d_clustering,
                   sizeof(int) * n, hipMemcpyDeviceToHost);
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
        vector<int> subspace_clustering(h_clustering, h_clustering + n);

        join(result, subspace_clustering, subspace, min_size, r);

//        join_gpu1(result, subspace_clustering, d_clustering, subspace, min_size, r, n);

        hipFree(d_clustering);

        hipDeviceSynchronize();
        nvtxRangePop();

        dim_no++;
    }
    gpuErrchk(hipPeekAtLastError());
}


void
InscyArrayGpu4(int *d_neighborhoods, int *d_neighborhood_end, TmpMalloc *tmps, ScyTreeArray *scy_tree,
               float *d_X, int n, int d, float neighborhood_size, float F,
               int num_obj,
               int min_size, map<vector<int>, int *, vec_cmp> &result, int first_dim_no,
               int total_number_of_dim, float r, int &calls, bool rectangular) {
    calls++;
    int total_inscy = pow(2, d);
    printf("InscyArrayGpu4(%d): %d%%      \r", calls, int((calls * 100) / total_inscy));

//    printf("\nsubspace: %d\n", scy_tree->number_of_points);
//    print_array_gpu<< <1,1>>>(scy_tree->d_restricted_dims, scy_tree->number_of_restricted_dims);

    int number_of_dims = total_number_of_dim - first_dim_no;
    int number_of_cells = scy_tree->number_of_cells;

    gpuErrchk(hipPeekAtLastError());

//    hipDeviceSynchronize();
//    nvtxRangePushA("restrict_merge_gpu_multi4");
    vector <vector<ScyTreeArray *>> L_merged = scy_tree->restrict_merge_gpu_multi4(tmps, first_dim_no, number_of_dims,
                                                                                   number_of_cells);
//    gpuErrchk(hipPeekAtLastError());
//    hipDeviceSynchronize();
//    nvtxRangePop();

    int dim_no = first_dim_no;
    while (dim_no < total_number_of_dim) {

        vector<int> subspace;
        int *d_clustering = tmps->malloc_points();// number_of_points
//        hipMalloc(&d_clustering, sizeof(int) * n);
        hipMemset(d_clustering, -1, sizeof(int) * n);

        int i = dim_no - first_dim_no;
//        if(L_merged[i].size()>1){
//            printf("3 We did split it! %d\n", L_merged[i].size());
//        }
        for (ScyTreeArray *restricted_scy_tree : L_merged[i]) {


            gpuErrchk(hipPeekAtLastError());
//            hipMemcpy(restricted_scy_tree->h_restricted_dims, restricted_scy_tree->d_restricted_dims,
//                       sizeof(int) * restricted_scy_tree->number_of_restricted_dims, hipMemcpyDeviceToHost);
            subspace = vector<int>(restricted_scy_tree->h_restricted_dims,
                                   restricted_scy_tree->h_restricted_dims +
                                   restricted_scy_tree->number_of_restricted_dims);


            int *d_new_neighborhoods;
            int *d_new_neighborhood_sizes;
            int *d_new_neighborhood_end;

//            nvtxRangePushA("find_neighborhoods_re4");
//            gpuErrchk(hipPeekAtLastError());
            find_neighborhoods_re4(tmps, d_neighborhoods, d_neighborhood_end,
                                   d_new_neighborhoods, d_new_neighborhood_end, d_new_neighborhood_sizes,
                                   d_X, n, d, scy_tree, restricted_scy_tree, neighborhood_size);
//            hipDeviceSynchronize();
//            gpuErrchk(hipPeekAtLastError());
//            nvtxRangePop();

//            hipDeviceSynchronize();
//            nvtxRangePushA("pruneRecursionAndRemove_gpu4");
            //pruneRecursion(restricted-tree); //prune sparse regions
            bool pruneRecursion = restricted_scy_tree->pruneRecursionAndRemove_gpu4(tmps, min_size, d_X, n, d,
                                                                                    neighborhood_size, F, num_obj,
                                                                                    d_new_neighborhoods,
                                                                                    d_new_neighborhood_end,
                                                                                    rectangular);
//            hipDeviceSynchronize();
//            nvtxRangePop();

            if (pruneRecursion) {

                //INSCY(restricted-tree,result); //depth-first via recursion
                InscyArrayGpu4(d_new_neighborhoods, d_new_neighborhood_end, tmps, restricted_scy_tree, d_X, n,
                               d,
                               neighborhood_size,
                               F, num_obj, min_size, result, dim_no + 1, total_number_of_dim, r, calls, rectangular);

                //pruneRedundancy(restricted-tree); //in-process-removal
//                hipDeviceSynchronize();
//                nvtxRangePushA("pruneRedundancy_gpu2");
//                bool pruneRedundancy = restricted_scy_tree->pruneRedundancy_gpu(r, result);
                bool pruneRedundancy = restricted_scy_tree->pruneRedundancy_gpu2(r, result, n, tmps);
//                pruneRecursion = true;
//                hipDeviceSynchronize();
//                nvtxRangePop();
                if (pruneRedundancy) {

//                    nvtxRangePushA("ClusteringGPUReAll");
//                    gpuErrchk(hipPeekAtLastError());
                    ClusteringGPUReAll(d_new_neighborhoods, d_new_neighborhood_end, tmps, d_clustering,
                                       restricted_scy_tree,
                                       d_X, n, d, neighborhood_size,
                                       F, num_obj, rectangular);
//                    hipDeviceSynchronize();
//                    gpuErrchk(hipPeekAtLastError());
//                    nvtxRangePop();
                } else {
                    printf("pruned due to prune Redundancy_gpu\n");
                }
            }
            if (restricted_scy_tree->number_of_points > 0) {
                hipDeviceSynchronize();
                hipFree(d_new_neighborhoods);
                gpuErrchk(hipPeekAtLastError());
            }
//            hipFree(d_new_neighborhood_sizes);
            tmps->free_points(d_new_neighborhood_sizes);
            gpuErrchk(hipPeekAtLastError());
//            hipFree(d_new_neighborhood_end);
            tmps->free_points(d_new_neighborhood_end);
            gpuErrchk(hipPeekAtLastError());
            delete restricted_scy_tree;
            gpuErrchk(hipPeekAtLastError());
        }


//        hipDeviceSynchronize();
//        nvtxRangePushA("join_gpu");
//        int *h_clustering = new int[n];
//        hipMemcpy(h_clustering, d_clustering,
//                   sizeof(int) * n, hipMemcpyDeviceToHost);
////        hipDeviceSynchronize();
//        gpuErrchk(hipPeekAtLastError());
//        vector<int> subspace_clustering(h_clustering, h_clustering + n);

//        join_gpu(result, subspace_clustering, subspace, min_size, r, n);
//        join(result, subspace_clustering, subspace, min_size, r);
        join_gpu(result, d_clustering, subspace, min_size, r, n, tmps);
        gpuErrchk(hipPeekAtLastError());

//        hipFree(d_clustering);
//        hipDeviceSynchronize();
//        nvtxRangePop();

        dim_no++;
    }
    gpuErrchk(hipPeekAtLastError());
}

void compare_gpu(int *d_tmp, int *d_correct, int n) {
    int h_tmp[n];
    int h_correct[n];
    hipMemcpy(h_tmp, d_tmp, n * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_correct, d_correct, n * sizeof(int), hipMemcpyDeviceToHost);
    int m = 0;
    int incorrect = 0;
    int correct = 0;
    for (int i = 0; i < n; i++) {
        if (h_tmp[i] != h_correct[i]) {
            incorrect++;
        } else {
            correct++;
            m = max(m, h_correct[i]);
        }
    }
    if (incorrect > 0)
        printf("correct! max:%d, correct: %d, incorrect:%d\n", m, correct, incorrect);
}

void
InscyArrayGpu5(int *d_neighborhoods, int *d_neighborhood_end, TmpMalloc *tmps, ScyTreeArray *scy_tree,
               float *d_X, int n, int d, float neighborhood_size, float F,
               int num_obj,
               int min_size, map<vector<int>, int *, vec_cmp> &result, int first_dim_no,
               int total_number_of_dim, float r, int &calls, bool rectangular) {
    calls++;
    int total_inscy = pow(2, d);
    printf("InscyArrayGpu5(%d): %d%%      \r", calls, int((calls * 100) / total_inscy));

    int number_of_dims = total_number_of_dim - first_dim_no;
    int number_of_cells = scy_tree->number_of_cells;

    vector <vector<ScyTreeArray *>> L_merged = scy_tree->restrict_merge_gpu_multi5(tmps, first_dim_no, number_of_dims,
                                                                                   number_of_cells);

    vector < ScyTreeArray * > restricted_scy_tree_list;
    vector<int *> clustering_list;
    vector<int *> new_neighborhoods_list;
    vector<int *> new_neighborhood_end_list;

    pair<int **, int **> p = find_neighborhoods_re5(tmps, d_neighborhoods, d_neighborhood_end,
                                                    d_X, n, d, scy_tree, L_merged, neighborhood_size);

    int **hd_new_neighborhoods_list = p.first;
    int **hd_new_neighborhood_end_list = p.second;


    int j = 0;
    int dim_no = first_dim_no;
    while (dim_no < total_number_of_dim) {

        int *d_clustering = tmps->malloc_points();
        hipMemset(d_clustering, -1, sizeof(int) * n);

        bool clustering_used = false;

        int i = dim_no - first_dim_no;
        for (ScyTreeArray *restricted_scy_tree : L_merged[i]) {

            int *d_new_neighborhoods = hd_new_neighborhoods_list[j];
            int *d_new_neighborhood_end = hd_new_neighborhood_end_list[j];
            j++;

            bool pruneRecursion = restricted_scy_tree->pruneRecursionAndRemove_gpu4(tmps, min_size, d_X, n, d,
                                                                                    neighborhood_size, F, num_obj,
                                                                                    d_new_neighborhoods,
                                                                                    d_new_neighborhood_end,
                                                                                    rectangular);

            if (pruneRecursion) {

                InscyArrayGpu5(d_new_neighborhoods, d_new_neighborhood_end, tmps, restricted_scy_tree,
                               d_X, n, d, neighborhood_size, F, num_obj, min_size,
                               result, dim_no + 1, total_number_of_dim, r, calls, rectangular);

                bool pruneRedundancy = restricted_scy_tree->pruneRedundancy_gpu2(r, result, n, tmps);
                if (pruneRedundancy) {

                    restricted_scy_tree_list.push_back(restricted_scy_tree);
                    clustering_list.push_back(d_clustering);
                    clustering_used = true;
                    new_neighborhoods_list.push_back(d_new_neighborhoods);
                    new_neighborhood_end_list.push_back(d_new_neighborhood_end);

                } else {
                    printf("pruned redundancy\n");
                    free_tree(tmps, restricted_scy_tree, d_new_neighborhoods, d_new_neighborhood_end);
                }
            } else {
                free_tree(tmps, restricted_scy_tree, d_new_neighborhoods, d_new_neighborhood_end);
            }
        }

        dim_no++;

        if (!clustering_used)
            tmps->free_points(d_clustering);
    }

    ClusteringGPUReAll5(new_neighborhoods_list, new_neighborhood_end_list, tmps,
                        clustering_list, restricted_scy_tree_list,
                        d_X, n, d, neighborhood_size,
                        F, num_obj, rectangular);

    for (int i = 0; i < restricted_scy_tree_list.size(); i++) {
        ScyTreeArray *restricted_scy_tree = restricted_scy_tree_list[i];
        int *d_clustering = clustering_list[i];
        int *d_new_neighborhoods = new_neighborhoods_list[i];
        int *d_new_neighborhood_end = new_neighborhood_end_list[i];

        if (i == restricted_scy_tree_list.size() - 1 ||
            (i < restricted_scy_tree_list.size() - 1 && d_clustering != clustering_list[i + 1])) {

            vector<int> subspace = vector<int>(restricted_scy_tree->h_restricted_dims,
                                               restricted_scy_tree->h_restricted_dims +
                                               restricted_scy_tree->number_of_restricted_dims);

            join_gpu(result, d_clustering, subspace, min_size, r, n, tmps);
        }
        free_tree(tmps, restricted_scy_tree, d_new_neighborhoods, d_new_neighborhood_end);
    }

}

void free_tree(TmpMalloc *tmps, ScyTreeArray *&restricted_scy_tree, int *&d_new_neighborhoods,
               int *&d_new_neighborhood_end) {
    //if (restricted_scy_tree->number_of_points > 0) {
//        hipDeviceSynchronize();
//        gpuErrchk(hipPeekAtLastError());
//        printf("test0.1\n");
        hipFree(d_new_neighborhoods);
//        printf("test0.2\n");
//        gpuErrchk(hipPeekAtLastError());
//    }
//    printf("test0.3\n");
    tmps->free_points(d_new_neighborhood_end);
//    printf("test0.4\n");
//    gpuErrchk(hipPeekAtLastError());
//    printf("test0.5\n");
    delete restricted_scy_tree;
//    printf("test0.6\n");
}
