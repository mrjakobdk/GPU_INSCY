//
// Created by mrjakobdk on 6/9/20.
//

#include "InscyArrayGpuMulti2ClMulti.cuh"
#include "../clustering/ClusteringGpuBlocks.cuh"
#include "../../structures/ScyTreeArray.h"
#include "../../utils/util.h"
#include "../../utils/TmpMalloc.cuh"


#include <math.h>
#include <map>
#include <vector>
#include "nvToolsExt.h"
#include "../clustering/ClusteringGpu.cuh"

using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

void
InscyArrayGpuMulti2ClMulti(TmpMalloc *tmps, ScyTreeArray *scy_tree, float *d_X, int n, int d, float neighborhood_size,
                           float F,
                           int num_obj,
                           int min_size, map <vector<int>, vector<int>, vec_cmp> &result, int first_dim_no,
                           int total_number_of_dim, float r, int &calls) {
    calls++;

    int number_of_dims = total_number_of_dim - first_dim_no;
    int number_of_cells = scy_tree->number_of_cells;


    nvtxRangePushA("restrict_merge_gpu_multi");
    vector <vector<ScyTreeArray *>> L_merged = scy_tree->restrict_merge_gpu_multi(tmps, first_dim_no, number_of_dims,
                                                                                  number_of_cells);
    hipDeviceSynchronize();
    nvtxRangePop();


    vector <vector<ScyTreeArray *>> L_pruned(number_of_dims);

    vector<int> subspace;
    int *d_clustering = tmps->d_clustering; // number_of_points
//        hipMalloc(&d_clustering, sizeof(int) * n);
    hipMemset(d_clustering, -1, sizeof(int) * n * number_of_dims);

    int dim_no = first_dim_no;
    while (dim_no < total_number_of_dim) {

        int i = dim_no - first_dim_no;
        for (ScyTreeArray *restricted_scy_tree : L_merged[i]) {



            //pruneRecursion(restricted-tree); //prune sparse regions
            if (restricted_scy_tree->pruneRecursion_gpu(min_size, d_X, n, d, neighborhood_size, F, num_obj)) {

                //INSCY(restricted-tree,result); //depth-first via recursion
                map <vector<int>, vector<int>, vec_cmp> sub_result;
                InscyArrayGpuMulti2ClMulti(tmps, restricted_scy_tree, d_X, n, d, neighborhood_size,
                                           F, num_obj, min_size, sub_result, dim_no + 1, total_number_of_dim, r, calls);
                result.insert(sub_result.begin(), sub_result.end());

                //pruneRedundancy(restricted-tree); //in-process-removal
                if (restricted_scy_tree->pruneRedundancy_gpu(r, result)) {

                    L_pruned[i].push_back(restricted_scy_tree);

                }
            }
//            delete restricted_scy_tree;
        }

        dim_no++;
    }


    nvtxRangePushA("clustering");
    ClusteringGPUBlocks(tmps, d_clustering, L_pruned, d_X, n, d, neighborhood_size,
                        F, num_obj, scy_tree->number_of_cells);
//    dim_no = first_dim_no;
//    while (dim_no < total_number_of_dim) {
//        int i = dim_no - first_dim_no;
//        for (int j = 0; j<L_pruned[i].size() ;j++) {
//            gpuErrchk(hipPeekAtLastError());
//
//            ScyTreeArray *restricted_scy_tree = L_pruned[i][j];
//            ClusteringGPU(tmps, d_clustering + i * n, restricted_scy_tree, d_X, n, d, neighborhood_size,
//                          F, num_obj);
//            hipDeviceSynchronize();
//        }
//        dim_no++;
//    }
    nvtxRangePop();


    nvtxRangePushA("joining");
    int *h_clustering = new int[n];
    dim_no = first_dim_no;
    while (dim_no < total_number_of_dim) {
        int i = dim_no - first_dim_no;
        if (L_pruned[i].size() > 0) {
            hipMemcpy(h_clustering, d_clustering + i * n,
                       sizeof(int) * n, hipMemcpyDeviceToHost);
            gpuErrchk(hipPeekAtLastError());

            ScyTreeArray *restricted_scy_tree = L_pruned[i][0];
            hipMemcpy(restricted_scy_tree->h_restricted_dims, restricted_scy_tree->d_restricted_dims,
                       sizeof(int) * restricted_scy_tree->number_of_restricted_dims, hipMemcpyDeviceToHost);
            subspace = vector<int>(restricted_scy_tree->h_restricted_dims,
                                   restricted_scy_tree->h_restricted_dims +
                                   restricted_scy_tree->number_of_restricted_dims);
//        hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());
            vector<int> subspace_clustering(h_clustering, h_clustering + n);

            join(result, subspace_clustering, subspace, min_size, r);
        }
        dim_no++;
    }
    hipDeviceSynchronize();
    nvtxRangePop();

    gpuErrchk(hipPeekAtLastError());

}
