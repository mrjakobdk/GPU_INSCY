#include "hip/hip_runtime.h"
#include "ScyTreeArray.h"
#include "../utils/RestrictUtils.h"
#include "../utils/MergeUtil.h"
#include "../utils/util.h"

#define BLOCKSIZE 16
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/*Check for safe return of all calls to the device */
void CUDA_SAFE_CALL(hipError_t call)
{
    hipError_t ret = call;
    //printf("RETURN FROM THE CUDA CALL:%d\t:",ret);
    switch(ret)
    {
        case hipSuccess:
            //              printf("Success\n");
            break;
            /*      case hipErrorInvalidValue:
                                    {
                                    printf("ERROR: InvalidValue:%i.\n",__LINE__);
                                    exit(-1);
                                    break;
                                    }
                    case hipErrorInvalidDevicePointer:
                                    {
                                    printf("ERROR:Invalid Device pointeri:%i.\n",__LINE__);
                                    exit(-1);
                                    break;
                                    }
                    case hipErrorInvalidMemcpyDirection:
                                    {
                                    printf("ERROR:Invalid memcpy direction:%i.\n",__LINE__);
                                    exit(-1);
                                    break;
                                    }                       */
        default:
        {
            printf(" ERROR at line :%i.%d' ' %s\n",__LINE__,ret,hipGetErrorString(ret));
            exit(-1);
            break;
        }
    }
}

__global__ void PrefixSum(int *dInArray, int *dOutArray, int arrayLen, int threadDim)
{
    //http://www.tezu.ernet.in/dcompsc/facility/HPCC/hypack/gpgpu-nvidia-cuda-prog-hypack-2013/gpu-comp-nvidia-cuda-num-comp-codes/cuda-prefix-sum.cu
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int tindex = (threadDim * tidx) + tidy;
    int maxNumThread = threadDim * threadDim;
    int pass = 0;
    int count ;
    int curEleInd;
    int tempResult = 0;

    while( (curEleInd = (tindex + maxNumThread * pass))  < arrayLen )
    {
        tempResult = 0;
        for( count = 0; count < curEleInd; count++)
            tempResult += dInArray[count];
        dOutArray[curEleInd] = tempResult;
        pass++;
    }
    __syncthreads();
}//end of Prefix sum function


#define BLOCK_WIDTH 64

void merge_using_gpu(int *d_parents_1, int *d_cells_1, int *d_counts_1,
                     int *d_dim_start_1, int *d_dims_1, int *d_restricted_dims_1,
                     int *d_points_1, int *d_points_placement_1,
                     int d_1, int n_1, int number_of_points_1, int number_of_restricted_dims_1,
                     int *d_parents_2, int *d_cells_2, int *d_counts_2,
                     int *d_dim_start_2, int *d_dims_2, int *d_restricted_dims_2,
                     int *d_points_2, int *d_points_placement_2,
                     int d_2, int n_2, int number_of_points_2, int number_of_restricted_dims_2,
                     int *&d_parents_3, int *&d_cells_3, int *&d_counts_3,
                     int *&d_dim_start_3, int *&d_dims_3, int *&d_restricted_dims_3,
                     int *&d_points_3, int *&d_points_placement_3,
                     int &d_3, int &n_3, int &number_of_points_3, int &number_of_restricted_dims_3) {

//    printf("d_1: %d, n_1:%d, points_1:%d\n", d_1, n_1, number_of_points_1);
//    printf("d_2: %d, n_2:%d, points_2:%d\n", d_2, n_2, number_of_points_2);

    gpuErrchk(hipPeekAtLastError());

    //compute sort keys for both using cell id cell_no and concat
    //sort - save permutation
    int n_total = n_1 + n_2;

    int numBlocks;

    int *d_map_to_old;
    int *d_map_to_new;
    int *d_is_included;
    int *d_new_indecies;
    hipMalloc(&d_map_to_new, n_total * sizeof(int));
    hipMemset(d_map_to_new, -1, n_total * sizeof(int));
    hipMalloc(&d_map_to_old, n_total * sizeof(int));
    hipMemset(d_map_to_old, -1, n_total * sizeof(int));
    hipMalloc(&d_is_included, n_total * sizeof(int));
    hipMemset(d_is_included, -1, n_total * sizeof(int));
    hipMemset(d_is_included, 1, sizeof(int));//root should always be included
    hipMalloc(&d_new_indecies, n_total * sizeof(int));
    hipMemset(d_new_indecies, 0, n_total * sizeof(int));

    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    int *h_dim_start_1 = new int[d_1];
    int *h_dim_start_2 = new int[d_2];
//    printf("d_1:%d, d_2:%d\n", d_1, d_2);
    gpuErrchk(hipPeekAtLastError());
    hipMemcpy(h_dim_start_1, d_dim_start_1, sizeof(int) * d_1, hipMemcpyDeviceToHost);
    gpuErrchk(hipPeekAtLastError());
    hipMemcpy(h_dim_start_2, d_dim_start_2, sizeof(int) * d_2, hipMemcpyDeviceToHost);
    gpuErrchk(hipPeekAtLastError());
    int step = 4; //todo find better

    int *pivots_1, *pivots_2;
    int n_pivots = (n_total / step + (n_total % step ? 1 : 0));
    hipMalloc(&pivots_1, n_pivots * sizeof(int));
    hipMalloc(&pivots_2, n_pivots * sizeof(int));
    hipMemset(pivots_1, -1, n_pivots * sizeof(int));
    hipMemset(pivots_2, -1, n_pivots * sizeof(int));

    gpuErrchk(hipPeekAtLastError());

    for (int d_i = -1; d_i < d_1; d_i++) {

        int start_1 = d_i == -1 ? 0 : h_dim_start_1[d_i];
        int start_2 = d_i == -1 ? 0 : h_dim_start_2[d_i];
        int end_1 = d_i == -1 ? 1 : (d_i + 1 < d_1 ? h_dim_start_1[d_i + 1] : n_1);
        int end_2 = d_i == -1 ? 1 : (d_i + 1 < d_1 ? h_dim_start_2[d_i + 1] : n_2);
        int start_toal = start_1 + start_2;
        int end_total = end_1 + end_2;
        int length = end_total - start_toal;

        numBlocks = length / (BLOCK_WIDTH * step);
        if (length % (BLOCK_WIDTH * step)) numBlocks++;

        merge_search_for_pivots << < numBlocks, BLOCK_WIDTH >> >
                                                (start_1, start_2, end_1, end_2, pivots_1, pivots_2, n_1, n_2, n_total, step,
                                                        cmp(d_new_indecies, d_map_to_new, d_parents_1, d_parents_2,
                                                            d_cells_1,
                                                            d_cells_2, d_counts_1, d_counts_2, n_1));
        gpuErrchk(hipPeekAtLastError());

        merge_check_path_from_pivots << < numBlocks, BLOCK_WIDTH >> >
                                                     (start_1, start_2, end_1, end_2, d_map_to_old, d_map_to_new, pivots_1, pivots_2, n_1, n_2, n_total, step,
                                                             cmp(d_new_indecies, d_map_to_new, d_parents_1, d_parents_2,
                                                                 d_cells_1,
                                                                 d_cells_2, d_counts_1, d_counts_2, n_1));
        gpuErrchk(hipPeekAtLastError());

        numBlocks = length / BLOCK_WIDTH;
        if (length % BLOCK_WIDTH) numBlocks++;
        compute_is_included_from_path << < numBlocks, BLOCK_WIDTH >> >
                                                      (start_1, start_2, d_is_included, d_map_to_old, d_parents_1, d_parents_2, d_cells_1, d_cells_2, d_counts_1, d_counts_2, n_1, end_total);

        gpuErrchk(hipPeekAtLastError());
        inclusive_scan(d_is_included, d_new_indecies, end_total);
        gpuErrchk(hipPeekAtLastError());
        //hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
    }

    gpuErrchk(hipPeekAtLastError());

    hipDeviceSynchronize();

    int *h_tmp = new int[1];
    hipMemcpy(h_tmp, d_new_indecies + n_total - 1, sizeof(int), hipMemcpyDeviceToHost);
    n_3 = h_tmp[0];


    d_3 = d_1;
    number_of_restricted_dims_3 = number_of_restricted_dims_1;


    //update parent id, cells and count

    hipMalloc(&d_parents_3, n_3 * sizeof(int));
    hipMalloc(&d_cells_3, n_3 * sizeof(int));
    hipMalloc(&d_counts_3, n_3 * sizeof(int));
    hipMemset(d_counts_3, 0, n_3 * sizeof(int));
    hipMalloc(&d_dim_start_3, d_3 * sizeof(int));
    hipMalloc(&d_dims_3, d_3 * sizeof(int));
    hipMalloc(&d_restricted_dims_3, number_of_restricted_dims_3 * sizeof(int));

    gpuErrchk(hipPeekAtLastError());


    numBlocks = n_total / BLOCK_WIDTH;
    if (n_total % BLOCK_WIDTH) numBlocks++;
    merge_move << < numBlocks, BLOCK_WIDTH >> >
                               (d_cells_1, d_cells_2, d_cells_3, d_parents_1, d_parents_2, d_parents_3, d_counts_1, d_counts_2, d_counts_3, d_new_indecies, d_map_to_new, d_map_to_old, n_total, n_1);


    gpuErrchk(hipPeekAtLastError());

    clone << < 1, BLOCK_WIDTH >> > (d_restricted_dims_3, d_restricted_dims_1, number_of_restricted_dims_3);

    if (d_3 > 0) {
        numBlocks = d_3 / BLOCK_WIDTH;
        if (d_3 % BLOCK_WIDTH) numBlocks++;
        merge_update_dim << < numBlocks, BLOCK_WIDTH >> >
                                         (d_dim_start_1, d_dims_1, d_dim_start_2, d_dims_2, d_dim_start_3, d_dims_3, d_new_indecies, d_map_to_new, d_3, n_1);


        gpuErrchk(hipPeekAtLastError());
    }
    hipDeviceSynchronize();
    //get number of points
    //number_of_points_3 = number_of_points_1 + number_of_points_2;
    hipMemcpy(h_tmp, d_counts_3, sizeof(int), hipMemcpyDeviceToHost);
    number_of_points_3 = h_tmp[0];


    gpuErrchk(hipPeekAtLastError());

    //construct new point arrays
    hipMalloc(&d_points_3, number_of_points_3 * sizeof(int));
    hipMemset(d_points_3, 0, number_of_points_3 * sizeof(int));
    hipMalloc(&d_points_placement_3, number_of_points_3 * sizeof(int));
    hipMemset(d_points_placement_3, 0, number_of_points_3 * sizeof(int));


    gpuErrchk(hipPeekAtLastError());

    // for each tree move points to new arrays
    numBlocks = number_of_points_3 / BLOCK_WIDTH;
    if (number_of_points_3 % BLOCK_WIDTH) numBlocks++;
    points_move << < numBlocks, BLOCK_WIDTH >> > (d_points_1, d_points_placement_1, number_of_points_1, n_1,
            d_points_2, d_points_placement_2, number_of_points_2,
            d_points_3, d_points_placement_3, number_of_points_3,
            d_new_indecies, d_map_to_new);


    gpuErrchk(hipPeekAtLastError());

    hipDeviceSynchronize();

    hipFree(d_map_to_old);
    hipFree(d_map_to_new);
    hipFree(d_is_included);
    hipFree(d_new_indecies);

    hipDeviceSynchronize();
}

ScyTreeArray *restrict(ScyTreeArray *scy_tree, int dim_no, int cell_no) {
    //finding sizes and indexes
    int n = scy_tree->number_of_nodes;
    int c = scy_tree->number_of_cells;
    int d = scy_tree->number_of_dims;

    hipMemcpy(scy_tree->h_dims, scy_tree->d_dims, sizeof(int) * d, hipMemcpyDeviceToHost);
    hipMemcpy(scy_tree->h_dim_start, scy_tree->d_dim_start, sizeof(int) * d, hipMemcpyDeviceToHost);
    //hipDeviceSynchronize();

    int dim_i = 0;
    for (int i = 0; i < d; i++) {
        if (scy_tree->h_dims[i] == dim_no) {
            dim_i = i;
        }
    }

    //allocate tmp arrays
    int *d_new_indecies, *d_new_counts, *d_is_included, *d_is_s_connected;
    hipMalloc(&d_new_indecies, n * sizeof(int));
    hipMemset(d_new_indecies, 0, n * sizeof(int));
    hipMalloc(&d_new_counts, n * sizeof(int));
    hipMemset(d_new_counts, 0, n * sizeof(int));
    hipMalloc(&d_is_included, n * sizeof(int));
    hipMemset(d_is_included, 0, n * sizeof(int));

    //hipDeviceSynchronize();

    memset << < 1, 1 >> > (d_is_included, 0, 1);//todo not a good way to do this
    hipMalloc(&d_is_s_connected, sizeof(int));
    hipMemset(d_is_s_connected, 0, sizeof(int));

    gpuErrchk(hipPeekAtLastError());

    //hipDeviceSynchronize();

    // 1. mark the nodes that should be included in the restriction
    //restrict dimension
    int lvl_size = scy_tree->get_lvl_size(dim_i);
    int number_of_blocks = lvl_size / BLOCK_WIDTH;
    if (lvl_size % BLOCK_WIDTH) number_of_blocks++;
    dim3 grid(number_of_blocks); //todo should be parallelized over c aswell
    dim3 block(BLOCK_WIDTH);
    restrict_dim << < grid, block >> > (scy_tree->d_parents, scy_tree->d_cells, scy_tree->d_counts, d_is_included,
            d_new_counts, cell_no, lvl_size, scy_tree->h_dim_start[dim_i], d_is_s_connected);


    gpuErrchk(hipPeekAtLastError());



    //propagrate up from restricted dim
    for (int d_j = dim_i - 1; d_j >= 0; d_j--) { // todo maybe in stream 2
        //todo maybe move for loop inside and stride instead of using blocks
        lvl_size = scy_tree->get_lvl_size(d_j);
        number_of_blocks = lvl_size / BLOCK_WIDTH;
        if (lvl_size % BLOCK_WIDTH) number_of_blocks++;
        dim3 grid_up(number_of_blocks);
        restrict_dim_prop_up << < grid_up, block >> >
                                           (scy_tree->d_parents, scy_tree->d_counts, d_is_included, d_new_counts,
                                                   lvl_size, scy_tree->h_dim_start[d_j]);
    }

    gpuErrchk(hipPeekAtLastError());

    //propagrate down from restricted dim
    if (dim_i + 1 < d) { //todo maybe in stream 1
        //todo maybe move for loop inside and stride instead of using blocks
        lvl_size = scy_tree->get_lvl_size(dim_i + 1);
        number_of_blocks = lvl_size / BLOCK_WIDTH;
        if (lvl_size % BLOCK_WIDTH) number_of_blocks++;
        dim3 grid_down(number_of_blocks);
        restrict_dim_prop_down_first << < grid_down, block >> > (scy_tree->d_parents, scy_tree->d_counts,
                scy_tree->d_cells, d_is_included, d_new_counts, cell_no, lvl_size, scy_tree->h_dim_start[dim_i + 1]);
    }

    gpuErrchk(hipPeekAtLastError());

    for (int d_j = dim_i + 2; d_j < d; d_j++) { //todo maybe in stream 1
        //todo maybe move for loop inside and stride instead of using blocks
        lvl_size = scy_tree->get_lvl_size(d_j);
        number_of_blocks = lvl_size / BLOCK_WIDTH;
        if (lvl_size % BLOCK_WIDTH) number_of_blocks++;
        dim3 grid_down(number_of_blocks);
        restrict_dim_prop_down << < grid_down, block >> >
                                               (scy_tree->d_parents, scy_tree->d_counts, d_is_included, d_new_counts, lvl_size, scy_tree->h_dim_start[d_j]);
    }

    gpuErrchk(hipPeekAtLastError());

    //hipDeviceSynchronize();


    // 2. do a scan to find the new indecies for the nodes in the restricted tree
    inclusive_scan(d_is_included, d_new_indecies, scy_tree->number_of_nodes);
    // 3. construct restricted tree

    gpuErrchk(hipPeekAtLastError());

    //hipDeviceSynchronize();



    int *h_tmp = new int[1];
    h_tmp[0] = 0;
    hipMemcpy(h_tmp, d_new_counts, sizeof(int), hipMemcpyDeviceToHost);
    int new_number_of_points = h_tmp[0];

    gpuErrchk(hipPeekAtLastError());


    hipMemcpy(h_tmp, scy_tree->d_counts, sizeof(int), hipMemcpyDeviceToHost);
    int number_of_points = h_tmp[0];

    hipMemcpy(h_tmp, d_new_indecies + scy_tree->number_of_nodes - 1, sizeof(int), hipMemcpyDeviceToHost);
    int new_number_of_nodes = h_tmp[0];


    gpuErrchk(hipPeekAtLastError());

    ScyTreeArray *restricted_scy_tree = new ScyTreeArray(new_number_of_nodes, scy_tree->number_of_dims - 1,
                                                         scy_tree->number_of_restricted_dims + 1,
                                                         new_number_of_points, scy_tree->number_of_cells);


    restricted_scy_tree->cell_size = scy_tree->cell_size;
    hipMemcpy(h_tmp, d_is_s_connected, sizeof(int), hipMemcpyDeviceToHost);
    restricted_scy_tree->is_s_connected = (bool) h_tmp[0];


    gpuErrchk(hipPeekAtLastError());


    number_of_blocks = scy_tree->number_of_nodes / BLOCK_WIDTH;
    if (scy_tree->number_of_nodes % BLOCK_WIDTH) number_of_blocks++;
    restrict_move << < number_of_blocks, BLOCK_WIDTH >> >
                                         (scy_tree->d_cells, restricted_scy_tree->d_cells,
                                                 scy_tree->d_parents, restricted_scy_tree->d_parents,
//                                                 scy_tree->d_node_order, restricted_scy_tree->d_node_order,
                                                 d_new_counts, restricted_scy_tree->d_counts,
                                                 d_new_indecies, d_is_included, scy_tree->number_of_nodes);


    gpuErrchk(hipPeekAtLastError());

    if (restricted_scy_tree->number_of_dims > 0) {

        number_of_blocks = restricted_scy_tree->number_of_dims / BLOCK_WIDTH;
        if (restricted_scy_tree->number_of_dims % BLOCK_WIDTH) number_of_blocks++;


        restrict_update_dim << < number_of_blocks, BLOCK_WIDTH >> >
                                                   (scy_tree->d_dim_start, scy_tree->d_dims, restricted_scy_tree->d_dim_start,
                                                           restricted_scy_tree->d_dims, d_new_indecies, dim_i,
                                                           restricted_scy_tree->number_of_dims);

        gpuErrchk(hipPeekAtLastError());
    }

    number_of_blocks = restricted_scy_tree->number_of_restricted_dims / BLOCK_WIDTH;
    if (restricted_scy_tree->number_of_restricted_dims % BLOCK_WIDTH) number_of_blocks++;
    restrict_update_restricted_dim << < number_of_blocks, BLOCK_WIDTH >> >
                                                          (dim_no, scy_tree->d_restricted_dims, restricted_scy_tree->d_restricted_dims, scy_tree->number_of_restricted_dims);

    //hipDeviceSynchronize();


    gpuErrchk(hipPeekAtLastError());

    int *d_is_point_included, *d_point_new_indecies;
    hipMalloc(&d_is_point_included, number_of_points * sizeof(int));
    hipMalloc(&d_point_new_indecies, number_of_points * sizeof(int));
    hipMemset(d_is_point_included, 0, number_of_points * sizeof(int));


    //gpuErrchk(hipPeekAtLastError());

    bool restricted_dim_is_leaf = (dim_i == scy_tree->number_of_dims - 1);

    number_of_blocks = number_of_points / BLOCK_WIDTH;
    if (number_of_points % BLOCK_WIDTH) number_of_blocks++;
    compute_is_points_included << < number_of_blocks, BLOCK_WIDTH >> > (
            scy_tree->d_points, scy_tree->d_points_placement, scy_tree->d_parents, scy_tree->d_cells, d_is_included, d_is_point_included,
                    scy_tree->number_of_nodes, number_of_points, new_number_of_points, restricted_dim_is_leaf, cell_no);


    gpuErrchk(hipPeekAtLastError());

    inclusive_scan(d_is_point_included, d_point_new_indecies, number_of_points);
//    dim3 dimBlock(BLOCKSIZE,BLOCKSIZE);
//    dim3 dimGrid(1,1);
//    PrefixSum<<<dimGrid,dimBlock>>>(d_point_new_indecies,d_is_point_included,number_of_points,BLOCKSIZE);

    CUDA_SAFE_CALL(hipDeviceSynchronize());
//    printf("d_is_point_included:\n");
//    print_array_gpu<<<1,1>>>(d_is_point_included, number_of_points);


    gpuErrchk(hipPeekAtLastError());

    move_points << < number_of_blocks, BLOCK_WIDTH >> > (scy_tree->d_parents, scy_tree->d_points,
            scy_tree->d_points_placement, restricted_scy_tree->d_points, restricted_scy_tree->d_points_placement,
            d_point_new_indecies, d_new_indecies, d_is_point_included, number_of_points, restricted_dim_is_leaf);

    hipDeviceSynchronize();


    gpuErrchk(hipPeekAtLastError());

    //todo hipFree() temps
//    hipFree(d_new_indecies);
//    gpuErrchk(hipPeekAtLastError());
//    hipFree(d_new_counts);
//    gpuErrchk(hipPeekAtLastError());
//    hipFree(d_is_included);
//    gpuErrchk(hipPeekAtLastError());

    hipDeviceSynchronize();

    gpuErrchk(hipPeekAtLastError());
    return restricted_scy_tree;
}

ScyTreeArray *restrict3(ScyTreeArray *scy_tree, int dim_no, int cell_no) {
    int number_of_blocks;
    dim3 block(512);
    //gpuErrchk(hipPeekAtLastError());

    //finding sizes and indexes
    //int n = scy_tree->number_of_nodes;
    int c = scy_tree->number_of_cells;
    int d = scy_tree->number_of_dims;

    int *d_dim_i;
    hipMalloc(&d_dim_i, sizeof(int));//todo use pre-allocated memory
    find_dim_i << < 1, 1 >> > (d_dim_i, scy_tree->d_dims, dim_no, scy_tree->number_of_dims);

    //allocate tmp arrays
    int *d_new_indecies, *d_new_counts, *d_is_included, *d_is_s_connected;
    hipMalloc(&d_new_indecies, scy_tree->number_of_nodes * sizeof(int));
    hipMemset(d_new_indecies, 0, scy_tree->number_of_nodes * sizeof(int));
    hipMalloc(&d_new_counts, scy_tree->number_of_nodes * sizeof(int));
    hipMemset(d_new_counts, 0, scy_tree->number_of_nodes * sizeof(int));
    hipMalloc(&d_is_included, scy_tree->number_of_nodes * sizeof(int));
    hipMemset(d_is_included, 0, scy_tree->number_of_nodes * sizeof(int));

    //hipDeviceSynchronize();

    memset << < 1, 1 >> > (d_is_included, 0, 1);//todo not a good way to do this
    hipMalloc(&d_is_s_connected, sizeof(int));
    hipMemset(d_is_s_connected, 0, sizeof(int));

    //gpuErrchk(hipPeekAtLastError());

    //hipDeviceSynchronize();

    // 1. mark the nodes that should be included in the restriction
    //restrict dimension
    restrict_dim_3 << < 1, block >> > (scy_tree->d_parents, scy_tree->d_cells, scy_tree->d_counts, d_is_included,
            d_new_counts, cell_no, scy_tree->d_dim_start, d_dim_i, d_is_s_connected, scy_tree->number_of_dims, scy_tree->number_of_nodes); //todo move h_dim_start[dim_i] to kernel


    //gpuErrchk(hipPeekAtLastError());



    //propagrate up from restricted dim

    restrict_dim_prop_up_3 << < 1, block >> >
                                   (scy_tree->d_parents, scy_tree->d_counts, d_is_included, d_new_counts,
                                           d_dim_i, scy_tree->d_dim_start, scy_tree->number_of_dims, scy_tree->number_of_nodes);


    //gpuErrchk(hipPeekAtLastError());

    //propagrate down from restricted dim
    restrict_dim_prop_down_first_3 << < 1, block >> >
                                           (scy_tree->d_parents, scy_tree->d_counts, scy_tree->d_cells, d_is_included, d_new_counts,
                                                   scy_tree->d_dim_start, d_dim_i,
                                                   cell_no, scy_tree->number_of_dims, scy_tree->number_of_nodes);

    //gpuErrchk(hipPeekAtLastError());

    restrict_dim_prop_down_3 << < 1, block >> >
                                     (scy_tree->d_parents, scy_tree->d_counts, d_is_included, d_new_counts,
                                             scy_tree->d_dim_start, d_dim_i,
                                             scy_tree->number_of_dims, scy_tree->number_of_nodes);

    //gpuErrchk(hipPeekAtLastError());

    //hipDeviceSynchronize();


    // 2. do a scan to find the new indecies for the nodes in the restricted tree
    inclusive_scan(d_is_included, d_new_indecies, scy_tree->number_of_nodes);
    // 3. construct restricted tree

    //gpuErrchk(hipPeekAtLastError());

    //hipDeviceSynchronize();



    int *h_tmp = new int[1];
    h_tmp[0] = 0;
    hipMemcpy(h_tmp, d_new_counts, sizeof(int), hipMemcpyDeviceToHost);
    int new_number_of_points = h_tmp[0];

    //gpuErrchk(hipPeekAtLastError());

    hipMemcpy(h_tmp, scy_tree->d_counts, sizeof(int), hipMemcpyDeviceToHost);
    int number_of_points = h_tmp[0];

    hipMemcpy(h_tmp, d_new_indecies + scy_tree->number_of_nodes - 1, sizeof(int), hipMemcpyDeviceToHost);
    int new_number_of_nodes = h_tmp[0];


    //gpuErrchk(hipPeekAtLastError());
    //ScyTreeArray(int number_of_nodes, int number_of_dims, int number_of_restricted_dims, int number_of_points, int number_of_cells)
    ScyTreeArray *restricted_scy_tree = new ScyTreeArray(new_number_of_nodes, scy_tree->number_of_dims - 1,
                                                         scy_tree->number_of_restricted_dims + 1,
                                                         new_number_of_points, scy_tree->number_of_cells);

    restricted_scy_tree->cell_size = scy_tree->cell_size;//todo maybe not used
    hipMemcpy(h_tmp, d_is_s_connected, sizeof(int), hipMemcpyDeviceToHost);
    restricted_scy_tree->is_s_connected = (bool) h_tmp[0];


    //gpuErrchk(hipPeekAtLastError());


    number_of_blocks = scy_tree->number_of_nodes / BLOCK_WIDTH;
    if (scy_tree->number_of_nodes % BLOCK_WIDTH) number_of_blocks++;
    restrict_move << < number_of_blocks, BLOCK_WIDTH >> >
                                         (scy_tree->d_cells, restricted_scy_tree->d_cells,
                                                 scy_tree->d_parents, restricted_scy_tree->d_parents,
//                                                 scy_tree->d_node_order, restricted_scy_tree->d_node_order,
                                                 d_new_counts, restricted_scy_tree->d_counts,
                                                 d_new_indecies, d_is_included, scy_tree->number_of_nodes);


    //gpuErrchk(hipPeekAtLastError());

    if (restricted_scy_tree->number_of_dims > 0) {

        number_of_blocks = restricted_scy_tree->number_of_dims / BLOCK_WIDTH;
        if (restricted_scy_tree->number_of_dims % BLOCK_WIDTH) number_of_blocks++;


        restrict_update_dim_3 << < number_of_blocks, BLOCK_WIDTH >> >
                                                     (scy_tree->d_dim_start, scy_tree->d_dims, restricted_scy_tree->d_dim_start,
                                                             restricted_scy_tree->d_dims, d_new_indecies,
                                                             d_dim_i,
                                                             restricted_scy_tree->number_of_dims);

        //gpuErrchk(hipPeekAtLastError());
    }

    number_of_blocks = restricted_scy_tree->number_of_restricted_dims / BLOCK_WIDTH;
    if (restricted_scy_tree->number_of_restricted_dims % BLOCK_WIDTH) number_of_blocks++;
    restrict_update_restricted_dim << < number_of_blocks, BLOCK_WIDTH >> >
                                                          (dim_no, scy_tree->d_restricted_dims, restricted_scy_tree->d_restricted_dims, scy_tree->number_of_restricted_dims);

    //hipDeviceSynchronize();


    //gpuErrchk(hipPeekAtLastError());

    int *d_is_point_included, *d_point_new_indecies;
    hipMalloc(&d_is_point_included, number_of_points * sizeof(int));
    hipMalloc(&d_point_new_indecies, number_of_points * sizeof(int));
    hipMemset(d_is_point_included, 0, number_of_points * sizeof(int));


    //gpuErrchk(hipPeekAtLastError());



    number_of_blocks = number_of_points / BLOCK_WIDTH;
    if (number_of_points % BLOCK_WIDTH) number_of_blocks++;
    compute_is_points_included_3 << < number_of_blocks, BLOCK_WIDTH >> >
                                                        (scy_tree->d_points_placement, scy_tree->d_cells, d_is_included,
                                                                d_is_point_included, d_dim_i,
                                                                scy_tree->number_of_dims, scy_tree->number_of_points, cell_no);


    //gpuErrchk(hipPeekAtLastError());

    inclusive_scan(d_is_point_included, d_point_new_indecies, number_of_points);


    //gpuErrchk(hipPeekAtLastError());

    move_points_3 << < number_of_blocks, BLOCK_WIDTH >> > (scy_tree->d_parents, scy_tree->d_points,
            scy_tree->d_points_placement, restricted_scy_tree->d_points, restricted_scy_tree->d_points_placement,
            d_point_new_indecies, d_new_indecies, d_is_point_included, d_dim_i,
            number_of_points, scy_tree->number_of_dims);

    //hipDeviceSynchronize();


    //gpuErrchk(hipPeekAtLastError());

    //todo hipFree() temps
    hipFree(d_new_indecies);
    hipFree(d_new_counts);
    hipFree(d_is_included);

    hipDeviceSynchronize();

    return restricted_scy_tree;
}

int ScyTreeArray::get_lvl_size(int d_i) {
    return (d_i == this->number_of_dims - 1 ? this->number_of_nodes : this->h_dim_start[d_i + 1]) -
           this->h_dim_start[d_i];
}

ScyTreeArray *ScyTreeArray::restrict_gpu(int dim_no, int cell_no) {
    ScyTreeArray *restricted_scy_tree = restrict(this, dim_no, cell_no);

    return restricted_scy_tree;
}

ScyTreeArray *ScyTreeArray::mergeWithNeighbors_gpu(ScyTreeArray *parent_scy_tree, int dim_no, int cell_no) {
    if (!this->is_s_connected) {
        return this;
    }

    ScyTreeArray *merged_scy_tree = this;
    ScyTreeArray *restricted_scy_tree = this;
    while (restricted_scy_tree->is_s_connected && cell_no < this->number_of_cells - 1) {
        cell_no++;
//        printf("%d\n",cell_no);
        gpuErrchk(hipPeekAtLastError());
        restricted_scy_tree = parent_scy_tree->restrict_gpu(dim_no, cell_no);
        gpuErrchk(hipPeekAtLastError());
        if (restricted_scy_tree->number_of_points > 0) {
            ScyTreeArray *merged_scy_tree_old = merged_scy_tree;
            gpuErrchk(hipPeekAtLastError());
            merged_scy_tree = merged_scy_tree->merge(restricted_scy_tree);
            gpuErrchk(hipPeekAtLastError());
            // delete merged_scy_tree_old;
        }
    }

    merged_scy_tree->is_s_connected = false;
    return merged_scy_tree;
}

ScyTreeArray *ScyTreeArray::merge(ScyTreeArray *sibling_scy_tree) {
    int *d_parents_3, *d_cells_3, *d_counts_3, *d_dim_start_3, *d_dims_3, *d_restricted_dims_3, *d_points_3, *d_points_placement_3;
    int n_3, d_3, number_of_points_3, number_of_restricted_dims_3;

    gpuErrchk(hipPeekAtLastError());
    merge_using_gpu(this->d_parents, this->d_cells, this->d_counts,
                    this->d_dim_start, this->d_dims, this->d_restricted_dims,
                    this->d_points, this->d_points_placement,
                    this->number_of_dims, this->number_of_nodes, this->number_of_points,
                    this->number_of_restricted_dims,
                    sibling_scy_tree->d_parents, sibling_scy_tree->d_cells, sibling_scy_tree->d_counts,
                    sibling_scy_tree->d_dim_start, sibling_scy_tree->d_dims, sibling_scy_tree->d_restricted_dims,
                    sibling_scy_tree->d_points, sibling_scy_tree->d_points_placement,
                    sibling_scy_tree->number_of_dims, sibling_scy_tree->number_of_nodes,
                    sibling_scy_tree->number_of_restricted_dims,
                    sibling_scy_tree->number_of_points,
                    d_parents_3, d_cells_3, d_counts_3,
                    d_dim_start_3, d_dims_3, d_restricted_dims_3,
                    d_points_3, d_points_placement_3,
                    d_3, n_3, number_of_points_3, number_of_restricted_dims_3);

//    printf("after merge_using_gpu\n");

    gpuErrchk(hipPeekAtLastError());
    ScyTreeArray *merged_scy_tree = new ScyTreeArray(n_3, this->number_of_dims, this->number_of_restricted_dims,
                                                     number_of_points_3, this->number_of_cells,
                                                     d_cells_3, d_parents_3, d_counts_3,
                                                     d_dim_start_3, d_dims_3, d_restricted_dims_3,
                                                     d_points_3, d_points_placement_3);

    gpuErrchk(hipPeekAtLastError());

    return merged_scy_tree;
}

int ScyTreeArray::get_dims_idx() {
    int sum = 0;

    hipMemcpy(this->h_restricted_dims, this->d_restricted_dims, sizeof(int) * number_of_restricted_dims,
               hipMemcpyDeviceToHost);
    for (int i = 0; i < this->number_of_restricted_dims; i++) {
        int re_dim = this->h_restricted_dims[i];
        sum += 1 << re_dim;
    }
    return sum;
}

ScyTreeArray::ScyTreeArray(int number_of_nodes, int number_of_dims, int number_of_restricted_dims, int number_of_points,
                           int number_of_cells) {
    this->number_of_nodes = number_of_nodes;
    this->number_of_dims = number_of_dims;
    this->number_of_restricted_dims = number_of_restricted_dims;
    this->number_of_points = number_of_points;
    this->number_of_cells = number_of_cells;

//    printf("\nScyTreeArray - small - number_of_nodes:%d, number_of_dims:%d, number_of_restricted_dims:%d, number_of_points:%d, number_of_cells:%d\n", number_of_nodes, number_of_dims, number_of_restricted_dims, number_of_points,
//           number_of_cells);

    this->h_parents = new int[number_of_nodes];
    zero(this->h_parents, number_of_nodes);

    this->h_cells = new int[number_of_nodes];
    zero(this->h_cells, number_of_nodes);

    this->h_counts = new int[number_of_nodes];
    zero(this->h_counts, number_of_nodes);

    this->h_dim_start = new int[number_of_dims];
    zero(this->h_dim_start, number_of_dims);

    this->h_dims = new int[number_of_dims];
    zero(this->h_dims, number_of_dims);

    this->h_points = new int[number_of_points];
    zero(this->h_points, number_of_points);

    this->h_points_placement = new int[number_of_points];
    zero(this->h_points_placement, number_of_points);

    this->h_restricted_dims = new int[number_of_restricted_dims];
    zero(this->h_restricted_dims, number_of_restricted_dims);


    hipMalloc(&this->d_parents, number_of_nodes * sizeof(int));
    hipMemset(this->d_parents, 0, number_of_nodes * sizeof(int));

    hipMalloc(&this->d_cells, number_of_nodes * sizeof(int));
    hipMemset(this->d_cells, 0, number_of_nodes * sizeof(int));

    hipMalloc(&this->d_counts, number_of_nodes * sizeof(int));
    hipMemset(this->d_counts, 0, number_of_nodes * sizeof(int));

    hipMalloc(&this->d_dim_start, number_of_dims * sizeof(int));
    hipMemset(this->d_dim_start, 0, number_of_dims * sizeof(int));

    hipMalloc(&this->d_dims, number_of_dims * sizeof(int));
    hipMemset(this->d_dims, 0, number_of_dims * sizeof(int));

    hipMalloc(&this->d_restricted_dims, number_of_restricted_dims * sizeof(int));
    hipMemset(this->d_restricted_dims, 0, number_of_restricted_dims * sizeof(int));

    hipMalloc(&this->d_points, number_of_points * sizeof(int));
    hipMemset(this->d_points, 0, number_of_points * sizeof(int));

    hipMalloc(&this->d_points_placement, number_of_points * sizeof(int));
    hipMemset(this->d_points_placement, 0, number_of_points * sizeof(int));
}

ScyTreeArray::ScyTreeArray(int number_of_nodes, int number_of_dims, int number_of_restricted_dims, int number_of_points,
                           int number_of_cells, int *d_cells, int *d_parents, int *d_counts, int *d_dim_start,
                           int *d_dims, int *d_restricted_dims, int *d_points, int *d_points_placement) {

//    printf("\nScyTreeArray - large - number_of_nodes:%d, number_of_dims:%d, number_of_restricted_dims:%d, number_of_points:%d, number_of_cells:%d\n", number_of_nodes, number_of_dims, number_of_restricted_dims, number_of_points,
//           number_of_cells);

    this->number_of_nodes = number_of_nodes;
    this->number_of_dims = number_of_dims;
    this->number_of_restricted_dims = number_of_restricted_dims;
    this->number_of_points = number_of_points;
    this->number_of_cells = number_of_cells;

    this->h_parents = new int[number_of_nodes];
    zero(this->h_parents, number_of_nodes);

    this->h_cells = new int[number_of_nodes];
    zero(this->h_cells, number_of_nodes);

    this->h_counts = new int[number_of_nodes];
    zero(this->h_counts, number_of_nodes);

    this->h_dim_start = new int[number_of_dims];
    zero(this->h_dim_start, number_of_dims);

    this->h_dims = new int[number_of_dims];
    zero(this->h_dims, number_of_dims);

    this->h_points = new int[number_of_points];
    zero(this->h_points, number_of_points);

    this->h_points_placement = new int[number_of_points];
    zero(this->h_points_placement, number_of_points);

    this->h_restricted_dims = new int[number_of_restricted_dims];
    zero(this->h_restricted_dims, number_of_restricted_dims);


    this->d_parents = d_parents;

    this->d_cells = d_cells;

    this->d_counts = d_counts;

    this->d_dim_start = d_dim_start;

    this->d_dims = d_dims;

    this->d_restricted_dims = d_restricted_dims;

    this->d_points = d_points;

    this->d_points_placement = d_points_placement;
}

void ScyTreeArray::copy_to_host() {
    hipMemcpy(h_parents, d_parents, sizeof(int) * number_of_nodes, hipMemcpyDeviceToHost);
    hipMemcpy(h_cells, d_cells, sizeof(int) * number_of_nodes, hipMemcpyDeviceToHost);
    hipMemcpy(h_counts, d_counts, sizeof(int) * number_of_nodes, hipMemcpyDeviceToHost);
    hipMemcpy(h_dim_start, d_dim_start, sizeof(int) * number_of_dims, hipMemcpyDeviceToHost);
    hipMemcpy(h_dims, d_dims, sizeof(int) * number_of_dims, hipMemcpyDeviceToHost);
    hipMemcpy(h_points, d_points, sizeof(int) * number_of_points, hipMemcpyDeviceToHost);
    hipMemcpy(h_points_placement, d_points_placement, sizeof(int) * number_of_points, hipMemcpyDeviceToHost);
    hipMemcpy(h_restricted_dims, d_restricted_dims, sizeof(int) * number_of_restricted_dims, hipMemcpyDeviceToHost);
}

void ScyTreeArray::print() {
    print_scy_tree(this->h_parents, this->h_cells, this->h_counts, this->h_dim_start, this->h_dims,
                   this->number_of_dims, this->number_of_nodes);
    printf("number_of_nodes: %d, number_of_points: %d, number_of_dims: %d, number_of_restricted_dims: %d, number_of_cells: %d\n",
           this->number_of_nodes, this->number_of_points, this->number_of_dims, this->number_of_restricted_dims,
           this->number_of_cells);
    printf("\n");
}
