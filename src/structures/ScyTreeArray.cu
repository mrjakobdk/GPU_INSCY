#include "hip/hip_runtime.h"
#include "ScyTreeArray.h"
#include "../utils/RestrictUtils.h"
#include "../utils/MergeUtil.h"
#include "../utils/util.h"
#include "../utils/TmpMalloc.cuh"
#include "../algorithms/clustering/ClusteringCpu.h"
//#include "../algorithms/clustering/ClusteringGpu.cuh"

#define BLOCKSIZE 16
#define BLOCK_SIZE 512
#define PI 3.14
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/*Check for safe return of all calls to the device */
void CUDA_SAFE_CALL(hipError_t call) {
    hipError_t ret = call;
    //printf("RETURN FROM THE CUDA CALL:%d\t:",ret);
    switch (ret) {
        case hipSuccess:
            //              printf("Success\n");
            break;
            /*      case hipErrorInvalidValue:
                                    {
                                    printf("ERROR: InvalidValue:%i.\n",__LINE__);
                                    exit(-1);
                                    break;
                                    }
                    case hipErrorInvalidDevicePointer:
                                    {
                                    printf("ERROR:Invalid Device pointeri:%i.\n",__LINE__);
                                    exit(-1);
                                    break;
                                    }
                    case hipErrorInvalidMemcpyDirection:
                                    {
                                    printf("ERROR:Invalid memcpy direction:%i.\n",__LINE__);
                                    exit(-1);
                                    break;
                                    }                       */
        default: {
            printf(" ERROR at line :%i.%d' ' %s\n", __LINE__, ret, hipGetErrorString(ret));
            exit(-1);
            break;
        }
    }
}

__global__ void PrefixSum(int *dInArray, int *dOutArray, int arrayLen, int threadDim) {
    //http://www.tezu.ernet.in/dcompsc/facility/HPCC/hypack/gpgpu-nvidia-cuda-prog-hypack-2013/gpu-comp-nvidia-cuda-num-comp-codes/cuda-prefix-sum.cu
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int tindex = (threadDim * tidx) + tidy;
    int maxNumThread = threadDim * threadDim;
    int pass = 0;
    int count;
    int curEleInd;
    int tempResult = 0;

    while ((curEleInd = (tindex + maxNumThread * pass)) < arrayLen) {
        tempResult = 0;
        for (count = 0; count < curEleInd; count++)
            tempResult += dInArray[count];
        dOutArray[curEleInd] = tempResult;
        pass++;
    }
    __syncthreads();
}//end of Prefix sum function


#define BLOCK_WIDTH 64

__global__
void
merge_search_for_pivots_new(int block_idx_x, int block_dim_x, int thread_idx_x, int start_1, int start_2, int end_1,
                            int end_2, int *pivots_1,
                            int *pivots_2,
                            int number_of_nodes_1,
                            int number_of_nodes_2,
                            int number_of_nodes_total,
                            int step, cmp c) {
    //this is very close to the code from:
    //https://web.cs.ucdavis.edu/~amenta/f15/GPUmp.pdf: GPU Merge Path - A GPU Merging Algorithm
    //also see Merge path - parallel merging made simple. In Parallel and Distributed Processing Symposium, International,may2012.
    int j = block_idx_x * block_dim_x + thread_idx_x;
    int i = j * step;
    int length_1 = end_1 - start_1;
    int length_2 = end_2 - start_2;

    if (i >= length_1 + length_2)
        return;

    //binary search
    int r_1 = min(end_1, start_1 + i);
    int r_2 = start_2 + max(0, i - (length_1));
    int l_1 = start_1 + max(0, i - (length_2));
    int l_2 = min(end_2, start_2 + i);
    int m_1 = 0;
    int m_2 = 0;

    if (i == 132) {
        printf("i:%d, j:%d, start_1: %d, start_2: %d, end_1: %d, end_2: %d\n", i, j, start_1, start_2, end_1, end_2);

        int offset = (r_1 - l_1) / 2;
        m_1 = r_1 - offset;
        m_2 = r_2 + offset;

        printf("m_1: %d, m_2: %d, r_1: %d, l_1: %d\n", m_1, m_2, r_1, l_1);
        bool not_above = (m_2 == 0 || m_1 == end_1 || !c(m_1, m_2 - 1));
        bool left_off = (m_1 == 0 || m_2 == end_2 || c(m_1 - 1, m_2));
        if (not_above) {
            printf("not_above %d\n", i);
            if (left_off) {
                printf("left_off %d\n", i);
            } else {
                printf("not_left_off %d\n", i);
            }
        } else {
            printf("above %d\n", i);
        }
        for (int x = m_1 - 2; x < m_1 + 3; x++) {
            for (int y = m_2 - 2; y < m_2 + 3; y++) {
                if (x >= end_1 || y >= end_2 || x < start_1 || y < start_2) {
                    printf("- ");
                } else if (c(x, y)) {
                    printf("1 ");
                } else {
                    printf("0 ");
                }
            }
            printf("\n");
        }
    }

    while (true) {//L <= R:
        int offset = (r_1 - l_1) / 2;
        m_1 = r_1 - offset;
        m_2 = r_2 + offset;

        bool not_above = (m_2 == 0 || m_1 == end_1 || !c(m_1, m_2 - 1));
        bool left_off = (m_1 == 0 || m_2 == end_2 || c(m_1 - 1, m_2));


        if (not_above) {
            if (left_off) {
                break;
            } else {
                r_1 = m_1 - 1;
                r_2 = m_2 + 1;
            }
        } else {
            l_1 = m_1 + 1;
            l_2 = m_2 - 1;
        }


    }

    pivots_1[j] = m_1;
    pivots_2[j] = m_2;
}

__global__
void print_c(cmp c, int i, int j) {
    if (c(i, j)) {
        printf("1 ");
    } else {
        printf("0 ");
    }
}

void merge_using_gpu(int *d_parents_1, int *d_cells_1, int *d_counts_1,
                     int *d_dim_start_1, int *d_dims_1, int *d_restricted_dims_1,
                     int *d_points_1, int *d_points_placement_1,
                     int d_1, int n_1, int number_of_points_1, int number_of_restricted_dims_1,
                     int *d_parents_2, int *d_cells_2, int *d_counts_2,
                     int *d_dim_start_2, int *d_dims_2, int *d_restricted_dims_2,
                     int *d_points_2, int *d_points_placement_2,
                     int d_2, int n_2, int number_of_points_2, int number_of_restricted_dims_2,
                     int *&d_parents_3, int *&d_cells_3, int *&d_counts_3,
                     int *&d_dim_start_3, int *&d_dims_3, int *&d_restricted_dims_3,
                     int *&d_points_3, int *&d_points_placement_3,
                     int &d_3, int &n_3, int &number_of_points_3, int &number_of_restricted_dims_3) {

//    printf("d_1: %d, n_1:%d, points_1:%d\n", d_1, n_1, number_of_points_1);
//    printf("d_2: %d, n_2:%d, points_2:%d\n", d_2, n_2, number_of_points_2);

    gpuErrchk(hipPeekAtLastError());

    //compute sort keys for both using cell id cell_no and concat
    //sort - save permutation
    int n_total = n_1 + n_2;

    int numBlocks;

    int *d_map_to_old;
    int *d_map_to_new;
    int *d_is_included;
    int *d_new_indecies;
    hipMalloc(&d_map_to_new, n_total * sizeof(int));
    hipMemset(d_map_to_new, -99, n_total * sizeof(int));
//    hipDeviceSynchronize();
    memset << < 1, 1 >> > (d_map_to_new, 0, 1);//q
    memset << < 1, 1 >> > (d_map_to_new, 0 + n_1, 0);//q

    hipMalloc(&d_map_to_old, n_total * sizeof(int));
    hipMemset(d_map_to_old, -88, n_total * sizeof(int));
//    hipDeviceSynchronize();
    memset << < 1, 1 >> > (d_map_to_old, 1, 0);//q
    memset << < 1, 1 >> > (d_map_to_old, 0, 0 + n_1);//q

    hipMalloc(&d_is_included, n_total * sizeof(int));
    hipMemset(d_is_included, -77, n_total * sizeof(int));
    memset << < 1, 1 >> > (d_is_included, 0, 1);//root should always be included
    memset << < 1, 1 >> > (d_is_included, 1, 0);//q

    hipMalloc(&d_new_indecies, n_total * sizeof(int));
    hipMemset(d_new_indecies, 0, n_total * sizeof(int));
//    hipDeviceSynchronize();
    memset << < 1, 1 >> > (d_new_indecies, 0, 1);//q
    memset << < 1, 1 >> > (d_new_indecies, 1, 1);//q

//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

//    printf("d_new_indecies\n");
//    print_array_gpu<<< 1, 1 >>>(d_new_indecies, n_total);
//    hipDeviceSynchronize();
//    gpuErrchk(hipPeekAtLastError());
//    printf("d_map_to_new\n");
//    print_array_gpu<<< 1, 1 >>>(d_map_to_new, n_total);
//    hipDeviceSynchronize();
//    gpuErrchk(hipPeekAtLastError());
//    printf("d_map_to_old\n");
//    print_array_gpu<<< 1, 1 >>>(d_map_to_old, n_total);
//    hipDeviceSynchronize();
//    gpuErrchk(hipPeekAtLastError());
//    printf("d_is_included\n");
//    print_array_gpu<<< 1, 1 >>>(d_is_included, n_total);
//    hipDeviceSynchronize();
//    gpuErrchk(hipPeekAtLastError());

    int *h_dim_start_1 = new int[d_1];
    int *h_dim_start_2 = new int[d_2];
//    printf("d_1:%d, d_2:%d\n", d_1, d_2);
    gpuErrchk(hipPeekAtLastError());
    hipMemcpy(h_dim_start_1, d_dim_start_1, sizeof(int) * d_1, hipMemcpyDeviceToHost);
    gpuErrchk(hipPeekAtLastError());
    hipMemcpy(h_dim_start_2, d_dim_start_2, sizeof(int) * d_2, hipMemcpyDeviceToHost);
    gpuErrchk(hipPeekAtLastError());
    int step = 4; //todo find better

    int *pivots_1, *pivots_2;
    int n_pivots = (n_total / step + (n_total % step ? 1 : 0));
    hipMalloc(&pivots_1, n_pivots * sizeof(int));
    hipMalloc(&pivots_2, n_pivots * sizeof(int));

    gpuErrchk(hipPeekAtLastError());

    for (int d_i = -1; d_i < d_1; d_i++) {//todo root always has the same result, so it can be avoided
        hipMemset(pivots_1, -1, n_pivots * sizeof(int));
        hipMemset(pivots_2, -1, n_pivots * sizeof(int));
//        printf("d_i:%d\n", d_i);
        int start_1 = d_i == -1 ? 0 : h_dim_start_1[d_i];
        int start_2 = d_i == -1 ? 0 : h_dim_start_2[d_i];
        int end_1 = d_i == -1 ? 1 : (d_i + 1 < d_1 ? h_dim_start_1[d_i + 1] : n_1);
        int end_2 = d_i == -1 ? 1 : (d_i + 1 < d_1 ? h_dim_start_2[d_i + 1] : n_2);
        int start_toal = start_1 + start_2;
        int end_total = end_1 + end_2;
        int length = end_total - start_toal;

        numBlocks = length / (BLOCK_WIDTH * step);
        if (length % (BLOCK_WIDTH * step)) numBlocks++;

//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

//        if (d_i == 6 && n_total == 2094) {
//            printf("start_1: %d, start_2: %d\n", start_1, start_2);
//            printf("end_1: %d, end_2: %d\n", end_1, end_2);
//            printf("d_map_to_new:\n");
//            print_array_gpu<<<1, 1>>>(d_map_to_new, start_1);
//            hipDeviceSynchronize();
//            print_array_gpu<<<1, 1>>>(d_map_to_new + n_1, start_2);
//            hipDeviceSynchronize();
//        }

        merge_search_for_pivots << < numBlocks, BLOCK_WIDTH >> >
                                                (start_1, start_2, end_1, end_2, pivots_1, pivots_2, n_1, n_2, n_total, step,
                                                        cmp(d_new_indecies, d_map_to_new,
                                                            d_parents_1, d_parents_2,
                                                            d_cells_1, d_cells_2,
                                                            d_counts_1, d_counts_2,
                                                            n_1, n_2));
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());


//        if (d_i == 5 && n_total == 2094) {
//
//
//            cmp c(d_new_indecies, d_map_to_new, d_parents_1, d_parents_2,
//                  d_cells_1, d_cells_2, d_counts_1, d_counts_2, n_1, n_2);
//
////            int i = 5;
////
////            int s_1 = i == -1 ? 0 : h_dim_start_1[i];
////            int s_2 = i == -1 ? 0 : h_dim_start_2[i];
////            int e_1 = i == -1 ? 1 : (i + 1 < d_1 ? h_dim_start_1[i + 1] : n_1);
////            int e_2 = i == -1 ? 1 : (i + 1 < d_1 ? h_dim_start_2[i + 1] : n_2);
////            for (int i = s_1; i < e_1; i++) {
////                for (int j = s_2; j < e_2; j++) {
////                    print_c <<<1, 1 >>>(c, i, j);
////                    hipDeviceSynchronize();
////                    gpuErrchk(hipPeekAtLastError());
////                }
////                printf("\n");
////            }
////            printf("\n");
////            printf("\n");
//            printf("n_total: %d\n", n_total);
//
//            printf("pivots_1:\n");
//            print_array_gpu<<<1, 1>>>(pivots_1, n_pivots);
//            hipDeviceSynchronize();
//
//            printf("pivots_2:\n");
//            print_array_gpu<<<1, 1>>>(pivots_2, n_pivots);
//            hipDeviceSynchronize();
//        }

        merge_check_path_from_pivots << < 1, 1 >> >
                                             (start_1, start_2, end_1, end_2, d_map_to_old, d_map_to_new, pivots_1, pivots_2, n_1, n_2, n_total, step,
                                                     cmp(d_new_indecies, d_map_to_new, d_parents_1,
                                                         d_parents_2,
                                                         d_cells_1, d_cells_2, d_counts_1, d_counts_2,
                                                         n_1, n_2));
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());


        numBlocks = length / BLOCK_WIDTH;
        if (length % BLOCK_WIDTH) numBlocks++;
        compute_is_included_from_path << < numBlocks, BLOCK_WIDTH >> >
                                                      (start_1, start_2, d_is_included, d_map_to_old, d_parents_1, d_parents_2, d_cells_1, d_cells_2, d_counts_1, d_counts_2, n_1, end_total);

//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        hipMemset(d_new_indecies, 0, n_total * sizeof(int));

//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        inclusive_scan(d_is_included, d_new_indecies, n_total);

//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
    }

//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

//    hipDeviceSynchronize();

    int *h_tmp = new int[1];
    hipMemcpy(h_tmp, d_new_indecies + n_total - 1, sizeof(int), hipMemcpyDeviceToHost);
    n_3 = h_tmp[0];


    d_3 = d_1;
    number_of_restricted_dims_3 = number_of_restricted_dims_1;


    //update parent id, cells and count

    gpuErrchk(hipPeekAtLastError());
    hipMalloc(&d_parents_3, n_3 * sizeof(int));
    gpuErrchk(hipPeekAtLastError());
    hipMalloc(&d_cells_3, n_3 * sizeof(int));
    gpuErrchk(hipPeekAtLastError());
    hipMalloc(&d_counts_3, n_3 * sizeof(int));
    gpuErrchk(hipPeekAtLastError());
    hipMemset(d_counts_3, 0, n_3 * sizeof(int));
    gpuErrchk(hipPeekAtLastError());
    hipMalloc(&d_dim_start_3, d_3 * sizeof(int));
    gpuErrchk(hipPeekAtLastError());
    hipMalloc(&d_dims_3, d_3 * sizeof(int));
    gpuErrchk(hipPeekAtLastError());
    hipMalloc(&d_restricted_dims_3, number_of_restricted_dims_3 * sizeof(int));

//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());


    numBlocks = n_total / BLOCK_WIDTH;
    if (n_total % BLOCK_WIDTH) numBlocks++;
    merge_move << < numBlocks, BLOCK_WIDTH >> >
                               (d_cells_1, d_cells_2, d_cells_3,
                                       d_parents_1, d_parents_2, d_parents_3,
                                       d_counts_1, d_counts_2, d_counts_3,
                                       d_new_indecies, d_map_to_new, d_map_to_old, n_total, n_1);


//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    clone << < 1, BLOCK_WIDTH >> > (d_restricted_dims_3, d_restricted_dims_1, number_of_restricted_dims_3);

    if (d_3 > 0) {
        numBlocks = d_3 / BLOCK_WIDTH;
        if (d_3 % BLOCK_WIDTH) numBlocks++;
        merge_update_dim << < numBlocks, BLOCK_WIDTH >> >
                                         (d_dim_start_1, d_dims_1, d_dim_start_2, d_dims_2, d_dim_start_3, d_dims_3, d_new_indecies, d_map_to_new, d_3, n_1);


//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
    }
//    hipDeviceSynchronize();
    //get number of points
    //number_of_points_3 = number_of_points_1 + number_of_points_2;
    hipMemcpy(h_tmp, d_counts_3, sizeof(int), hipMemcpyDeviceToHost);
    number_of_points_3 = h_tmp[0];


//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    //construct new point arrays
    hipMalloc(&d_points_3, number_of_points_3 * sizeof(int));
    hipMemset(d_points_3, 0, number_of_points_3 * sizeof(int));
    hipMalloc(&d_points_placement_3, number_of_points_3 * sizeof(int));
    hipMemset(d_points_placement_3, 0, number_of_points_3 * sizeof(int));


//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    // for each tree move points to new arrays
    numBlocks = number_of_points_3 / BLOCK_WIDTH;
    if (number_of_points_3 % BLOCK_WIDTH) numBlocks++;
    points_move << < numBlocks, BLOCK_WIDTH >> > (d_points_1, d_points_placement_1, number_of_points_1, n_1,
            d_points_2, d_points_placement_2, number_of_points_2,
            d_points_3, d_points_placement_3, number_of_points_3,
            d_new_indecies, d_map_to_new);


//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());


    if (n_3 == 1033) {
//        printf("\n\nMerged - Look here n=%d\nd_parents:\n", n_3);
//        print_array_gpu<<<1, 1>>>(d_parents_3, n_3);
//        hipDeviceSynchronize();
//        gpuErrchk(hipPeekAtLastError());
//
//        printf("d_new_indecies:\n");
//        print_array_gpu<<<1, 1>>>(d_new_indecies, n_total);
//        hipDeviceSynchronize();
//        gpuErrchk(hipPeekAtLastError());
//
//        printf("d_is_included:\n");
//        print_array_gpu<<<1, 1>>>(d_is_included, n_total);
//        hipDeviceSynchronize();
//        gpuErrchk(hipPeekAtLastError());
//
//        printf("d_map_to_new:\n");
//        print_array_gpu<<<1, 1>>>(d_map_to_new, n_total);
//        hipDeviceSynchronize();
//        gpuErrchk(hipPeekAtLastError());
//
//        printf("d_map_to_old:\n");
//        print_array_gpu<<<1, 1>>>(d_map_to_old, n_total);
//        hipDeviceSynchronize();
//        gpuErrchk(hipPeekAtLastError());

//        cmp c(d_new_indecies, d_map_to_new, d_parents_1, d_parents_2,
//                  d_cells_1, d_cells_2, d_counts_1, d_counts_2, n_1, n_2);
//
//        for (int i = -1; i < d_1; i++) {
//
//            int s_1 = i == -1 ? 0 : h_dim_start_1[i];
//            int s_2 = i == -1 ? 0 : h_dim_start_2[i];
//            int e_1 = i == -1 ? 1 : (i + 1 < d_1 ? h_dim_start_1[i + 1] : n_1);
//            int e_2 = i == -1 ? 1 : (i + 1 < d_1 ? h_dim_start_2[i + 1] : n_2);
//            for (int i = s_1; i < e_1; i++) {
//                for (int j = s_2; j < e_2; j++) {
//                    print_c <<<1, 1 >>>(c, i, j);
//                    hipDeviceSynchronize();
//                    gpuErrchk(hipPeekAtLastError());
//                }
//                printf("\n");
//            }
//            printf("\n");
//            printf("\n");
//        }
    }


//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());


    hipFree(d_map_to_old);
    hipFree(d_map_to_new);
    hipFree(d_is_included);
    hipFree(d_new_indecies);
    hipFree(pivots_1);
    hipFree(pivots_2);

//    hipDeviceSynchronize();
}

ScyTreeArray *restrict(ScyTreeArray *scy_tree, int dim_no, int cell_no) {
    //finding sizes and indexes
    int n = scy_tree->number_of_nodes;
    int c = scy_tree->number_of_cells;
    int d = scy_tree->number_of_dims;

    hipMemcpy(scy_tree->h_dims, scy_tree->d_dims, sizeof(int) * d, hipMemcpyDeviceToHost);
    hipMemcpy(scy_tree->h_dim_start, scy_tree->d_dim_start, sizeof(int) * d, hipMemcpyDeviceToHost);
    //hipDeviceSynchronize();

    int dim_i = 0;
    for (int i = 0; i < d; i++) {
        if (scy_tree->h_dims[i] == dim_no) {
            dim_i = i;
        }
    }

    //allocate tmp arrays
    int *d_new_indecies, *d_new_counts, *d_is_included, *d_is_s_connected;
    hipMalloc(&d_new_indecies, n * sizeof(int));
    hipMemset(d_new_indecies, 0, n * sizeof(int));
    hipMalloc(&d_new_counts, n * sizeof(int));
    hipMemset(d_new_counts, 0, n * sizeof(int));
    hipMalloc(&d_is_included, n * sizeof(int));
    hipMemset(d_is_included, 0, n * sizeof(int));
    gpuErrchk(hipPeekAtLastError());

    //hipDeviceSynchronize();

    memset << < 1, 1 >> > (d_is_included, 0, 1);//todo not a good way to do this
    hipMalloc(&d_is_s_connected, sizeof(int));
    hipMemset(d_is_s_connected, 0, sizeof(int));

    gpuErrchk(hipPeekAtLastError());

    //hipDeviceSynchronize();

    // 1. mark the nodes that should be included in the restriction
    //restrict dimension
    int lvl_size = scy_tree->get_lvl_size(dim_i);
    int number_of_blocks = lvl_size / BLOCK_WIDTH;
    if (lvl_size % BLOCK_WIDTH) number_of_blocks++;
    dim3 grid(number_of_blocks); //todo should be parallelized over c aswell
    dim3 block(BLOCK_WIDTH);
    restrict_dim << < grid, block >> > (scy_tree->d_parents, scy_tree->d_cells, scy_tree->d_counts, d_is_included,
            d_new_counts, cell_no, lvl_size, scy_tree->h_dim_start[dim_i], d_is_s_connected);


    gpuErrchk(hipPeekAtLastError());



    //propagrate up from restricted dim
    for (int d_j = dim_i - 1; d_j >= 0; d_j--) { // todo maybe in stream 2
        //todo maybe move for loop inside and stride instead of using blocks
        lvl_size = scy_tree->get_lvl_size(d_j);
        number_of_blocks = lvl_size / BLOCK_WIDTH;
        if (lvl_size % BLOCK_WIDTH) number_of_blocks++;
        dim3 grid_up(number_of_blocks);
        restrict_dim_prop_up << < grid_up, block >> >
                                           (scy_tree->d_parents, scy_tree->d_counts, d_is_included, d_new_counts,
                                                   lvl_size, scy_tree->h_dim_start[d_j]);
    }

    gpuErrchk(hipPeekAtLastError());

    //propagrate down from restricted dim
    if (dim_i + 1 < d) { //todo maybe in stream 1
        //todo maybe move for loop inside and stride instead of using blocks
        lvl_size = scy_tree->get_lvl_size(dim_i + 1);
        number_of_blocks = lvl_size / BLOCK_WIDTH;
        if (lvl_size % BLOCK_WIDTH) number_of_blocks++;
        dim3 grid_down(number_of_blocks);
        restrict_dim_prop_down_first << < grid_down, block >> > (scy_tree->d_parents, scy_tree->d_counts,
                scy_tree->d_cells, d_is_included, d_new_counts, cell_no, lvl_size, scy_tree->h_dim_start[dim_i +
                                                                                                         1]);
    }

    gpuErrchk(hipPeekAtLastError());

    for (int d_j = dim_i + 2; d_j < d; d_j++) { //todo maybe in stream 1
        //todo maybe move for loop inside and stride instead of using blocks
        lvl_size = scy_tree->get_lvl_size(d_j);
        number_of_blocks = lvl_size / BLOCK_WIDTH;
        if (lvl_size % BLOCK_WIDTH) number_of_blocks++;
        dim3 grid_down(number_of_blocks);
        restrict_dim_prop_down << < grid_down, block >> >
                                               (scy_tree->d_parents, scy_tree->d_counts, d_is_included, d_new_counts, lvl_size, scy_tree->h_dim_start[d_j]);
    }

//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    //hipDeviceSynchronize();

//    if (scy_tree->number_of_nodes == 3012) {
//
////        printf("d_is_included:\n");
////        print_array_gpu<<<1, 1>>>(d_is_included, scy_tree->number_of_nodes);
////        hipDeviceSynchronize();
////        gpuErrchk(hipPeekAtLastError());
////        int *h_is_included = new int[scy_tree->number_of_nodes];
////        hipMemcpy(h_is_included, d_is_included, sizeof(int) * scy_tree->number_of_nodes, hipMemcpyDeviceToHost);
////        hipDeviceSynchronize();
////        gpuErrchk(hipPeekAtLastError());
////        int sum = 0;
////        for (int i = 0; i < scy_tree->number_of_nodes; i++) {
////            sum += h_is_included[i];
////        }
////        printf("sum: %d\n", sum);
//
//    }


    // 2. do a scan to find the new indecies for the nodes in the restricted tree
    inclusive_scan(d_is_included, d_new_indecies, scy_tree->number_of_nodes);
    // 3. construct restricted tree

    gpuErrchk(hipPeekAtLastError());

    //hipDeviceSynchronize();



    int *h_tmp = new int[1];
    h_tmp[0] = 0;
    hipMemcpy(h_tmp, d_new_counts, sizeof(int), hipMemcpyDeviceToHost);
    int new_number_of_points = h_tmp[0];

    gpuErrchk(hipPeekAtLastError());


    hipMemcpy(h_tmp, scy_tree->d_counts, sizeof(int), hipMemcpyDeviceToHost);
    int number_of_points = h_tmp[0];

    hipMemcpy(h_tmp, d_new_indecies + scy_tree->number_of_nodes - 1, sizeof(int), hipMemcpyDeviceToHost);
    int new_number_of_nodes = h_tmp[0];


    gpuErrchk(hipPeekAtLastError());

    ScyTreeArray *restricted_scy_tree = new ScyTreeArray(new_number_of_nodes, scy_tree->number_of_dims - 1,
                                                         scy_tree->number_of_restricted_dims + 1,
                                                         new_number_of_points, scy_tree->number_of_cells);


    restricted_scy_tree->cell_size = scy_tree->cell_size;
    hipMemcpy(h_tmp, d_is_s_connected, sizeof(int), hipMemcpyDeviceToHost);
    restricted_scy_tree->is_s_connected = (bool) h_tmp[0];


    gpuErrchk(hipPeekAtLastError());


    number_of_blocks = scy_tree->number_of_nodes / BLOCK_WIDTH;
    if (scy_tree->number_of_nodes % BLOCK_WIDTH) number_of_blocks++;
    restrict_move << < number_of_blocks, BLOCK_WIDTH >> >
                                         (scy_tree->d_cells, restricted_scy_tree->d_cells,
                                                 scy_tree->d_parents, restricted_scy_tree->d_parents,
//                                                 scy_tree->d_node_order, restricted_scy_tree->d_node_order,
                                                 d_new_counts, restricted_scy_tree->d_counts,
                                                 d_new_indecies, d_is_included, scy_tree->number_of_nodes);


//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    if (restricted_scy_tree->number_of_dims > 0) {

        number_of_blocks = restricted_scy_tree->number_of_dims / BLOCK_WIDTH;
        if (restricted_scy_tree->number_of_dims % BLOCK_WIDTH) number_of_blocks++;


        restrict_update_dim << < number_of_blocks, BLOCK_WIDTH >> >
                                                   (scy_tree->d_dim_start, scy_tree->d_dims, restricted_scy_tree->d_dim_start,
                                                           restricted_scy_tree->d_dims, d_new_indecies, dim_i,
                                                           restricted_scy_tree->number_of_dims);

        gpuErrchk(hipPeekAtLastError());
    }

    number_of_blocks = restricted_scy_tree->number_of_restricted_dims / BLOCK_WIDTH;
    if (restricted_scy_tree->number_of_restricted_dims % BLOCK_WIDTH) number_of_blocks++;
    restrict_update_restricted_dim << < number_of_blocks, BLOCK_WIDTH >> >
                                                          (dim_no, scy_tree->d_restricted_dims, restricted_scy_tree->d_restricted_dims, scy_tree->number_of_restricted_dims);

    //hipDeviceSynchronize();


//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    int *d_is_point_included, *d_point_new_indecies;
    hipMalloc(&d_is_point_included, number_of_points * sizeof(int));
    hipMalloc(&d_point_new_indecies, number_of_points * sizeof(int));
    hipMemset(d_is_point_included, 0, number_of_points * sizeof(int));


    //gpuErrchk(hipPeekAtLastError());

    bool restricted_dim_is_leaf = (dim_i == scy_tree->number_of_dims - 1);

    number_of_blocks = number_of_points / BLOCK_WIDTH;
    if (number_of_points % BLOCK_WIDTH) number_of_blocks++;
    compute_is_points_included << < number_of_blocks, BLOCK_WIDTH >> > (
            scy_tree->d_points, scy_tree->d_points_placement, scy_tree->d_parents, scy_tree->d_cells, d_is_included, d_is_point_included,
                    scy_tree->number_of_nodes, number_of_points, new_number_of_points, restricted_dim_is_leaf, cell_no);


    gpuErrchk(hipPeekAtLastError());

    inclusive_scan(d_is_point_included, d_point_new_indecies, number_of_points);
//    dim3 dimBlock(BLOCKSIZE,BLOCKSIZE);
//    dim3 dimGrid(1,1);
//    PrefixSum<<<dimGrid,dimBlock>>>(d_point_new_indecies,d_is_point_included,number_of_points,BLOCKSIZE);

//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());
//    printf("d_is_point_included:\n");
//    print_array_gpu<<<1,1>>>(d_is_point_included, number_of_points);



    move_points << < number_of_blocks, BLOCK_WIDTH >> > (scy_tree->d_parents, scy_tree->d_points,
            scy_tree->d_points_placement, restricted_scy_tree->d_points, restricted_scy_tree->d_points_placement,
            d_point_new_indecies, d_new_indecies, d_is_point_included, number_of_points, restricted_dim_is_leaf);

//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    //todo hipFree() temps
    //int *d_new_indecies, *d_new_counts, *d_is_included, *d_is_s_connected;
    //int *d_is_point_included, *d_point_new_indecies;
    hipFree(d_new_indecies);
    gpuErrchk(hipPeekAtLastError());
    hipFree(d_new_counts);
    gpuErrchk(hipPeekAtLastError());
    hipFree(d_is_included);
    gpuErrchk(hipPeekAtLastError());
    hipFree(d_is_s_connected);
    gpuErrchk(hipPeekAtLastError());
    hipFree(d_is_point_included);
    gpuErrchk(hipPeekAtLastError());
    hipFree(d_point_new_indecies);
    gpuErrchk(hipPeekAtLastError());

//    hipDeviceSynchronize();

//    if (restricted_scy_tree->number_of_nodes == 975) {
//        printf("\n\nLook here n=%d\nd_parents:\n", restricted_scy_tree->number_of_nodes);
//        print_array_gpu<<<1, 1>>>(restricted_scy_tree->d_parents, restricted_scy_tree->number_of_nodes);
//        hipDeviceSynchronize();
//        gpuErrchk(hipPeekAtLastError());
//
//        printf("d_new_indecies:\n");
//        print_array_gpu<<<1, 1>>>(d_new_indecies, scy_tree->number_of_nodes);
//        hipDeviceSynchronize();
//        gpuErrchk(hipPeekAtLastError());
//
//        printf("d_is_included:\n");
//        print_array_gpu<<<1, 1>>>(d_is_included, scy_tree->number_of_nodes);
//        hipDeviceSynchronize();
//        gpuErrchk(hipPeekAtLastError());
//
//        printf("old d_parents:\n");
//        print_array_gpu<<<1, 1>>>(scy_tree->d_parents, scy_tree->number_of_nodes);
//        hipDeviceSynchronize();
//
//        printf("old size:%d\n", scy_tree->number_of_nodes);
//    }

    return restricted_scy_tree;
}

ScyTreeArray *restrict3(ScyTreeArray *scy_tree, int dim_no, int cell_no) {
    int number_of_blocks;
    dim3 block(512);
    //gpuErrchk(hipPeekAtLastError());

    //finding sizes and indexes
    //int n = scy_tree->number_of_nodes;
    int c = scy_tree->number_of_cells;
    int d = scy_tree->number_of_dims;

    int *d_dim_i;
    hipMalloc(&d_dim_i, sizeof(int));//todo use pre-allocated memory
    find_dim_i << < 1, 1 >> > (d_dim_i, scy_tree->d_dims, dim_no, scy_tree->number_of_dims);

    //allocate tmp arrays
    int *d_new_indecies, *d_new_counts, *d_is_included, *d_is_s_connected;
    hipMalloc(&d_new_indecies, scy_tree->number_of_nodes * sizeof(int));
    hipMemset(d_new_indecies, 0, scy_tree->number_of_nodes * sizeof(int));
    hipMalloc(&d_new_counts, scy_tree->number_of_nodes * sizeof(int));
    hipMemset(d_new_counts, 0, scy_tree->number_of_nodes * sizeof(int));
    hipMalloc(&d_is_included, scy_tree->number_of_nodes * sizeof(int));
    hipMemset(d_is_included, 0, scy_tree->number_of_nodes * sizeof(int));

    //hipDeviceSynchronize();

    memset << < 1, 1 >> > (d_is_included, 0, 1);//todo not a good way to do this
    hipMalloc(&d_is_s_connected, sizeof(int));
    hipMemset(d_is_s_connected, 0, sizeof(int));

//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    //hipDeviceSynchronize();

    // 1. mark the nodes that should be included in the restriction
    //restrict dimension
    restrict_dim_3 << < 1, block >> > (scy_tree->d_parents, scy_tree->d_cells, scy_tree->d_counts, d_is_included,
            d_new_counts, cell_no, scy_tree->d_dim_start, d_dim_i, d_is_s_connected, scy_tree->number_of_dims, scy_tree->number_of_nodes); //todo move h_dim_start[dim_i] to kernel


//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());



    //propagrate up from restricted dim

    restrict_dim_prop_up_3 << < 1, block >> >
                                   (scy_tree->d_parents, scy_tree->d_counts, d_is_included, d_new_counts,
                                           d_dim_i, scy_tree->d_dim_start, scy_tree->number_of_dims, scy_tree->number_of_nodes);


//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    //propagrate down from restricted dim
    restrict_dim_prop_down_first_3 << < 1, block >> >
                                           (scy_tree->d_parents, scy_tree->d_counts, scy_tree->d_cells, d_is_included, d_new_counts,
                                                   scy_tree->d_dim_start, d_dim_i,
                                                   cell_no, scy_tree->number_of_dims, scy_tree->number_of_nodes);

//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    restrict_dim_prop_down_3 << < 1, block >> >
                                     (scy_tree->d_parents, scy_tree->d_counts, d_is_included, d_new_counts,
                                             scy_tree->d_dim_start, d_dim_i,
                                             scy_tree->number_of_dims, scy_tree->number_of_nodes);

//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

//    if (scy_tree->number_of_nodes == 3012) {
//
////        printf("d_is_included:\n");
////        print_array_gpu<<<1, 1>>>(d_is_included, scy_tree->number_of_nodes);
////        hipDeviceSynchronize();
////        gpuErrchk(hipPeekAtLastError());
////        int *h_is_included = new int[scy_tree->number_of_nodes];
////        hipMemcpy(h_is_included, d_is_included, sizeof(int) * scy_tree->number_of_nodes, hipMemcpyDeviceToHost);
////        hipDeviceSynchronize();
////        gpuErrchk(hipPeekAtLastError());
////        int sum = 0;
////        for (int i = 0; i < scy_tree->number_of_nodes; i++) {
////            sum += h_is_included[i];
////        }
////        printf("sum: %d\n", sum);
//
//    }

    // 2. do a scan to find the new indecies for the nodes in the restricted tree
    inclusive_scan(d_is_included, d_new_indecies, scy_tree->number_of_nodes);

//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    // 3. construct restricted tree
    int *h_tmp = new int[1];
    h_tmp[0] = 0;
    hipMemcpy(h_tmp, d_new_counts, sizeof(int), hipMemcpyDeviceToHost);
    int new_number_of_points = h_tmp[0];

    //gpuErrchk(hipPeekAtLastError());

    hipMemcpy(h_tmp, scy_tree->d_counts, sizeof(int), hipMemcpyDeviceToHost);
    int number_of_points = h_tmp[0];

    hipMemcpy(h_tmp, d_new_indecies + scy_tree->number_of_nodes - 1, sizeof(int), hipMemcpyDeviceToHost);
    int new_number_of_nodes = h_tmp[0];


    //gpuErrchk(hipPeekAtLastError());
    //ScyTreeArray(int number_of_nodes, int number_of_dims, int number_of_restricted_dims, int number_of_points, int number_of_cells)
    ScyTreeArray *restricted_scy_tree = new ScyTreeArray(new_number_of_nodes, scy_tree->number_of_dims - 1,
                                                         scy_tree->number_of_restricted_dims + 1,
                                                         new_number_of_points, scy_tree->number_of_cells);

    restricted_scy_tree->cell_size = scy_tree->cell_size;//todo maybe not used
    hipMemcpy(h_tmp, d_is_s_connected, sizeof(int), hipMemcpyDeviceToHost);
    restricted_scy_tree->is_s_connected = (bool) h_tmp[0];


    //gpuErrchk(hipPeekAtLastError());


    number_of_blocks = scy_tree->number_of_nodes / BLOCK_WIDTH;
    if (scy_tree->number_of_nodes % BLOCK_WIDTH) number_of_blocks++;
    restrict_move << < number_of_blocks, BLOCK_WIDTH >> >
                                         (scy_tree->d_cells, restricted_scy_tree->d_cells,
                                                 scy_tree->d_parents, restricted_scy_tree->d_parents,
//                                                 scy_tree->d_node_order, restricted_scy_tree->d_node_order,
                                                 d_new_counts, restricted_scy_tree->d_counts,
                                                 d_new_indecies, d_is_included, scy_tree->number_of_nodes);


    //gpuErrchk(hipPeekAtLastError());

    if (restricted_scy_tree->number_of_dims > 0) {

        number_of_blocks = restricted_scy_tree->number_of_dims / BLOCK_WIDTH;
        if (restricted_scy_tree->number_of_dims % BLOCK_WIDTH) number_of_blocks++;


        restrict_update_dim_3 << < number_of_blocks, BLOCK_WIDTH >> >
                                                     (scy_tree->d_dim_start, scy_tree->d_dims, restricted_scy_tree->d_dim_start,
                                                             restricted_scy_tree->d_dims, d_new_indecies,
                                                             d_dim_i,
                                                             restricted_scy_tree->number_of_dims);

        //gpuErrchk(hipPeekAtLastError());
    }

    number_of_blocks = restricted_scy_tree->number_of_restricted_dims / BLOCK_WIDTH;
    if (restricted_scy_tree->number_of_restricted_dims % BLOCK_WIDTH) number_of_blocks++;
    restrict_update_restricted_dim << < number_of_blocks, BLOCK_WIDTH >> >
                                                          (dim_no, scy_tree->d_restricted_dims, restricted_scy_tree->d_restricted_dims, scy_tree->number_of_restricted_dims);

    //hipDeviceSynchronize();


    //gpuErrchk(hipPeekAtLastError());

    int *d_is_point_included, *d_point_new_indecies;
    hipMalloc(&d_is_point_included, number_of_points * sizeof(int));
    hipMalloc(&d_point_new_indecies, number_of_points * sizeof(int));
    hipMemset(d_is_point_included, 0, number_of_points * sizeof(int));


    //gpuErrchk(hipPeekAtLastError());



    number_of_blocks = number_of_points / BLOCK_WIDTH;
    if (number_of_points % BLOCK_WIDTH) number_of_blocks++;
    compute_is_points_included_3 << < number_of_blocks, BLOCK_WIDTH >> >
                                                        (scy_tree->d_points_placement, scy_tree->d_cells, d_is_included,
                                                                d_is_point_included, d_dim_i,
                                                                scy_tree->number_of_dims, scy_tree->number_of_points, cell_no);


    //gpuErrchk(hipPeekAtLastError());

    inclusive_scan(d_is_point_included, d_point_new_indecies, number_of_points);


    //gpuErrchk(hipPeekAtLastError());

    move_points_3 << < number_of_blocks, BLOCK_WIDTH >> > (scy_tree->d_parents, scy_tree->d_points,
            scy_tree->d_points_placement, restricted_scy_tree->d_points, restricted_scy_tree->d_points_placement,
            d_point_new_indecies, d_new_indecies, d_is_point_included, d_dim_i,
            number_of_points, scy_tree->number_of_dims);

    //hipDeviceSynchronize();


    //gpuErrchk(hipPeekAtLastError());



//    if (restricted_scy_tree->number_of_nodes == 946) {
//        printf("\n\nLook here n=%d\nd_parents:\n", restricted_scy_tree->number_of_nodes);
//        print_array_gpu<<<1, 1>>>(restricted_scy_tree->d_parents, restricted_scy_tree->number_of_nodes);
//        hipDeviceSynchronize();
//        gpuErrchk(hipPeekAtLastError());
//
//        printf("d_new_indecies:\n");
//        print_array_gpu<<<1, 1>>>(d_new_indecies, scy_tree->number_of_nodes);
//        hipDeviceSynchronize();
//        gpuErrchk(hipPeekAtLastError());
//
//        printf("d_is_included:\n");
//        print_array_gpu<<<1, 1>>>(d_is_included, scy_tree->number_of_nodes);
//        hipDeviceSynchronize();
//        gpuErrchk(hipPeekAtLastError());
//
//        printf("old d_parents:\n");
//        print_array_gpu<<<1, 1>>>(scy_tree->d_parents, scy_tree->number_of_nodes);
//        hipDeviceSynchronize();
//
//        printf("old size:%d\n", scy_tree->number_of_nodes);
//    }



    //todo hipFree() temps
    hipFree(d_new_indecies);
    hipFree(d_new_counts);
    hipFree(d_is_included);

//    hipDeviceSynchronize();

    return restricted_scy_tree;
}

int ScyTreeArray::get_lvl_size(int d_i) {
    return (d_i == this->number_of_dims - 1 ? this->number_of_nodes : this->h_dim_start[d_i + 1]) -
           this->h_dim_start[d_i];
}

ScyTreeArray *ScyTreeArray::restrict_gpu(int dim_no, int cell_no) {
    ScyTreeArray *restricted_scy_tree = restrict(this, dim_no, cell_no);

    return restricted_scy_tree;
}

ScyTreeArray *ScyTreeArray::restrict3_gpu(int dim_no, int cell_no) {
    ScyTreeArray *restricted_scy_tree = restrict3(this, dim_no, cell_no);

    return restricted_scy_tree;
}

vector <vector<ScyTreeArray *>>
ScyTreeArray::restrict_gpu_multi(int first_dim_no, int number_of_dims,
                                 int number_of_cells) {//todo  number_of_dims is different from this->number_of_dims find a better name

    //restricted-tree := restrict(scy-tree, descriptor);

    ScyTreeArray *scy_tree = this;

    int number_of_blocks;
    dim3 block(128);
    dim3 grid(number_of_dims, number_of_cells);
//    printf("grid(%d, %d)\n", number_of_dims, number_of_cells);

    int c = scy_tree->number_of_cells;
    int d = scy_tree->number_of_dims;

    int total_number_of_dim = first_dim_no + number_of_dims;
    int number_of_restrictions = number_of_dims * number_of_cells;

    vector <vector<ScyTreeArray *>> L(number_of_dims);



    //todo needs to be allocated for each - only dependent on the scy_tree
    //allocate tmp arrays - start
    int *d_new_indecies, *d_new_counts, *d_is_included;
    hipMalloc(&d_new_indecies, scy_tree->number_of_nodes * number_of_restrictions * sizeof(int));
    hipMalloc(&d_new_counts, scy_tree->number_of_nodes * number_of_restrictions * sizeof(int));
    hipMalloc(&d_is_included, scy_tree->number_of_nodes * number_of_restrictions * sizeof(int));

    hipMemset(d_new_indecies, 0, scy_tree->number_of_nodes * number_of_restrictions * sizeof(int));
    hipMemset(d_new_counts, 0, scy_tree->number_of_nodes * number_of_restrictions * sizeof(int));
    hipMemset(d_is_included, 0, scy_tree->number_of_nodes * number_of_restrictions * sizeof(int));
    for (int i = 0; i < number_of_dims; i++) {
        for (int cell_no = 0; cell_no < number_of_cells; cell_no++) {
            int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
            memset << < 1, 1 >> > (d_is_included + node_offset, 0, 1);//todo not a good way to do this
        }
    }

    int *d_is_point_included, *d_point_new_indecies;
    hipMalloc(&d_is_point_included, number_of_points * number_of_restrictions * sizeof(int));
    hipMalloc(&d_point_new_indecies, number_of_points * number_of_restrictions * sizeof(int));
    hipMemset(d_is_point_included, 0, number_of_points * number_of_restrictions * sizeof(int));

    int *d_is_s_connected;
    hipMalloc(&d_is_s_connected, number_of_restrictions * sizeof(int));
    hipMemset(d_is_s_connected, 0, number_of_restrictions * sizeof(int));

    int *d_dim_i;
    hipMalloc(&d_dim_i, number_of_dims * sizeof(int));

//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    int *h_new_number_of_points = new int[number_of_restrictions];
    int *h_new_number_of_nodes = new int[number_of_restrictions];
    //allocate tmp arrays - end

    int dim_no = first_dim_no;
    while (dim_no < total_number_of_dim) {
        int i = dim_no - first_dim_no;
        L[i] = vector<ScyTreeArray *>(number_of_cells);

        //todo find each dim that are being restricted - same for all cells - dependent on the scy_tree and dim
        find_dim_i << < 1, 1 >> >
                           (d_dim_i + i, scy_tree->d_dims, dim_no, scy_tree->number_of_dims);
        dim_no++;
    }

    if (number_of_dims > 0) {

        if (true) {
            restrict_dim_multi << < grid, block >> >
                                          (scy_tree->d_parents, scy_tree->d_cells, scy_tree->d_counts, scy_tree->d_dim_start,
                                                  d_is_included, d_new_counts, d_is_s_connected, d_dim_i,
                                                  scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                  scy_tree->number_of_cells, scy_tree->number_of_points);
//            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());

            restrict_dim_prop_up_multi << < grid, block >> >
                                                  (scy_tree->d_parents, scy_tree->d_counts, scy_tree->d_dim_start,
                                                          d_is_included, d_new_counts, d_dim_i,
                                                          scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                          scy_tree->number_of_cells, scy_tree->number_of_points);

//            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());

            restrict_dim_prop_down_first_multi << < grid, block >> >
                                                          (scy_tree->d_parents, scy_tree->d_counts, scy_tree->d_cells, scy_tree->d_dim_start,
                                                                  d_is_included, d_new_counts, d_dim_i,
                                                                  scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                                  scy_tree->number_of_cells, scy_tree->number_of_points);

//            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());

            restrict_dim_prop_down_multi << < grid, block >> >
                                                    (scy_tree->d_parents, scy_tree->d_counts, scy_tree->d_dim_start,
                                                            d_is_included, d_new_counts, d_dim_i,
                                                            scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                            scy_tree->number_of_cells, scy_tree->number_of_points);
        } else {
            restrict_dim_once_and_for_all << < grid, block >> >
                                                     (scy_tree->d_parents, scy_tree->d_cells, scy_tree->d_counts, scy_tree->d_dim_start,
                                                             d_is_included, d_new_counts, d_is_s_connected, d_dim_i,
                                                             scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                             scy_tree->number_of_cells, scy_tree->number_of_points);
//            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());
        }

//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;


                // 2. do a scan to find the new indecies for the nodes in the restricted tree
                //todo should be done partial for each restriction - maybe this can be parallellized over blocks for each restriction
                //todo make a inclusive_scan_multi
                inclusive_scan(d_is_included + node_offset, d_new_indecies + node_offset,
                               scy_tree->number_of_nodes);

                // 3. construct restricted tree
//                hipDeviceSynchronize();
                gpuErrchk(hipPeekAtLastError());

                //todo find new_number_of_points and new_number_of_nodes for each restricted scy_tree
                int *h_tmp = new int[1];
                h_tmp[0] = 0;
                hipMemcpy(h_tmp, d_new_counts + node_offset, sizeof(int), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                gpuErrchk(hipPeekAtLastError());
                int new_number_of_points = h_tmp[0];

                hipMemcpy(h_tmp, d_new_indecies + node_offset + scy_tree->number_of_nodes - 1, sizeof(int),
                           hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                gpuErrchk(hipPeekAtLastError());
                int new_number_of_nodes = h_tmp[0];
//            hipMemcpy(h_new_number_of_nodes + one_offset, d_new_indecies + node_offset + scy_tree->number_of_nodes - 1,
//                       sizeof(int),
//                       hipMemcpyDeviceToHost);
//            hipMemcpy(h_new_number_of_points + one_offset, d_new_counts + node_offset, sizeof(int),
//                       hipMemcpyDeviceToHost);


//                hipDeviceSynchronize();
                gpuErrchk(hipPeekAtLastError());

                //todo this is not needed??? we already have number_of_points
//                hipMemcpy(h_tmp, scy_tree->d_counts, sizeof(int), hipMemcpyDeviceToHost);
//                int number_of_points = h_tmp[0];
                //int number_of_points = scy_tree->number_of_points;//todo we are allready in this object???



                //gpuErrchk(hipPeekAtLastError());
                //todo create a new restricted scy_tree for each restriction

                if (new_number_of_points > scy_tree->number_of_points) {
                    printf("old_number_of_points: %d, new_number_of_points: %d\n", scy_tree->number_of_points,
                           new_number_of_points);
                    printf("old_number_of_nodes: %d, new_number_of_nodes: %d\n", new_number_of_nodes,
                           scy_tree->number_of_nodes);
                }
                ScyTreeArray *restricted_scy_tree = new ScyTreeArray(new_number_of_nodes,
                                                                     scy_tree->number_of_dims - 1,
                                                                     scy_tree->number_of_restricted_dims + 1,
                                                                     new_number_of_points,
                                                                     scy_tree->number_of_cells);
//                hipDeviceSynchronize();
                gpuErrchk(hipPeekAtLastError());

                L[i][cell_no] = restricted_scy_tree;

                //todo set is s-connected for each restriction
                restricted_scy_tree->cell_size = scy_tree->cell_size;//todo maybe not used
                hipMemcpy(h_tmp, d_is_s_connected + one_offset, sizeof(int), hipMemcpyDeviceToHost);
                restricted_scy_tree->is_s_connected = (bool) h_tmp[0];


//                hipDeviceSynchronize();
                gpuErrchk(hipPeekAtLastError());

                cell_no++;
            }
            dim_no++;
        }

        hipStream_t streams[number_of_restrictions];
        for (int i = 0; i < number_of_restrictions; i++) {
            hipStreamCreate(&streams[i]);
        }

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;
                ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                //todo parallellilize over restrictions
                number_of_blocks = scy_tree->number_of_nodes / BLOCK_WIDTH;
                if (scy_tree->number_of_nodes % BLOCK_WIDTH) number_of_blocks++;
                restrict_move << < number_of_blocks, BLOCK_WIDTH, 0, streams[one_offset] >> >
                                                                     (scy_tree->d_cells, restricted_scy_tree->d_cells,
                                                                             scy_tree->d_parents, restricted_scy_tree->d_parents,
                                                                             d_new_counts +
                                                                             node_offset, restricted_scy_tree->d_counts,
                                                                             d_new_indecies + node_offset,
                                                                             d_is_included +
                                                                             node_offset, scy_tree->number_of_nodes);

                cell_no++;
            }
            dim_no++;
        }
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;
                ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                //todo this if statement would be the same for all restrictions because it is allways restricted on one more than scy_tree - which is really nice!
                if (scy_tree->number_of_dims > 1) {//if not restricted on all dimensions

                    number_of_blocks = restricted_scy_tree->number_of_dims / BLOCK_WIDTH;
                    if (restricted_scy_tree->number_of_dims % BLOCK_WIDTH) number_of_blocks++;

                    //todo parallellilize over restrictions - maybe stride instead of distribute onto blocks - it would be easier to read and code
                    restrict_update_dim_3 << < number_of_blocks, BLOCK_WIDTH, 0, streams[one_offset] >> >
                                                                                 (scy_tree->d_dim_start, scy_tree->d_dims,
                                                                                         restricted_scy_tree->d_dim_start,
                                                                                         restricted_scy_tree->d_dims,
                                                                                         d_new_indecies + node_offset,
                                                                                         d_dim_i +
                                                                                         i, restricted_scy_tree->number_of_dims);

                }

                cell_no++;
            }
            dim_no++;
        }
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;
                ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                //todo parallellilize over restrictions - maybe stride instead of distribute onto blocks - it would be easier to read and code
                number_of_blocks = restricted_scy_tree->number_of_restricted_dims / BLOCK_WIDTH;
                if (restricted_scy_tree->number_of_restricted_dims % BLOCK_WIDTH) number_of_blocks++;
                restrict_update_restricted_dim << < number_of_blocks, BLOCK_WIDTH, 0, streams[one_offset] >> >
                                                                                      (dim_no, scy_tree->d_restricted_dims, restricted_scy_tree->d_restricted_dims, scy_tree->number_of_restricted_dims);


                cell_no++;
            }
            dim_no++;
        }
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;
                ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                number_of_blocks = number_of_points / BLOCK_WIDTH;
                if (number_of_points % BLOCK_WIDTH) number_of_blocks++;
                compute_is_points_included_3 << < number_of_blocks, BLOCK_WIDTH, 0, streams[one_offset] >> >
                                                                                    (scy_tree->d_points_placement, scy_tree->d_cells,
                                                                                            d_is_included + node_offset,
                                                                                            d_is_point_included +
                                                                                            point_offset,
                                                                                            d_dim_i + i,
                                                                                            scy_tree->number_of_dims, scy_tree->number_of_points, cell_no);


                cell_no++;
            }
            dim_no++;
        }
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;
                ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                hipStreamSynchronize(streams[one_offset]);
                inclusive_scan(d_is_point_included + point_offset,
                               d_point_new_indecies + point_offset,
                               number_of_points);

                cell_no++;
            }
            dim_no++;
        }
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;
                ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                //todo parallellilize over restrictions - maybe stride instead of distribute onto blocks - it would be easier to read and code
                move_points_3 << < number_of_blocks, BLOCK_WIDTH, 0, streams[one_offset] >> > (scy_tree->d_parents,
                        scy_tree->d_points, scy_tree->d_points_placement, restricted_scy_tree->d_points,
                        restricted_scy_tree->d_points_placement, d_point_new_indecies + point_offset,
                        d_new_indecies + node_offset, d_is_point_included + point_offset, d_dim_i + i,
                        number_of_points, scy_tree->number_of_dims);

                cell_no++;
            }
            dim_no++;
        }
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

    }

//    hipFree(d_new_indecies);
//    hipFree(d_new_counts);
//    hipFree(d_is_included);
//    hipFree(d_is_s_connected);
//    hipFree(d_is_point_included);
//    hipFree(d_point_new_indecies);
    return L;
}

vector <vector<ScyTreeArray *>>
ScyTreeArray::restrict_gpu_multi(TmpMalloc *tmps, int first_dim_no, int number_of_dims,
                                 int number_of_cells) {//todo  number_of_dims is different from this->number_of_dims find a better name

    //restricted-tree := restrict(scy-tree, descriptor);
    tmps->reset_counters();

    ScyTreeArray *scy_tree = this;

    int number_of_blocks;
    dim3 block(128);
    dim3 grid(number_of_dims, number_of_cells);
//    printf("grid(%d, %d)\n", number_of_dims, number_of_cells);

    int c = scy_tree->number_of_cells;
    int d = scy_tree->number_of_dims;

    int total_number_of_dim = first_dim_no + number_of_dims;
    int number_of_restrictions = number_of_dims * number_of_cells;

    vector <vector<ScyTreeArray *>> L(number_of_dims);



    //todo needs to be allocated for each - only dependent on the scy_tree
    //allocate tmp arrays - start
    int *d_new_indecies = tmps->get_int_array(tmps->int_array_counter++, scy_tree->number_of_nodes *
                                                                         number_of_restrictions);//tmps->d_new_indecies;
    int *d_new_counts = tmps->get_int_array(tmps->int_array_counter++,
                                            scy_tree->number_of_nodes * number_of_restrictions);//tmps->d_new_counts;
    int *d_is_included = tmps->get_int_array(tmps->int_array_counter++,
                                             scy_tree->number_of_nodes * number_of_restrictions);//tmps->d_is_included;

    hipMemset(d_new_indecies, 0, scy_tree->number_of_nodes * number_of_restrictions * sizeof(int));
    hipMemset(d_new_counts, 0, scy_tree->number_of_nodes * number_of_restrictions * sizeof(int));
    hipMemset(d_is_included, 0, scy_tree->number_of_nodes * number_of_restrictions * sizeof(int));
    for (int i = 0; i < number_of_dims; i++) {
        for (int cell_no = 0; cell_no < number_of_cells; cell_no++) {
            int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
            memset << < 1, 1 >> > (d_is_included + node_offset, 0, 1);//todo not a good way to do this
        }
    }

    int *d_is_point_included = tmps->get_int_array(tmps->int_array_counter++, scy_tree->number_of_points *
                                                                              number_of_restrictions);//tmps->d_is_point_included;
    int *d_point_new_indecies = tmps->get_int_array(tmps->int_array_counter++, scy_tree->number_of_points *
                                                                               number_of_restrictions);//tmps->d_point_new_indecies;
    hipMemset(d_is_point_included, 0, number_of_points * number_of_restrictions * sizeof(int));

    int *d_is_s_connected = tmps->get_int_array(tmps->int_array_counter++,
                                                number_of_restrictions);//tmps->d_is_s_connected;
    hipMemset(d_is_s_connected, 0, number_of_restrictions * sizeof(int));

    int *d_dim_i = tmps->get_int_array(tmps->int_array_counter++, number_of_dims);//tmps->d_dim_i;

//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    int *h_new_number_of_points = new int[number_of_restrictions];
    int *h_new_number_of_nodes = new int[number_of_restrictions];
    //allocate tmp arrays - end

    int dim_no = first_dim_no;
    while (dim_no < total_number_of_dim) {
        int i = dim_no - first_dim_no;
        L[i] = vector<ScyTreeArray *>(number_of_cells);

        //todo find each dim that are being restricted - same for all cells - dependent on the scy_tree and dim
        find_dim_i << < 1, 1 >> >
                           (d_dim_i + i, scy_tree->d_dims, dim_no, scy_tree->number_of_dims);
        dim_no++;
    }

    if (number_of_dims > 0) {

        if (true) {
            restrict_dim_multi << < grid, block >> >
                                          (scy_tree->d_parents, scy_tree->d_cells, scy_tree->d_counts, scy_tree->d_dim_start,
                                                  d_is_included, d_new_counts, d_is_s_connected, d_dim_i,
                                                  scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                  scy_tree->number_of_cells, scy_tree->number_of_points);
//            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());

            restrict_dim_prop_up_multi << < grid, block >> >
                                                  (scy_tree->d_parents, scy_tree->d_counts, scy_tree->d_dim_start,
                                                          d_is_included, d_new_counts, d_dim_i,
                                                          scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                          scy_tree->number_of_cells, scy_tree->number_of_points);

//            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());

            restrict_dim_prop_down_first_multi << < grid, block >> >
                                                          (scy_tree->d_parents, scy_tree->d_counts, scy_tree->d_cells, scy_tree->d_dim_start,
                                                                  d_is_included, d_new_counts, d_dim_i,
                                                                  scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                                  scy_tree->number_of_cells, scy_tree->number_of_points);

//            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());

            restrict_dim_prop_down_multi << < grid, block >> >
                                                    (scy_tree->d_parents, scy_tree->d_counts, scy_tree->d_dim_start,
                                                            d_is_included, d_new_counts, d_dim_i,
                                                            scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                            scy_tree->number_of_cells, scy_tree->number_of_points);
        } else {
            restrict_dim_once_and_for_all << < grid, block >> >
                                                     (scy_tree->d_parents, scy_tree->d_cells, scy_tree->d_counts, scy_tree->d_dim_start,
                                                             d_is_included, d_new_counts, d_is_s_connected, d_dim_i,
                                                             scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                             scy_tree->number_of_cells, scy_tree->number_of_points);
//            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());
        }

//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;


                // 2. do a scan to find the new indecies for the nodes in the restricted tree
                //todo should be done partial for each restriction - maybe this can be parallellized over blocks for each restriction
                //todo make a inclusive_scan_multi
                inclusive_scan(d_is_included + node_offset, d_new_indecies + node_offset,
                               scy_tree->number_of_nodes);

                // 3. construct restricted tree
//                hipDeviceSynchronize();
                gpuErrchk(hipPeekAtLastError());

                //todo find new_number_of_points and new_number_of_nodes for each restricted scy_tree
                int *h_tmp = new int[1];
                h_tmp[0] = 0;
                hipMemcpy(h_tmp, d_new_counts + node_offset, sizeof(int), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                gpuErrchk(hipPeekAtLastError());
                int new_number_of_points = h_tmp[0];

                hipMemcpy(h_tmp, d_new_indecies + node_offset + scy_tree->number_of_nodes - 1, sizeof(int),
                           hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                gpuErrchk(hipPeekAtLastError());
                int new_number_of_nodes = h_tmp[0];

//                hipDeviceSynchronize();
                gpuErrchk(hipPeekAtLastError());

                //gpuErrchk(hipPeekAtLastError());
                //todo create a new restricted scy_tree for each restriction

                if (new_number_of_points > scy_tree->number_of_points) {
                    printf("old_number_of_points: %d, new_number_of_points: %d\n", scy_tree->number_of_points,
                           new_number_of_points);
                    printf("old_number_of_nodes: %d, new_number_of_nodes: %d\n", new_number_of_nodes,
                           scy_tree->number_of_nodes);
                }
                ScyTreeArray *restricted_scy_tree = new ScyTreeArray(new_number_of_nodes,
                                                                     scy_tree->number_of_dims - 1,
                                                                     scy_tree->number_of_restricted_dims + 1,
                                                                     new_number_of_points,
                                                                     scy_tree->number_of_cells);
//                hipDeviceSynchronize();
                gpuErrchk(hipPeekAtLastError());

                L[i][cell_no] = restricted_scy_tree;

                //todo set is s-connected for each restriction
                restricted_scy_tree->cell_size = scy_tree->cell_size;//todo maybe not used
                hipMemcpy(h_tmp, d_is_s_connected + one_offset, sizeof(int), hipMemcpyDeviceToHost);
                restricted_scy_tree->is_s_connected = (bool) h_tmp[0];


//                hipDeviceSynchronize();
                gpuErrchk(hipPeekAtLastError());

                cell_no++;
            }
            dim_no++;
        }

        hipStream_t streams[min(10, number_of_restrictions)];
        for (int i = 0; i < min(10, number_of_restrictions); i++) {
            hipStreamCreate(&streams[i]);
        }

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;
                ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                //todo parallellilize over restrictions
                number_of_blocks = scy_tree->number_of_nodes / BLOCK_WIDTH;
                if (scy_tree->number_of_nodes % BLOCK_WIDTH) number_of_blocks++;
                restrict_move << < number_of_blocks, BLOCK_WIDTH, 0, streams[one_offset % 10] >> >
                                                                     (scy_tree->d_cells, restricted_scy_tree->d_cells,
                                                                             scy_tree->d_parents, restricted_scy_tree->d_parents,
                                                                             d_new_counts +
                                                                             node_offset, restricted_scy_tree->d_counts,
                                                                             d_new_indecies + node_offset,
                                                                             d_is_included +
                                                                             node_offset, scy_tree->number_of_nodes);

                cell_no++;
            }
            dim_no++;
        }
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;
                ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                //todo this if statement would be the same for all restrictions because it is allways restricted on one more than scy_tree - which is really nice!
                if (scy_tree->number_of_dims > 1) {//if not restricted on all dimensions

                    number_of_blocks = restricted_scy_tree->number_of_dims / BLOCK_WIDTH;
                    if (restricted_scy_tree->number_of_dims % BLOCK_WIDTH) number_of_blocks++;

                    //todo parallellilize over restrictions - maybe stride instead of distribute onto blocks - it would be easier to read and code
                    restrict_update_dim_3 << < number_of_blocks, BLOCK_WIDTH, 0, streams[one_offset % 10] >> >
                                                                                 (scy_tree->d_dim_start, scy_tree->d_dims,
                                                                                         restricted_scy_tree->d_dim_start,
                                                                                         restricted_scy_tree->d_dims,
                                                                                         d_new_indecies + node_offset,
                                                                                         d_dim_i +
                                                                                         i, restricted_scy_tree->number_of_dims);

                }

                cell_no++;
            }
            dim_no++;
        }
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;
                ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                //todo parallellilize over restrictions - maybe stride instead of distribute onto blocks - it would be easier to read and code
                number_of_blocks = restricted_scy_tree->number_of_restricted_dims / BLOCK_WIDTH;
                if (restricted_scy_tree->number_of_restricted_dims % BLOCK_WIDTH) number_of_blocks++;
                restrict_update_restricted_dim << < number_of_blocks, BLOCK_WIDTH, 0, streams[one_offset % 10] >> >
                                                                                      (dim_no, scy_tree->d_restricted_dims, restricted_scy_tree->d_restricted_dims, scy_tree->number_of_restricted_dims);


                cell_no++;
            }
            dim_no++;
        }
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;
                ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                number_of_blocks = number_of_points / BLOCK_WIDTH;
                if (number_of_points % BLOCK_WIDTH) number_of_blocks++;
                compute_is_points_included_3 << < number_of_blocks, BLOCK_WIDTH, 0, streams[one_offset % 10] >> >
                                                                                    (scy_tree->d_points_placement, scy_tree->d_cells,
                                                                                            d_is_included + node_offset,
                                                                                            d_is_point_included +
                                                                                            point_offset,
                                                                                            d_dim_i + i,
                                                                                            scy_tree->number_of_dims, scy_tree->number_of_points, cell_no);


                cell_no++;
            }
            dim_no++;
        }
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;
                ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                hipStreamSynchronize(streams[one_offset % 10]);
                inclusive_scan(d_is_point_included + point_offset,
                               d_point_new_indecies + point_offset,
                               number_of_points);

                cell_no++;
            }
            dim_no++;
        }
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;
                ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                //todo parallellilize over restrictions - maybe stride instead of distribute onto blocks - it would be easier to read and code
                move_points_3 << < number_of_blocks, BLOCK_WIDTH, 0, streams[one_offset % 10] >> > (scy_tree->d_parents,
                        scy_tree->d_points, scy_tree->d_points_placement, restricted_scy_tree->d_points,
                        restricted_scy_tree->d_points_placement, d_point_new_indecies + point_offset,
                        d_new_indecies + node_offset, d_is_point_included + point_offset, d_dim_i + i,
                        number_of_points, scy_tree->number_of_dims);

                cell_no++;
            }
            dim_no++;
        }
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
        for (int i = 0; i < min(number_of_restrictions, 10); i++) {
            hipStreamDestroy(streams[i]);
        }
    }


    return L;
}


vector <vector<ScyTreeArray *>>
ScyTreeArray::restrict_merge_gpu_multi(TmpMalloc *tmps, int first_dim_no, int number_of_dims,
                                       int number_of_cells) {//todo  number_of_dims is different from this->number_of_dims find a better name

    //restricted-tree := restrict(scy-tree, descriptor);

    ScyTreeArray *scy_tree = this;

    tmps->reset_counters();


    int number_of_blocks;
    dim3 block(128);
    dim3 grid(number_of_dims, number_of_cells);

    int c = scy_tree->number_of_cells;
    int d = scy_tree->number_of_dims;

    int total_number_of_dim = first_dim_no + number_of_dims;
    int number_of_restrictions = number_of_dims * number_of_cells;

    vector <vector<ScyTreeArray *>> L(number_of_dims);

    vector <vector<ScyTreeArray *>> L_merged(number_of_dims);

    if (scy_tree->number_of_nodes * number_of_restrictions == 0)
        return L_merged;

    //allocate tmp arrays - start
    int *d_new_indecies = tmps->get_int_array(tmps->int_array_counter++, scy_tree->number_of_nodes *
                                                                         number_of_restrictions);//tmps->d_new_indecies;
    int *d_new_counts = tmps->get_int_array(tmps->int_array_counter++,
                                            scy_tree->number_of_nodes * number_of_restrictions);//tmps->d_new_counts;
    int *d_is_included = tmps->get_int_array(tmps->int_array_counter++,
                                             scy_tree->number_of_nodes * number_of_restrictions);//tmps->d_is_included;

    hipMemset(d_new_indecies, 0, scy_tree->number_of_nodes * number_of_restrictions * sizeof(int));
    hipMemset(d_new_counts, 0, scy_tree->number_of_nodes * number_of_restrictions * sizeof(int));
    hipMemset(d_is_included, 0, scy_tree->number_of_nodes * number_of_restrictions * sizeof(int));
    for (int i = 0; i < number_of_dims; i++) {
        for (int cell_no = 0; cell_no < number_of_cells; cell_no++) {
            int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
            memset << < 1, 1 >> > (d_is_included + node_offset, 0, 1);//todo not a good way to do this
        }
    }
    gpuErrchk(hipPeekAtLastError());

    int *d_is_point_included = tmps->get_int_array(tmps->int_array_counter++, this->number_of_points *
                                                                              number_of_restrictions);//tmps->d_is_point_included;
    gpuErrchk(hipPeekAtLastError());
    int *d_point_new_indecies = tmps->get_int_array(tmps->int_array_counter++, this->number_of_points *
                                                                               number_of_restrictions);//tmps->d_point_new_indecies;
    gpuErrchk(hipPeekAtLastError());

    hipMemset(d_is_point_included, 0, this->number_of_points * number_of_restrictions * sizeof(int));
    gpuErrchk(hipPeekAtLastError());

    int *d_is_s_connected = tmps->get_int_array(tmps->int_array_counter++,
                                                number_of_restrictions);//tmps->d_is_s_connected;
    hipMemset(d_is_s_connected, 0, number_of_restrictions * sizeof(int));
    gpuErrchk(hipPeekAtLastError());

    int *d_dim_i = tmps->get_int_array(tmps->int_array_counter++, number_of_dims);//tmps->d_dim_i;

    gpuErrchk(hipPeekAtLastError());

    int *h_new_number_of_points = new int[number_of_restrictions];
    int *h_new_number_of_nodes = new int[number_of_restrictions];

    int *d_merge_map = tmps->get_int_array(tmps->int_array_counter++, number_of_restrictions);
    int *h_merge_map = new int[number_of_restrictions];
//    hipMalloc(&d_merge_map, number_of_restrictions * sizeof(int));
    //allocate tmp arrays - end

    int dim_no = first_dim_no;
    while (dim_no < total_number_of_dim) {
        int i = dim_no - first_dim_no;
        L[i] = vector<ScyTreeArray *>(number_of_cells);

        //todo find each dim that are being restricted - same for all cells - dependent on the scy_tree and dim
        find_dim_i << < 1, 1 >> >
                           (d_dim_i + i, scy_tree->d_dims, dim_no, scy_tree->number_of_dims);
        dim_no++;
    }

    if (number_of_dims > 0) {

        if (true) {

            check_is_s_connected << < number_of_dims, block >> >
                                                      (scy_tree->d_parents, scy_tree->d_cells, scy_tree->d_counts, scy_tree->d_dim_start,
                                                              d_is_included, d_new_counts, d_is_s_connected, d_dim_i,
                                                              scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                              scy_tree->number_of_cells, scy_tree->number_of_points);
            gpuErrchk(hipPeekAtLastError());

            compute_merge_map << < 1, number_of_dims >> >//todo fix num_of_dims>1024
                                      (d_is_s_connected, d_merge_map, scy_tree->number_of_cells);
            gpuErrchk(hipPeekAtLastError());
            hipMemcpy(h_merge_map, d_merge_map, number_of_restrictions * sizeof(int), hipMemcpyDeviceToHost);
//            printf("is_s_connected:\n");
//            print_array_gpu<< <1,1>>>(d_is_s_connected, number_of_restrictions);
//            hipDeviceSynchronize();
//            printf("h_merge_map:\n");
//            print_array(h_merge_map, number_of_restrictions);

            gpuErrchk(hipPeekAtLastError());

            restrict_merge_dim_multi << < number_of_dims, block >> >
                                                          (scy_tree->d_parents, scy_tree->d_cells, scy_tree->d_counts, scy_tree->d_dim_start,
                                                                  d_is_included, d_new_counts, d_is_s_connected, d_dim_i, d_merge_map,
                                                                  scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                                  scy_tree->number_of_cells, scy_tree->number_of_points);
//            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());

            restrict_dim_prop_up_multi << < grid, block >> >
                                                  (scy_tree->d_parents, scy_tree->d_counts, scy_tree->d_dim_start,
                                                          d_is_included, d_new_counts, d_dim_i,
                                                          scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                          scy_tree->number_of_cells, scy_tree->number_of_points);

//            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());

            restrict_merge_dim_prop_down_first_multi << < grid, block >> >
                                                                (scy_tree->d_parents, scy_tree->d_counts, scy_tree->d_cells, scy_tree->d_dim_start,
                                                                        d_is_included, d_new_counts, d_dim_i, d_merge_map,
                                                                        scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                                        scy_tree->number_of_cells, scy_tree->number_of_points);

//            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());

            restrict_dim_prop_down_multi << < grid, block >> >
                                                    (scy_tree->d_parents, scy_tree->d_counts, scy_tree->d_dim_start,
                                                            d_is_included, d_new_counts, d_dim_i,
                                                            scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                            scy_tree->number_of_cells, scy_tree->number_of_points);
        } else {
            restrict_dim_once_and_for_all << < grid, block >> >
                                                     (scy_tree->d_parents, scy_tree->d_cells, scy_tree->d_counts, scy_tree->d_dim_start,
                                                             d_is_included, d_new_counts, d_is_s_connected, d_dim_i,
                                                             scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                             scy_tree->number_of_cells, scy_tree->number_of_points);
//            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());
        }

//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;

                if (cell_no == 0 || h_merge_map[one_offset - 1] != h_merge_map[one_offset]) {
                    // 2. do a scan to find the new indecies for the nodes in the restricted tree
                    //todo should be done partial for each restriction - maybe this can be parallellized over blocks for each restriction
                    //todo make a inclusive_scan_multi
                    inclusive_scan(d_is_included + node_offset, d_new_indecies + node_offset,
                                   scy_tree->number_of_nodes);

                    // 3. construct restricted tree
//                hipDeviceSynchronize();
                    gpuErrchk(hipPeekAtLastError());

                    //todo find new_number_of_points and new_number_of_nodes for each restricted scy_tree
                    int *h_tmp = new int[1];
                    h_tmp[0] = 0;
                    hipMemcpy(h_tmp, d_new_counts + node_offset, sizeof(int), hipMemcpyDeviceToHost);
                    hipDeviceSynchronize();
                    gpuErrchk(hipPeekAtLastError());
                    int new_number_of_points = h_tmp[0];

                    hipMemcpy(h_tmp, d_new_indecies + node_offset + scy_tree->number_of_nodes - 1, sizeof(int),
                               hipMemcpyDeviceToHost);
                    hipDeviceSynchronize();
                    gpuErrchk(hipPeekAtLastError());
                    int new_number_of_nodes = h_tmp[0];

//                hipDeviceSynchronize();
                    gpuErrchk(hipPeekAtLastError());

                    //gpuErrchk(hipPeekAtLastError());
                    //todo create a new restricted scy_tree for each restriction

                    if (new_number_of_points > scy_tree->number_of_points) {
                        printf("new_number_of_points: %d\n", new_number_of_points);
                        printf("new_number_of_nodes: %d\n", new_number_of_nodes);
                    }
                    ScyTreeArray *restricted_scy_tree = new ScyTreeArray(new_number_of_nodes,
                                                                         scy_tree->number_of_dims - 1,
                                                                         scy_tree->number_of_restricted_dims + 1,
                                                                         new_number_of_points,
                                                                         scy_tree->number_of_cells);
                    hipDeviceSynchronize();
                    gpuErrchk(hipPeekAtLastError());

                    L[i][cell_no] = restricted_scy_tree;
                    L_merged[i].push_back(restricted_scy_tree);

                    //todo set is s-connected for each restriction
                    restricted_scy_tree->cell_size = scy_tree->cell_size;//todo maybe not used
                    hipMemcpy(h_tmp, d_is_s_connected + one_offset, sizeof(int), hipMemcpyDeviceToHost);
                    restricted_scy_tree->is_s_connected = (bool) h_tmp[0];


                    hipDeviceSynchronize();
                    gpuErrchk(hipPeekAtLastError());
                    delete[] h_tmp;
                }
                cell_no++;
            }
            dim_no++;
        }

//        hipStream_t streams[number_of_restrictions];
//        for (int i = 0; i < min(number_of_restrictions, 10); i++) {
//            hipStreamCreate(&streams[i % 10]);
//        }

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;
                if (cell_no == 0 || h_merge_map[one_offset - 1] != h_merge_map[one_offset]) {
                    ScyTreeArray *restricted_scy_tree = L[i][cell_no];


                    if (scy_tree->number_of_nodes == 0) {
                        printf("something strange happened!\n");
                    }

                    //todo parallellilize over restrictions
                    number_of_blocks = scy_tree->number_of_nodes / BLOCK_WIDTH;
                    if (scy_tree->number_of_nodes % BLOCK_WIDTH) number_of_blocks++;
                    restrict_move << < number_of_blocks, BLOCK_WIDTH, 0 >> >//, streams[one_offset % 10]
                                                                      (scy_tree->d_cells, restricted_scy_tree->d_cells,
                                                                              scy_tree->d_parents, restricted_scy_tree->d_parents,
                                                                              d_new_counts +
                                                                              node_offset, restricted_scy_tree->d_counts,
                                                                              d_new_indecies + node_offset,
                                                                              d_is_included +
                                                                              node_offset, scy_tree->number_of_nodes);
                }
                cell_no++;
            }
            dim_no++;
        }
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;
                if (cell_no == 0 || h_merge_map[one_offset - 1] != h_merge_map[one_offset]) {
                    ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                    //todo this if statement would be the same for all restrictions because it is allways restricted on one more than scy_tree - which is really nice!
                    if (scy_tree->number_of_dims > 1) {//if not restricted on all dimensions

                        number_of_blocks = restricted_scy_tree->number_of_dims / BLOCK_WIDTH;
                        if (restricted_scy_tree->number_of_dims % BLOCK_WIDTH) number_of_blocks++;

                        //todo parallellilize over restrictions - maybe stride instead of distribute onto blocks - it would be easier to read and code
                        restrict_update_dim_3 << < number_of_blocks, BLOCK_WIDTH, 0 >> >
                                                                                  //, streams[one_offset % 10] >> >
                                                                                  (scy_tree->d_dim_start, scy_tree->d_dims,
                                                                                          restricted_scy_tree->d_dim_start,
                                                                                          restricted_scy_tree->d_dims,
                                                                                          d_new_indecies +
                                                                                          node_offset,
                                                                                          d_dim_i +
                                                                                          i, restricted_scy_tree->number_of_dims);

                    }
                }
                cell_no++;
            }
            dim_no++;
        }
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;
                if (cell_no == 0 || h_merge_map[one_offset - 1] != h_merge_map[one_offset]) {
                    ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                    //todo parallellilize over restrictions - maybe stride instead of distribute onto blocks - it would be easier to read and code
                    number_of_blocks = restricted_scy_tree->number_of_restricted_dims / BLOCK_WIDTH;
                    if (restricted_scy_tree->number_of_restricted_dims % BLOCK_WIDTH) number_of_blocks++;
                    restrict_update_restricted_dim << < number_of_blocks, BLOCK_WIDTH, 0 >> >
                                                                                       //, streams[one_offset % 10] >> >
                                                                                       (dim_no, scy_tree->d_restricted_dims, restricted_scy_tree->d_restricted_dims, scy_tree->number_of_restricted_dims);

                }
                cell_no++;
            }
            dim_no++;
        }
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;

                if (cell_no == 0 || h_merge_map[one_offset - 1] != h_merge_map[one_offset]) {
                    ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                    number_of_blocks = number_of_points / BLOCK_WIDTH;
                    if (number_of_points % BLOCK_WIDTH) number_of_blocks++;
                    restrict_merge_is_points_included
                            << < number_of_blocks, BLOCK_WIDTH, 0 >> >//, streams[one_offset % 10] >> >
                                                                (scy_tree->d_points_placement, scy_tree->d_cells,
                                                                        d_is_included + node_offset,
                                                                        d_is_point_included + point_offset,
                                                                        d_dim_i + i,
                                                                        d_merge_map + i * number_of_cells,
                                                                        scy_tree->number_of_dims, scy_tree->number_of_points, cell_no);

                }
                cell_no++;
            }
            dim_no++;
        }
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;

                if (cell_no == 0 || h_merge_map[one_offset - 1] != h_merge_map[one_offset]) {
                    ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                    //hipStreamSynchronize(streams[one_offset % 10]);
                    inclusive_scan(d_is_point_included + point_offset,
                                   d_point_new_indecies + point_offset,
                                   number_of_points);
                }
                cell_no++;
            }
            dim_no++;
        }
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;
                if (cell_no == 0 || h_merge_map[one_offset - 1] != h_merge_map[one_offset]) {
                    ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                    //todo parallellilize over restrictions - maybe stride instead of distribute onto blocks - it would be easier to read and code
                    move_points_3 << < number_of_blocks, BLOCK_WIDTH, 0 >> >//, streams[one_offset % 10] >> >
                                                                      (scy_tree->d_parents,
                                                                              scy_tree->d_points, scy_tree->d_points_placement, restricted_scy_tree->d_points,
                                                                              restricted_scy_tree->d_points_placement,
                                                                              d_point_new_indecies + point_offset,
                                                                              d_new_indecies + node_offset,
                                                                              d_is_point_included + point_offset,
                                                                              d_dim_i + i,
                                                                              number_of_points, scy_tree->number_of_dims);
                }
                cell_no++;
            }
            dim_no++;
        }
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

//        for (int i = 0; i < min(number_of_restrictions, 10); i++) {
//            hipStreamDestroy(streams[i]);
//        }
    }

    delete[] h_new_number_of_points;
    delete[] h_new_number_of_nodes;
    delete[] h_merge_map;

    return L_merged;
}

__global__
void check_parents(int *d_point_placement, int number_of_points, int *d_parents, int number_of_nodes) {
    if (threadIdx.x == 0)
        printf("checking...\n");
    for (int i = threadIdx.x; i < number_of_points; i += blockDim.x) {
        int node = d_point_placement[i];
        while (d_parents[node] != node) {
            if (node <= d_parents[node]) {
                printf("hmmm... node:%d, parent:%d, number_of_nodes:%d, d_point_placement[i]:%d\n", node,
                       d_parents[node], number_of_nodes, d_point_placement[i]);
                break;
            }
            node = d_parents[node];
        }
    }
}

vector <vector<ScyTreeArray *>>
ScyTreeArray::restrict_merge_gpu_multi2(TmpMalloc *tmps, int first_dim_no, int number_of_dims,
                                        int number_of_cells) {//todo  number_of_dims is different from this->number_of_dims find a better name

    //restricted-tree := restrict(scy-tree, descriptor);

    ScyTreeArray *scy_tree = this;

    tmps->reset_counters();


    int number_of_blocks;
    dim3 block(128);
    dim3 grid(number_of_dims, number_of_cells);

    int c = scy_tree->number_of_cells;
    int d = scy_tree->number_of_dims;

    int total_number_of_dim = first_dim_no + number_of_dims;
    int number_of_restrictions = number_of_dims * number_of_cells;

    vector <vector<ScyTreeArray *>> L(number_of_dims);

    vector <vector<ScyTreeArray *>> L_merged(number_of_dims);

    if (scy_tree->number_of_nodes * number_of_restrictions == 0)
        return L_merged;
    gpuErrchk(hipPeekAtLastError());

    //allocate tmp arrays - start
    int *d_new_indecies = tmps->get_int_array(tmps->int_array_counter++, scy_tree->number_of_nodes *
                                                                         number_of_restrictions);//tmps->d_new_indecies;
    gpuErrchk(hipPeekAtLastError());
    int *d_new_counts = tmps->get_int_array(tmps->int_array_counter++,
                                            scy_tree->number_of_nodes * number_of_restrictions);//tmps->d_new_counts;
    gpuErrchk(hipPeekAtLastError());
    int *d_is_included = tmps->get_int_array(tmps->int_array_counter++,
                                             scy_tree->number_of_nodes * number_of_restrictions);//tmps->d_is_included;
    gpuErrchk(hipPeekAtLastError());
    int *d_children_full = tmps->get_int_array(tmps->int_array_counter++,
                                               2 * scy_tree->number_of_nodes * number_of_restrictions *
                                               scy_tree->number_of_cells);
    gpuErrchk(hipPeekAtLastError());

    int *d_parents_full = tmps->get_int_array(tmps->int_array_counter++,
                                              scy_tree->number_of_nodes * number_of_restrictions);
    gpuErrchk(hipPeekAtLastError());

    hipMemset(d_new_indecies, 0, scy_tree->number_of_nodes * number_of_restrictions * sizeof(int));
    hipMemset(d_new_counts, 0, scy_tree->number_of_nodes * number_of_restrictions * sizeof(int));
    hipMemset(d_is_included, 0, scy_tree->number_of_nodes * number_of_restrictions * sizeof(int));
    hipMemset(d_parents_full, -1, scy_tree->number_of_nodes * number_of_restrictions * sizeof(int));
    hipMemset(d_children_full, -1,
               2 * scy_tree->number_of_nodes * number_of_restrictions * scy_tree->number_of_cells * sizeof(int));
    for (int i = 0; i < number_of_dims; i++) {
        for (int cell_no = 0; cell_no < number_of_cells; cell_no++) {
            int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
            memset << < 1, 1 >> > (d_is_included + node_offset, 0, 1);//todo not a good way to do this
        }
    }
    gpuErrchk(hipPeekAtLastError());

    int *d_is_point_included = tmps->get_int_array(tmps->int_array_counter++, this->number_of_points *
                                                                              number_of_restrictions);//tmps->d_is_point_included;
    gpuErrchk(hipPeekAtLastError());
    int *d_point_new_indecies = tmps->get_int_array(tmps->int_array_counter++, this->number_of_points *
                                                                               number_of_restrictions);//tmps->d_point_new_indecies;
    gpuErrchk(hipPeekAtLastError());

    hipMemset(d_is_point_included, 0, this->number_of_points * number_of_restrictions * sizeof(int));
    gpuErrchk(hipPeekAtLastError());

    int *d_is_s_connected = tmps->get_int_array(tmps->int_array_counter++,
                                                number_of_restrictions);//tmps->d_is_s_connected;
    hipMemset(d_is_s_connected, 0, number_of_restrictions * sizeof(int));
    gpuErrchk(hipPeekAtLastError());

    int *d_dim_i = tmps->get_int_array(tmps->int_array_counter++, number_of_dims);//tmps->d_dim_i;

    gpuErrchk(hipPeekAtLastError());

    int *h_new_number_of_points = new int[number_of_restrictions];
    int *h_new_number_of_nodes = new int[number_of_restrictions];

    int *d_merge_map = tmps->get_int_array(tmps->int_array_counter++, number_of_restrictions);
    int *h_merge_map = new int[number_of_restrictions];
//    hipMalloc(&d_merge_map, number_of_restrictions * sizeof(int));
    //allocate tmp arrays - end

    int dim_no = first_dim_no;
    while (dim_no < total_number_of_dim) {
        int i = dim_no - first_dim_no;
        L[i] = vector<ScyTreeArray *>(number_of_cells);

        //todo find each dim that are being restricted - same for all cells - dependent on the scy_tree and dim
        find_dim_i << < 1, 1 >> >
                           (d_dim_i + i, scy_tree->d_dims, dim_no, scy_tree->number_of_dims);
        dim_no++;
    }

    if (number_of_dims > 0) {

        if (true) {

            check_is_s_connected << < number_of_dims, block >> >
                                                      (scy_tree->d_parents, scy_tree->d_cells, scy_tree->d_counts, scy_tree->d_dim_start,
                                                              d_is_included, d_new_counts, d_is_s_connected, d_dim_i,
                                                              scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                              scy_tree->number_of_cells, scy_tree->number_of_points);
            gpuErrchk(hipPeekAtLastError());

            compute_merge_map << < 1, number_of_dims >> >//todo fix num_of_dims>1024
                                      (d_is_s_connected, d_merge_map, scy_tree->number_of_cells);
            gpuErrchk(hipPeekAtLastError());
            hipMemcpy(h_merge_map, d_merge_map, number_of_restrictions * sizeof(int), hipMemcpyDeviceToHost);
//            printf("is_s_connected:\n");
//            print_array_gpu<< <1,1>>>(d_is_s_connected, number_of_restrictions);
//            hipDeviceSynchronize();
//            printf("h_merge_map:\n");
//            print_array(h_merge_map, number_of_restrictions);

            gpuErrchk(hipPeekAtLastError());

            restrict_merge_dim_multi2 << < number_of_dims, block >> >
                                                           (d_parents_full, scy_tree->d_parents, scy_tree->d_cells, scy_tree->d_counts, scy_tree->d_dim_start,
                                                                   d_is_included, d_new_counts, d_is_s_connected, d_dim_i, d_merge_map,
                                                                   scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                                   scy_tree->number_of_cells, scy_tree->number_of_points);
//            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());

            restrict_dim_prop_up_multi2 << < grid, block >> >
                                                   (d_parents_full, d_children_full, scy_tree->d_parents, scy_tree->d_counts, scy_tree->d_cells, scy_tree->d_dim_start,
                                                           d_is_included, d_new_counts, d_dim_i,
                                                           scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                           scy_tree->number_of_cells, scy_tree->number_of_points);

            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());

            restrict_merge_dim_prop_down_first_multi2 << < grid, block >> >
                                                                 (d_parents_full, d_children_full, scy_tree->d_parents, scy_tree->d_counts, scy_tree->d_cells, scy_tree->d_dim_start,
                                                                         d_is_included, d_new_counts, d_dim_i, d_merge_map,
                                                                         scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                                         scy_tree->number_of_cells, scy_tree->number_of_points);

            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());

            restrict_dim_prop_down_multi2 << < grid, block >> >
                                                     (d_parents_full, d_children_full, scy_tree->d_parents, scy_tree->d_counts, scy_tree->d_cells, scy_tree->d_dim_start,
                                                             d_is_included, d_new_counts, d_dim_i,
                                                             scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                             scy_tree->number_of_cells, scy_tree->number_of_points);


        } else {
            restrict_dim_once_and_for_all << < grid, block >> >
                                                     (scy_tree->d_parents, scy_tree->d_cells, scy_tree->d_counts, scy_tree->d_dim_start,
                                                             d_is_included, d_new_counts, d_is_s_connected, d_dim_i,
                                                             scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                             scy_tree->number_of_cells, scy_tree->number_of_points);
//            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());
        }
//        printf("test1\n");

//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

//        printf("test3\n");
        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;

                if (cell_no == 0 || h_merge_map[one_offset - 1] != h_merge_map[one_offset]) {
                    // 2. do a scan to find the new indecies for the nodes in the restricted tree
                    //todo should be done partial for each restriction - maybe this can be parallellized over blocks for each restriction
                    //todo make a inclusive_scan_multi
                    inclusive_scan(d_is_included + node_offset, d_new_indecies + node_offset,
                                   scy_tree->number_of_nodes);

                    // 3. construct restricted tree
//                hipDeviceSynchronize();
                    gpuErrchk(hipPeekAtLastError());

                    //todo find new_number_of_points and new_number_of_nodes for each restricted scy_tree
                    int *h_tmp = new int[1];
                    h_tmp[0] = 0;
                    hipMemcpy(h_tmp, d_new_counts + node_offset, sizeof(int), hipMemcpyDeviceToHost);
                    hipDeviceSynchronize();
                    gpuErrchk(hipPeekAtLastError());
                    int new_number_of_points = h_tmp[0];

                    hipMemcpy(h_tmp, d_new_indecies + node_offset + scy_tree->number_of_nodes - 1, sizeof(int),
                               hipMemcpyDeviceToHost);
                    hipDeviceSynchronize();
                    gpuErrchk(hipPeekAtLastError());
                    int new_number_of_nodes = h_tmp[0];

                    hipDeviceSynchronize();
                    gpuErrchk(hipPeekAtLastError());

                    //gpuErrchk(hipPeekAtLastError());
                    //todo create a new restricted scy_tree for each restriction

                    if (new_number_of_points > scy_tree->number_of_points || new_number_of_points < 0
                        || new_number_of_nodes > scy_tree->number_of_nodes || new_number_of_nodes < 0) {
                        printf("old_number_of_points: %d, new_number_of_points: %d\n", scy_tree->number_of_points,
                               new_number_of_points);
                        printf("old_number_of_nodes: %d, new_number_of_nodes: %d\n", scy_tree->number_of_nodes,
                               new_number_of_nodes);
                    }
                    ScyTreeArray *restricted_scy_tree = new ScyTreeArray(new_number_of_nodes,
                                                                         scy_tree->number_of_dims - 1,
                                                                         scy_tree->number_of_restricted_dims + 1,
                                                                         new_number_of_points,
                                                                         scy_tree->number_of_cells);
                    hipDeviceSynchronize();
                    gpuErrchk(hipPeekAtLastError());

                    L[i][cell_no] = restricted_scy_tree;
                    L_merged[i].push_back(restricted_scy_tree);

                    //todo set is s-connected for each restriction
                    restricted_scy_tree->cell_size = scy_tree->cell_size;//todo maybe not used
//                    hipMemcpy(h_tmp, d_is_s_connected + one_offset, sizeof(int), hipMemcpyDeviceToHost);
                    restricted_scy_tree->is_s_connected = false;//(bool) h_tmp[0];

                    hipDeviceSynchronize();
                    gpuErrchk(hipPeekAtLastError());
                    delete[] h_tmp;
                }
                cell_no++;
            }
            dim_no++;
        }

//        hipStream_t streams[number_of_restrictions];
//        for (int i = 0; i < min(number_of_restrictions, 10); i++) {
//            hipStreamCreate(&streams[i % 10]);
//        }

//        printf("test5\n");

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;
                if (cell_no == 0 || h_merge_map[one_offset - 1] != h_merge_map[one_offset]) {
                    ScyTreeArray *restricted_scy_tree = L[i][cell_no];


                    if (scy_tree->number_of_nodes == 0) {
                        printf("something strange happened!\n");
                    }

                    //todo parallellilize over restrictions
                    number_of_blocks = scy_tree->number_of_nodes / BLOCK_WIDTH;
                    if (scy_tree->number_of_nodes % BLOCK_WIDTH) number_of_blocks++;
                    restrict_move_multi2<<< number_of_blocks, BLOCK_WIDTH, 0 >>>//, streams[one_offset % 10]
                            (d_parents_full + node_offset, scy_tree->d_cells, restricted_scy_tree->d_cells,
                             scy_tree->d_parents, restricted_scy_tree->d_parents, d_new_counts + node_offset,
                             restricted_scy_tree->d_counts, d_new_indecies + node_offset,
                             d_is_included + node_offset, scy_tree->number_of_nodes);
                }
                cell_no++;
            }
            dim_no++;
        }
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;
                if (cell_no == 0 || h_merge_map[one_offset - 1] != h_merge_map[one_offset]) {
                    ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                    //todo this if statement would be the same for all restrictions because it is allways restricted on one more than scy_tree - which is really nice!
                    if (scy_tree->number_of_dims > 1) {//if not restricted on all dimensions

                        number_of_blocks = restricted_scy_tree->number_of_dims / BLOCK_WIDTH;
                        if (restricted_scy_tree->number_of_dims % BLOCK_WIDTH) number_of_blocks++;

                        //todo parallellilize over restrictions - maybe stride instead of distribute onto blocks - it would be easier to read and code
                        restrict_update_dim_3 << < number_of_blocks, BLOCK_WIDTH, 0 >> >
                                                                                  //, streams[one_offset % 10] >> >
                                                                                  (scy_tree->d_dim_start, scy_tree->d_dims,
                                                                                          restricted_scy_tree->d_dim_start,
                                                                                          restricted_scy_tree->d_dims,
                                                                                          d_new_indecies +
                                                                                          node_offset,
                                                                                          d_dim_i +
                                                                                          i, restricted_scy_tree->number_of_dims);

                    }
                }
                cell_no++;
            }
            dim_no++;
        }
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;
                if (cell_no == 0 || h_merge_map[one_offset - 1] != h_merge_map[one_offset]) {
                    ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                    //todo parallellilize over restrictions - maybe stride instead of distribute onto blocks - it would be easier to read and code
                    number_of_blocks = restricted_scy_tree->number_of_restricted_dims / BLOCK_WIDTH;
                    if (restricted_scy_tree->number_of_restricted_dims % BLOCK_WIDTH) number_of_blocks++;
                    restrict_update_restricted_dim << < number_of_blocks, BLOCK_WIDTH, 0 >> >
                                                                                       //, streams[one_offset % 10] >> >
                                                                                       (dim_no, scy_tree->d_restricted_dims, restricted_scy_tree->d_restricted_dims, scy_tree->number_of_restricted_dims);

                }
                cell_no++;
            }
            dim_no++;
        }
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;

                if (cell_no == 0 || h_merge_map[one_offset - 1] != h_merge_map[one_offset]) {
                    ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                    number_of_blocks = number_of_points / BLOCK_WIDTH;
                    if (number_of_points % BLOCK_WIDTH) number_of_blocks++;
                    restrict_merge_is_points_included_multi2
                    <<< number_of_blocks, BLOCK_WIDTH, 0 >>>//, streams[one_offset % 10] >> >
                            (d_parents_full + node_offset, scy_tree->d_points_placement, scy_tree->d_cells,
                             d_is_included + node_offset,
                             d_is_point_included + point_offset,
                             d_dim_i + i,
                             d_merge_map + i * number_of_cells,
                             scy_tree->number_of_dims, scy_tree->number_of_points, cell_no);

                }
                cell_no++;
            }
            dim_no++;
        }
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;

                if (cell_no == 0 || h_merge_map[one_offset - 1] != h_merge_map[one_offset]) {
                    ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                    //hipStreamSynchronize(streams[one_offset % 10]);
                    inclusive_scan(d_is_point_included + point_offset,
                                   d_point_new_indecies + point_offset,
                                   number_of_points);
                }
                cell_no++;
            }
            dim_no++;
        }
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

//        for (int i = 0; i < min(number_of_restrictions, 10); i++) {
//            hipStreamDestroy(streams[i]);
//        }

//        dim_no = first_dim_no;
//        while (dim_no < total_number_of_dim) {
//            int i = dim_no - first_dim_no;
//            int cell_no = 0;
//            while (cell_no < number_of_cells) {
//                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
//                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
//                int one_offset = i * number_of_cells + cell_no;
//
//                hipDeviceSynchronize();
//                gpuErrchk(hipPeekAtLastError());
//                printf("d_children_full:\n");
//                print_array_gpu<<<1, 1>>>(
//                        d_children_full + 2 * i * number_of_cells * number_of_cells * number_of_nodes +
//                        2 * cell_no * number_of_cells * number_of_nodes, 2 * scy_tree->number_of_nodes *
//                                                                         scy_tree->number_of_cells);
//                hipDeviceSynchronize();
//                printf("d_is_included:\n");
//                print_array_gpu<<<1, 1>>>(d_is_included + node_offset, scy_tree->number_of_nodes);
//                hipDeviceSynchronize();
//                printf("d_is_point_included:\n");
//                print_array_gpu<<<1, 1>>>(d_is_point_included + point_offset, scy_tree->number_of_points);
//                hipDeviceSynchronize();
//                printf("\nd_parents_full:\n");
//                print_array_gpu<<<1, 1>>>(d_parents_full + node_offset, scy_tree->number_of_nodes);
//                hipDeviceSynchronize();
//                gpuErrchk(hipPeekAtLastError());
//                printf("\nd_new_counts:\n");
//                print_array_gpu<<<1, 1>>>(d_new_counts + node_offset, scy_tree->number_of_nodes);
//                hipDeviceSynchronize();
//                gpuErrchk(hipPeekAtLastError());
//                printf("d_parents:\n");
//                print_array_gpu<<<1, 1>>>(scy_tree->d_parents, scy_tree->number_of_nodes);
//                hipDeviceSynchronize();
//                printf("\nd_counts:\n");
//                print_array_gpu<<<1, 1>>>(scy_tree->d_counts, scy_tree->number_of_nodes);
//                hipDeviceSynchronize();
//                printf("\nd_cells:\n");
//                print_array_gpu<<<1, 1>>>(scy_tree->d_cells, scy_tree->number_of_nodes);
//                hipDeviceSynchronize();
//                printf("\nd_merge_map:\n");
//                print_array_gpu<<<1, 1>>>(d_merge_map, number_of_restrictions);
//                hipDeviceSynchronize();
//                printf("\nd_is_s_connected:\n");
//                print_array_gpu<<<1, 1>>>(d_is_s_connected, number_of_restrictions);
//                hipDeviceSynchronize();
//                gpuErrchk(hipPeekAtLastError());
//
//                cell_no++;
//            }
//            dim_no++;
//        }

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;
                if (cell_no == 0 || h_merge_map[one_offset - 1] != h_merge_map[one_offset]) {
                    ScyTreeArray *restricted_scy_tree = L[i][cell_no];

//                    printf("number_of_nodes:%d, number_of_points:%d\n", restricted_scy_tree->number_of_nodes,
//                           restricted_scy_tree->number_of_points);
                    if (restricted_scy_tree->number_of_points > 0) {


                        //todo parallellilize over restrictions - maybe stride instead of distribute onto blocks - it would be easier to read and code
                        move_points_3_multi2 <<< number_of_blocks, BLOCK_WIDTH, 0 >>>//, streams[one_offset % 10] >> >
                                (d_parents_full + node_offset, d_children_full
                                                               + 2 * i * number_of_cells * number_of_cells *
                                                                 number_of_nodes
                                                               + 2 * cell_no * number_of_nodes * number_of_cells,
                                 scy_tree->d_parents, scy_tree->d_cells,
                                 scy_tree->d_points, scy_tree->d_points_placement, restricted_scy_tree->d_points,
                                 restricted_scy_tree->d_points_placement,
                                 d_point_new_indecies +
                                 point_offset,
                                 d_new_indecies + node_offset,
                                 d_is_point_included + point_offset,
                                 d_dim_i + i,
                                 number_of_points, scy_tree->number_of_dims, scy_tree->number_of_cells);
//                        printf("d:%d, c:%d\n", i, cell_no);
                        hipDeviceSynchronize();
                        gpuErrchk(hipPeekAtLastError());

//                        check_parents<<<1, 512>>>(restricted_scy_tree->d_points_placement,
//                                                  restricted_scy_tree->number_of_points, restricted_scy_tree->d_parents,
//                                                  restricted_scy_tree->number_of_nodes);
                    }
                }
                cell_no++;
            }
            dim_no++;
        }
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
    }

    delete[] h_new_number_of_points;
    delete[] h_new_number_of_nodes;
    delete[] h_merge_map;

//    printf("test10\n");
    return L_merged;
}


__global__
void remove_empty_nodes(int *d_new_parents_full, int *d_children_full,
                        int *d_parents, int *d_counts, int *d_cells,
                        int *d_dim_start,
                        int *d_is_included_full, int *d_new_counts_full, int *d_dim_i_full,
                        int number_of_dims, int number_of_nodes, int number_of_cells,
                        int number_of_points) {
    int i = blockIdx.x;
    int cell_no = blockIdx.y;

    int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;

    int *d_is_included = d_is_included_full + node_offset;
    int *d_new_counts = d_new_counts_full + node_offset;
    int *d_new_parents = d_new_parents_full + node_offset;
    int *d_children = d_children_full
                      + 2 * i * number_of_cells * number_of_cells * number_of_nodes
                      + 2 * cell_no * number_of_nodes * number_of_cells;

    for (int i = threadIdx.x; i < number_of_nodes; i += blockDim.x) {
        if (d_is_included[i] && i != 0) {
            int new_parent = d_new_parents[i];
            int count = d_new_counts[d_children[new_parent * number_of_cells * 2
                                                + 2 * d_cells[i]]];
            if (count == 0) {
                d_is_included[i] = 0;
            }
        }
    }

}

vector <vector<ScyTreeArray *>>
ScyTreeArray::restrict_merge_gpu_multi3(TmpMalloc *tmps, int first_dim_no, int number_of_dims,
                                        int number_of_cells) {//todo  number_of_dims is different from this->number_of_dims find a better name

    ScyTreeArray *scy_tree = this;

    tmps->reset_counters();


    int number_of_blocks;
    dim3 block(128);
    dim3 grid(number_of_dims, number_of_cells);

    int c = scy_tree->number_of_cells;
    int d = scy_tree->number_of_dims;

    int total_number_of_dim = first_dim_no + number_of_dims;
    int number_of_restrictions = number_of_dims * number_of_cells;

    vector <vector<ScyTreeArray *>> L(number_of_dims);

    vector <vector<ScyTreeArray *>> L_merged(number_of_dims);

    if (scy_tree->number_of_nodes * number_of_restrictions == 0)
        return L_merged;
    gpuErrchk(hipPeekAtLastError());

    //allocate tmp arrays - start
    int *d_new_indecies = tmps->get_int_array(tmps->int_array_counter++, scy_tree->number_of_nodes *
                                                                         number_of_restrictions);//tmps->d_new_indecies;
    gpuErrchk(hipPeekAtLastError());
    int *d_new_counts = tmps->get_int_array(tmps->int_array_counter++,
                                            scy_tree->number_of_nodes * number_of_restrictions);//tmps->d_new_counts;
    gpuErrchk(hipPeekAtLastError());
    int *d_is_included = tmps->get_int_array(tmps->int_array_counter++,
                                             scy_tree->number_of_nodes * number_of_restrictions);//tmps->d_is_included;
    gpuErrchk(hipPeekAtLastError());
    int *d_children_full = tmps->get_int_array(tmps->int_array_counter++,
                                               2 * scy_tree->number_of_nodes * number_of_restrictions *
                                               scy_tree->number_of_cells);
    gpuErrchk(hipPeekAtLastError());

    int *d_parents_full = tmps->get_int_array(tmps->int_array_counter++,
                                              scy_tree->number_of_nodes * number_of_restrictions);
    gpuErrchk(hipPeekAtLastError());

    hipMemset(d_new_indecies, 0, scy_tree->number_of_nodes * number_of_restrictions * sizeof(int));
    hipMemset(d_new_counts, 0, scy_tree->number_of_nodes * number_of_restrictions * sizeof(int));
    hipMemset(d_is_included, 0, scy_tree->number_of_nodes * number_of_restrictions * sizeof(int));
    hipMemset(d_parents_full, -1, scy_tree->number_of_nodes * number_of_restrictions * sizeof(int));
    hipMemset(d_children_full, -1,
               2 * scy_tree->number_of_nodes * number_of_restrictions * scy_tree->number_of_cells * sizeof(int));
    for (int i = 0; i < number_of_dims; i++) {
        for (int cell_no = 0; cell_no < number_of_cells; cell_no++) {
            int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
            memset << < 1, 1 >> > (d_is_included + node_offset, 0, 1);//todo not a good way to do this
        }
    }
    gpuErrchk(hipPeekAtLastError());

    int *d_is_point_included = tmps->get_int_array(tmps->int_array_counter++, this->number_of_points *
                                                                              number_of_restrictions);//tmps->d_is_point_included;
    gpuErrchk(hipPeekAtLastError());
    int *d_point_new_indecies = tmps->get_int_array(tmps->int_array_counter++, this->number_of_points *
                                                                               number_of_restrictions);//tmps->d_point_new_indecies;
    gpuErrchk(hipPeekAtLastError());

    hipMemset(d_is_point_included, 0, this->number_of_points * number_of_restrictions * sizeof(int));
    gpuErrchk(hipPeekAtLastError());

    int *d_is_s_connected = tmps->get_int_array(tmps->int_array_counter++,
                                                number_of_restrictions);//tmps->d_is_s_connected;
    hipMemset(d_is_s_connected, 0, number_of_restrictions * sizeof(int));
    gpuErrchk(hipPeekAtLastError());

    int *d_dim_i = tmps->get_int_array(tmps->int_array_counter++, number_of_dims);//tmps->d_dim_i;

    gpuErrchk(hipPeekAtLastError());

    int *h_new_number_of_points = new int[number_of_restrictions];
    int *h_new_number_of_nodes = new int[number_of_restrictions];

    int *d_merge_map = tmps->get_int_array(tmps->int_array_counter++, number_of_restrictions);
    int *h_merge_map = new int[number_of_restrictions];
    //allocate tmp arrays - end

    int dim_no = first_dim_no;
    while (dim_no < total_number_of_dim) {
        int i = dim_no - first_dim_no;
        L[i] = vector<ScyTreeArray *>(number_of_cells);

        //todo find each dim that are being restricted - same for all cells - dependent on the scy_tree and dim
        find_dim_i << < 1, 1 >> >
                           (d_dim_i + i, scy_tree->d_dims, dim_no, scy_tree->number_of_dims);
        dim_no++;
    }

    if (number_of_dims > 0) {

        if (true) {

            check_is_s_connected << < number_of_dims, block >> >
                                                      (scy_tree->d_parents, scy_tree->d_cells, scy_tree->d_counts, scy_tree->d_dim_start,
                                                              d_is_included, d_new_counts, d_is_s_connected, d_dim_i,
                                                              scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                              scy_tree->number_of_cells, scy_tree->number_of_points);
            gpuErrchk(hipPeekAtLastError());

            compute_merge_map << < 1, number_of_dims >> >//todo fix num_of_dims>1024
                                      (d_is_s_connected, d_merge_map, scy_tree->number_of_cells);
            gpuErrchk(hipPeekAtLastError());
            hipMemcpy(h_merge_map, d_merge_map, number_of_restrictions * sizeof(int), hipMemcpyDeviceToHost);

            gpuErrchk(hipPeekAtLastError());

            restrict_merge_dim_multi2 << < number_of_dims, block >> >
                                                           (d_parents_full, scy_tree->d_parents, scy_tree->d_cells, scy_tree->d_counts, scy_tree->d_dim_start,
                                                                   d_is_included, d_new_counts, d_is_s_connected, d_dim_i, d_merge_map,
                                                                   scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                                   scy_tree->number_of_cells, scy_tree->number_of_points);
//            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());

            restrict_dim_prop_up_multi2 << < grid, block >> >
                                                   (d_parents_full, d_children_full, scy_tree->d_parents, scy_tree->d_counts, scy_tree->d_cells, scy_tree->d_dim_start,
                                                           d_is_included, d_new_counts, d_dim_i,
                                                           scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                           scy_tree->number_of_cells, scy_tree->number_of_points);

            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());

            restrict_merge_dim_prop_down_first_multi2 << < grid, block >> >
                                                                 (d_parents_full, d_children_full, scy_tree->d_parents, scy_tree->d_counts, scy_tree->d_cells, scy_tree->d_dim_start,
                                                                         d_is_included, d_new_counts, d_dim_i, d_merge_map,
                                                                         scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                                         scy_tree->number_of_cells, scy_tree->number_of_points);

            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());

            restrict_dim_prop_down_multi2 << < grid, block >> >
                                                     (d_parents_full, d_children_full, scy_tree->d_parents, scy_tree->d_counts, scy_tree->d_cells, scy_tree->d_dim_start,
                                                             d_is_included, d_new_counts, d_dim_i,
                                                             scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                             scy_tree->number_of_cells, scy_tree->number_of_points);


//            hipDeviceSynchronize();
//            gpuErrchk(hipPeekAtLastError());
//
//            remove_empty_nodes << < grid, block >> >
//                                          (d_parents_full, d_children_full, scy_tree->d_parents, scy_tree->d_counts, scy_tree->d_cells, scy_tree->d_dim_start,
//                                                  d_is_included, d_new_counts, d_dim_i,
//                                                  scy_tree->number_of_dims, scy_tree->number_of_nodes,
//                                                  scy_tree->number_of_cells, scy_tree->number_of_points);


        } else {
            restrict_dim_once_and_for_all << < grid, block >> >
                                                     (scy_tree->d_parents, scy_tree->d_cells, scy_tree->d_counts, scy_tree->d_dim_start,
                                                             d_is_included, d_new_counts, d_is_s_connected, d_dim_i,
                                                             scy_tree->number_of_dims, scy_tree->number_of_nodes,
                                                             scy_tree->number_of_cells, scy_tree->number_of_points);
//            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());
        }

//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;

                if (cell_no == 0 || h_merge_map[one_offset - 1] != h_merge_map[one_offset]) {
                    // 2. do a scan to find the new indecies for the nodes in the restricted tree
                    //todo should be done partial for each restriction - maybe this can be parallellized over blocks for each restriction
                    //todo make a inclusive_scan_multi
                    inclusive_scan(d_is_included + node_offset, d_new_indecies + node_offset,
                                   scy_tree->number_of_nodes);

                    // 3. construct restricted tree
//                hipDeviceSynchronize();
                    gpuErrchk(hipPeekAtLastError());

                    //todo find new_number_of_points and new_number_of_nodes for each restricted scy_tree
                    int *h_tmp = new int[1];
                    h_tmp[0] = 0;
                    hipMemcpy(h_tmp, d_new_counts + node_offset, sizeof(int), hipMemcpyDeviceToHost);
                    hipDeviceSynchronize();
                    gpuErrchk(hipPeekAtLastError());
                    int new_number_of_points = h_tmp[0];

                    hipMemcpy(h_tmp, d_new_indecies + node_offset + scy_tree->number_of_nodes - 1, sizeof(int),
                               hipMemcpyDeviceToHost);
                    hipDeviceSynchronize();
                    gpuErrchk(hipPeekAtLastError());
                    int new_number_of_nodes = h_tmp[0];

                    hipDeviceSynchronize();
                    gpuErrchk(hipPeekAtLastError());

                    //gpuErrchk(hipPeekAtLastError());
                    //todo create a new restricted scy_tree for each restriction

                    if (new_number_of_points > scy_tree->number_of_points || new_number_of_points < 0
                        || new_number_of_nodes > scy_tree->number_of_nodes || new_number_of_nodes < 0) {
                        printf("old_number_of_points: %d, new_number_of_points: %d\n", scy_tree->number_of_points,
                               new_number_of_points);
                        printf("old_number_of_nodes: %d, new_number_of_nodes: %d\n", scy_tree->number_of_nodes,
                               new_number_of_nodes);
                    }
                    ScyTreeArray *restricted_scy_tree = new ScyTreeArray(new_number_of_nodes,
                                                                         scy_tree->number_of_dims - 1,
                                                                         scy_tree->number_of_restricted_dims + 1,
                                                                         new_number_of_points,
                                                                         scy_tree->number_of_cells);
                    hipDeviceSynchronize();
                    gpuErrchk(hipPeekAtLastError());

                    L[i][cell_no] = restricted_scy_tree;
                    L_merged[i].push_back(restricted_scy_tree);

                    //todo set is s-connected for each restriction
                    restricted_scy_tree->cell_size = scy_tree->cell_size;//todo maybe not used
                    restricted_scy_tree->is_s_connected = false;//(bool) h_tmp[0];

                    hipDeviceSynchronize();
                    gpuErrchk(hipPeekAtLastError());
                    delete[] h_tmp;
                }
                cell_no++;
            }
            dim_no++;
        }

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;
                if (cell_no == 0 || h_merge_map[one_offset - 1] != h_merge_map[one_offset]) {
                    ScyTreeArray *restricted_scy_tree = L[i][cell_no];


                    if (scy_tree->number_of_nodes == 0) {
                        printf("something strange happened!\n");
                    }

                    //todo parallellilize over restrictions
                    number_of_blocks = scy_tree->number_of_nodes / BLOCK_WIDTH;
                    if (scy_tree->number_of_nodes % BLOCK_WIDTH) number_of_blocks++;
                    restrict_move_multi2<<< number_of_blocks, BLOCK_WIDTH, 0 >>>//, streams[one_offset % 10]
                            (d_parents_full + node_offset, scy_tree->d_cells, restricted_scy_tree->d_cells,
                             scy_tree->d_parents, restricted_scy_tree->d_parents, d_new_counts + node_offset,
                             restricted_scy_tree->d_counts, d_new_indecies + node_offset,
                             d_is_included + node_offset, scy_tree->number_of_nodes);
                }
                cell_no++;
            }
            dim_no++;
        }
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;
                if (cell_no == 0 || h_merge_map[one_offset - 1] != h_merge_map[one_offset]) {
                    ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                    //todo this if statement would be the same for all restrictions because it is allways restricted on one more than scy_tree - which is really nice!
                    if (scy_tree->number_of_dims > 1) {//if not restricted on all dimensions

                        number_of_blocks = restricted_scy_tree->number_of_dims / BLOCK_WIDTH;
                        if (restricted_scy_tree->number_of_dims % BLOCK_WIDTH) number_of_blocks++;

                        //todo parallellilize over restrictions - maybe stride instead of distribute onto blocks - it would be easier to read and code
                        restrict_update_dim_3 << < number_of_blocks, BLOCK_WIDTH, 0 >> >
                                                                                  //, streams[one_offset % 10] >> >
                                                                                  (scy_tree->d_dim_start, scy_tree->d_dims,
                                                                                          restricted_scy_tree->d_dim_start,
                                                                                          restricted_scy_tree->d_dims,
                                                                                          d_new_indecies +
                                                                                          node_offset,
                                                                                          d_dim_i +
                                                                                          i, restricted_scy_tree->number_of_dims);

                    }
                }
                cell_no++;
            }
            dim_no++;
        }
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;
                if (cell_no == 0 || h_merge_map[one_offset - 1] != h_merge_map[one_offset]) {
                    ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                    //todo parallellilize over restrictions - maybe stride instead of distribute onto blocks - it would be easier to read and code
                    number_of_blocks = restricted_scy_tree->number_of_restricted_dims / BLOCK_WIDTH;
                    if (restricted_scy_tree->number_of_restricted_dims % BLOCK_WIDTH) number_of_blocks++;
                    restrict_update_restricted_dim << < number_of_blocks, BLOCK_WIDTH, 0 >> >
                                                                                       //, streams[one_offset % 10] >> >
                                                                                       (dim_no, scy_tree->d_restricted_dims, restricted_scy_tree->d_restricted_dims, scy_tree->number_of_restricted_dims);

                }
                cell_no++;
            }
            dim_no++;
        }
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;

                if (cell_no == 0 || h_merge_map[one_offset - 1] != h_merge_map[one_offset]) {
                    ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                    number_of_blocks = number_of_points / BLOCK_WIDTH;
                    if (number_of_points % BLOCK_WIDTH) number_of_blocks++;
                    restrict_merge_is_points_included_multi2
                    <<< number_of_blocks, BLOCK_WIDTH, 0 >>>//, streams[one_offset % 10] >> >
                            (d_parents_full + node_offset, scy_tree->d_points_placement, scy_tree->d_cells,
                             d_is_included + node_offset,
                             d_is_point_included + point_offset,
                             d_dim_i + i,
                             d_merge_map + i * number_of_cells,
                             scy_tree->number_of_dims, scy_tree->number_of_points, cell_no);

                }
                cell_no++;
            }
            dim_no++;
        }
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;

                if (cell_no == 0 || h_merge_map[one_offset - 1] != h_merge_map[one_offset]) {
                    ScyTreeArray *restricted_scy_tree = L[i][cell_no];

                    //hipStreamSynchronize(streams[one_offset % 10]);
                    inclusive_scan(d_is_point_included + point_offset,
                                   d_point_new_indecies + point_offset,
                                   number_of_points);
                }
                cell_no++;
            }
            dim_no++;
        }
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        dim_no = first_dim_no;
        while (dim_no < total_number_of_dim) {
            int i = dim_no - first_dim_no;
            int cell_no = 0;
            while (cell_no < number_of_cells) {
                int point_offset = i * number_of_cells * number_of_points + cell_no * number_of_points;
                int node_offset = i * number_of_cells * number_of_nodes + cell_no * number_of_nodes;
                int one_offset = i * number_of_cells + cell_no;
                if (cell_no == 0 || h_merge_map[one_offset - 1] != h_merge_map[one_offset]) {
                    ScyTreeArray *restricted_scy_tree = L[i][cell_no];

//                    printf("number_of_nodes:%d, number_of_points:%d\n", restricted_scy_tree->number_of_nodes,
//                           restricted_scy_tree->number_of_points);
                    if (restricted_scy_tree->number_of_points > 0) {


                        //todo parallellilize over restrictions - maybe stride instead of distribute onto blocks - it would be easier to read and code
                        move_points_3_multi2 <<< number_of_blocks, BLOCK_WIDTH, 0 >>>//, streams[one_offset % 10] >> >
                                (d_parents_full + node_offset, d_children_full
                                                               + 2 * i * number_of_cells * number_of_cells *
                                                                 number_of_nodes
                                                               + 2 * cell_no * number_of_nodes * number_of_cells,
                                 scy_tree->d_parents, scy_tree->d_cells,
                                 scy_tree->d_points, scy_tree->d_points_placement, restricted_scy_tree->d_points,
                                 restricted_scy_tree->d_points_placement,
                                 d_point_new_indecies +
                                 point_offset,
                                 d_new_indecies + node_offset,
                                 d_is_point_included + point_offset,
                                 d_dim_i + i,
                                 number_of_points, scy_tree->number_of_dims, scy_tree->number_of_cells);
//                        printf("d:%d, c:%d\n", i, cell_no);
                        hipDeviceSynchronize();
                        gpuErrchk(hipPeekAtLastError());

//                        check_parents<<<1, 512>>>(restricted_scy_tree->d_points_placement,
//                                                  restricted_scy_tree->number_of_points, restricted_scy_tree->d_parents,
//                                                  restricted_scy_tree->number_of_nodes);
                    }
                }
                cell_no++;
            }
            dim_no++;
        }
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
    }

    delete[] h_new_number_of_points;
    delete[] h_new_number_of_nodes;
    delete[] h_merge_map;

//    printf("test10\n");
    return L_merged;
}

ScyTreeArray *ScyTreeArray::mergeWithNeighbors_gpu1(ScyTreeArray *parent_scy_tree, int dim_no, int &cell_no) {
    if (!this->is_s_connected) {
        return this;
    }

    ScyTreeArray *merged_scy_tree = this;
    ScyTreeArray *restricted_scy_tree = this;
    while (restricted_scy_tree->is_s_connected && cell_no < this->number_of_cells - 1) {
        cell_no++;
        gpuErrchk(hipPeekAtLastError());
        restricted_scy_tree = parent_scy_tree->restrict_gpu(dim_no, cell_no);
        gpuErrchk(hipPeekAtLastError());
        if (restricted_scy_tree->number_of_points > 0) {
            ScyTreeArray *merged_scy_tree_old = merged_scy_tree;
            gpuErrchk(hipPeekAtLastError());
            merged_scy_tree = merged_scy_tree->merge(restricted_scy_tree);
            gpuErrchk(hipPeekAtLastError());
            hipDeviceSynchronize();
            if (merged_scy_tree_old != this)
                delete merged_scy_tree_old;
        }
    }

    merged_scy_tree->is_s_connected = false;
    return merged_scy_tree;
}

ScyTreeArray *ScyTreeArray::merge(ScyTreeArray *sibling_scy_tree) {
    int *d_parents_3, *d_cells_3, *d_counts_3, *d_dim_start_3, *d_dims_3, *d_restricted_dims_3, *d_points_3, *d_points_placement_3;
    int n_3, d_3, number_of_points_3, number_of_restricted_dims_3;

    gpuErrchk(hipPeekAtLastError());
    merge_using_gpu(this->d_parents, this->d_cells, this->d_counts,
                    this->d_dim_start, this->d_dims, this->d_restricted_dims,
                    this->d_points, this->d_points_placement,
                    this->number_of_dims, this->number_of_nodes, this->number_of_points,
                    this->number_of_restricted_dims,
                    sibling_scy_tree->d_parents, sibling_scy_tree->d_cells, sibling_scy_tree->d_counts,
                    sibling_scy_tree->d_dim_start, sibling_scy_tree->d_dims, sibling_scy_tree->d_restricted_dims,
                    sibling_scy_tree->d_points, sibling_scy_tree->d_points_placement,
                    sibling_scy_tree->number_of_dims, sibling_scy_tree->number_of_nodes,
                    sibling_scy_tree->number_of_restricted_dims,
                    sibling_scy_tree->number_of_points,
                    d_parents_3, d_cells_3, d_counts_3,
                    d_dim_start_3, d_dims_3, d_restricted_dims_3,
                    d_points_3, d_points_placement_3,
                    d_3, n_3, number_of_points_3, number_of_restricted_dims_3);

    gpuErrchk(hipPeekAtLastError());
    ScyTreeArray *merged_scy_tree = new ScyTreeArray(n_3, this->number_of_dims, this->number_of_restricted_dims,
                                                     number_of_points_3, this->number_of_cells,
                                                     d_cells_3, d_parents_3, d_counts_3,
                                                     d_dim_start_3, d_dims_3, d_restricted_dims_3,
                                                     d_points_3, d_points_placement_3);


    gpuErrchk(hipPeekAtLastError());

    return merged_scy_tree;
}


void merge_using_gpu(TmpMalloc *tmps, int *d_parents_1, int *d_cells_1, int *d_counts_1,
                     int *d_dim_start_1, int *d_dims_1, int *d_restricted_dims_1,
                     int *d_points_1, int *d_points_placement_1,
                     int d_1, int n_1, int number_of_points_1, int number_of_restricted_dims_1,
                     int *d_parents_2, int *d_cells_2, int *d_counts_2,
                     int *d_dim_start_2, int *d_dims_2, int *d_restricted_dims_2,
                     int *d_points_2, int *d_points_placement_2,
                     int d_2, int n_2, int number_of_points_2, int number_of_restricted_dims_2,
                     int *&d_parents_3, int *&d_cells_3, int *&d_counts_3,
                     int *&d_dim_start_3, int *&d_dims_3, int *&d_restricted_dims_3,
                     int *&d_points_3, int *&d_points_placement_3,
                     int &d_3, int &n_3, int &number_of_points_3, int &number_of_restricted_dims_3) {


    gpuErrchk(hipPeekAtLastError());

    tmps->reset_counters();

    //compute sort keys for both using cell id cell_no and concat
    //sort - save permutation
    int n_total = n_1 + n_2;

    int numBlocks;

    int *d_map_to_old = tmps->get_int_array(tmps->int_array_counter++, n_total);//tmps->d_map_to_old;
    int *d_map_to_new = tmps->get_int_array(tmps->int_array_counter++, n_total);//tmps->d_map_to_new;
    int *d_is_included = tmps->get_int_array(tmps->int_array_counter++, n_total);//tmps->d_is_included_merge;
    int *d_new_indecies = tmps->get_int_array(tmps->int_array_counter++, n_total);//tmps->d_new_indecies_merge;
//    hipMalloc(&d_map_to_new, n_total * sizeof(int));
    hipMemset(d_map_to_new, -99, n_total * sizeof(int));
    gpuErrchk(hipPeekAtLastError());
//    hipDeviceSynchronize();
    memset << < 1, 1 >> > (d_map_to_new, 0, 1);//q
    memset << < 1, 1 >> > (d_map_to_new, 0 + n_1, 0);//q
    gpuErrchk(hipPeekAtLastError());

//    hipMalloc(&d_map_to_old, n_total * sizeof(int));
    hipMemset(d_map_to_old, -88, n_total * sizeof(int));
//    hipDeviceSynchronize();
    memset << < 1, 1 >> > (d_map_to_old, 1, 0);//q
    memset << < 1, 1 >> > (d_map_to_old, 0, 0 + n_1);//q
    gpuErrchk(hipPeekAtLastError());

//    hipMalloc(&d_is_included, n_total * sizeof(int));
    hipMemset(d_is_included, -77, n_total * sizeof(int));
    memset << < 1, 1 >> > (d_is_included, 0, 1);//root should always be included
    memset << < 1, 1 >> > (d_is_included, 1, 0);//q
    gpuErrchk(hipPeekAtLastError());

//    hipMalloc(&d_new_indecies, n_total * sizeof(int));
    hipMemset(d_new_indecies, 0, n_total * sizeof(int));
//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());
    memset << < 1, 1 >> > (d_new_indecies, 0, 1);//q
    memset << < 1, 1 >> > (d_new_indecies, 1, 1);//q

//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());


    int *h_dim_start_1 = new int[d_1];
    int *h_dim_start_2 = new int[d_2];
//    printf("d_1:%d, d_2:%d\n", d_1, d_2);
    gpuErrchk(hipPeekAtLastError());
    hipMemcpy(h_dim_start_1, d_dim_start_1, sizeof(int) * d_1, hipMemcpyDeviceToHost);
    gpuErrchk(hipPeekAtLastError());
    hipMemcpy(h_dim_start_2, d_dim_start_2, sizeof(int) * d_2, hipMemcpyDeviceToHost);
    gpuErrchk(hipPeekAtLastError());
    int step = 4; //todo find better

    int n_pivots = (n_total / step + (n_total % step ? 1 : 0));
    int *pivots_1 = tmps->get_int_array(tmps->int_array_counter++, n_pivots);//tmps->pivots_1;
    int *pivots_2 = tmps->get_int_array(tmps->int_array_counter++, n_pivots);//tmps->pivots_2;
//    hipMalloc(&pivots_1, n_pivots * sizeof(int));
//    hipMalloc(&pivots_2, n_pivots * sizeof(int));

    gpuErrchk(hipPeekAtLastError());

    for (int d_i = -1; d_i < d_1; d_i++) {//todo root always has the same result, so it can be avoided
        hipMemset(pivots_1, -1, n_pivots * sizeof(int));
        hipMemset(pivots_2, -1, n_pivots * sizeof(int));
//        printf("d_i:%d\n", d_i);
        int start_1 = d_i == -1 ? 0 : h_dim_start_1[d_i];
        int start_2 = d_i == -1 ? 0 : h_dim_start_2[d_i];
        int end_1 = d_i == -1 ? 1 : (d_i + 1 < d_1 ? h_dim_start_1[d_i + 1] : n_1);
        int end_2 = d_i == -1 ? 1 : (d_i + 1 < d_1 ? h_dim_start_2[d_i + 1] : n_2);
        int start_toal = start_1 + start_2;
        int end_total = end_1 + end_2;
        int length = end_total - start_toal;

        numBlocks = length / (BLOCK_WIDTH * step);
        if (length % (BLOCK_WIDTH * step)) numBlocks++;

//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());


        merge_search_for_pivots << < numBlocks, BLOCK_WIDTH >> >
                                                (start_1, start_2, end_1, end_2, pivots_1, pivots_2, n_1, n_2, n_total, step,
                                                        cmp(d_new_indecies, d_map_to_new,
                                                            d_parents_1, d_parents_2,
                                                            d_cells_1, d_cells_2,
                                                            d_counts_1, d_counts_2,
                                                            n_1, n_2));
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());


        merge_check_path_from_pivots << < 1, 1 >> >
                                             (start_1, start_2, end_1, end_2, d_map_to_old, d_map_to_new, pivots_1, pivots_2, n_1, n_2, n_total, step,
                                                     cmp(d_new_indecies, d_map_to_new, d_parents_1,
                                                         d_parents_2,
                                                         d_cells_1, d_cells_2, d_counts_1, d_counts_2,
                                                         n_1, n_2));
//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());


        numBlocks = length / BLOCK_WIDTH;
        if (length % BLOCK_WIDTH) numBlocks++;
        compute_is_included_from_path << < numBlocks, BLOCK_WIDTH >> >
                                                      (start_1, start_2, d_is_included, d_map_to_old, d_parents_1, d_parents_2, d_cells_1, d_cells_2, d_counts_1, d_counts_2, n_1, end_total);

//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        hipMemset(d_new_indecies, 0, n_total * sizeof(int));

//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        inclusive_scan(d_is_included, d_new_indecies, n_total);

//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
    }

//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

//    hipDeviceSynchronize();

    int *h_tmp = new int[1];
    hipMemcpy(h_tmp, d_new_indecies + n_total - 1, sizeof(int), hipMemcpyDeviceToHost);
    n_3 = h_tmp[0];


    d_3 = d_1;
    number_of_restricted_dims_3 = number_of_restricted_dims_1;


    //update parent id, cells and count

    gpuErrchk(hipPeekAtLastError());
    hipMalloc(&d_parents_3, n_3 * sizeof(int));
    gpuErrchk(hipPeekAtLastError());
    hipMalloc(&d_cells_3, n_3 * sizeof(int));
    gpuErrchk(hipPeekAtLastError());
    hipMalloc(&d_counts_3, n_3 * sizeof(int));
    gpuErrchk(hipPeekAtLastError());
    hipMemset(d_counts_3, 0, n_3 * sizeof(int));
    gpuErrchk(hipPeekAtLastError());
    hipMalloc(&d_dim_start_3, d_3 * sizeof(int));
    gpuErrchk(hipPeekAtLastError());
    hipMalloc(&d_dims_3, d_3 * sizeof(int));
    gpuErrchk(hipPeekAtLastError());
    hipMalloc(&d_restricted_dims_3, number_of_restricted_dims_3 * sizeof(int));

//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());


    numBlocks = n_total / BLOCK_WIDTH;
    if (n_total % BLOCK_WIDTH) numBlocks++;
    merge_move << < numBlocks, BLOCK_WIDTH >> >
                               (d_cells_1, d_cells_2, d_cells_3,
                                       d_parents_1, d_parents_2, d_parents_3,
                                       d_counts_1, d_counts_2, d_counts_3,
                                       d_new_indecies, d_map_to_new, d_map_to_old, n_total, n_1);


//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    clone << < 1, BLOCK_WIDTH >> > (d_restricted_dims_3, d_restricted_dims_1, number_of_restricted_dims_3);

    if (d_3 > 0) {
        numBlocks = d_3 / BLOCK_WIDTH;
        if (d_3 % BLOCK_WIDTH) numBlocks++;
        merge_update_dim << < numBlocks, BLOCK_WIDTH >> >
                                         (d_dim_start_1, d_dims_1, d_dim_start_2, d_dims_2, d_dim_start_3, d_dims_3, d_new_indecies, d_map_to_new, d_3, n_1);


//        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
    }
//    hipDeviceSynchronize();
    //get number of points
    hipMemcpy(h_tmp, d_counts_3, sizeof(int), hipMemcpyDeviceToHost);
    number_of_points_3 = h_tmp[0];


//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    //construct new point arrays
    hipMalloc(&d_points_3, number_of_points_3 * sizeof(int));
    hipMemset(d_points_3, 0, number_of_points_3 * sizeof(int));
    hipMalloc(&d_points_placement_3, number_of_points_3 * sizeof(int));
    hipMemset(d_points_placement_3, 0, number_of_points_3 * sizeof(int));


//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    // for each tree move points to new arrays
    numBlocks = number_of_points_3 / BLOCK_WIDTH;
    if (number_of_points_3 % BLOCK_WIDTH) numBlocks++;
    points_move << < numBlocks, BLOCK_WIDTH >> > (d_points_1, d_points_placement_1, number_of_points_1, n_1,
            d_points_2, d_points_placement_2, number_of_points_2,
            d_points_3, d_points_placement_3, number_of_points_3,
            d_new_indecies, d_map_to_new);


//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());


//    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());
}

ScyTreeArray *ScyTreeArray::merge(TmpMalloc *tmps, ScyTreeArray *sibling_scy_tree) {
    int *d_parents_3, *d_cells_3, *d_counts_3, *d_dim_start_3, *d_dims_3, *d_restricted_dims_3, *d_points_3, *d_points_placement_3;
    int n_3, d_3, number_of_points_3, number_of_restricted_dims_3;

    gpuErrchk(hipPeekAtLastError());
    merge_using_gpu(tmps, this->d_parents, this->d_cells, this->d_counts,
                    this->d_dim_start, this->d_dims, this->d_restricted_dims,
                    this->d_points, this->d_points_placement,
                    this->number_of_dims, this->number_of_nodes, this->number_of_points,
                    this->number_of_restricted_dims,
                    sibling_scy_tree->d_parents, sibling_scy_tree->d_cells, sibling_scy_tree->d_counts,
                    sibling_scy_tree->d_dim_start, sibling_scy_tree->d_dims, sibling_scy_tree->d_restricted_dims,
                    sibling_scy_tree->d_points, sibling_scy_tree->d_points_placement,
                    sibling_scy_tree->number_of_dims, sibling_scy_tree->number_of_nodes,
                    sibling_scy_tree->number_of_restricted_dims,
                    sibling_scy_tree->number_of_points,
                    d_parents_3, d_cells_3, d_counts_3,
                    d_dim_start_3, d_dims_3, d_restricted_dims_3,
                    d_points_3, d_points_placement_3,
                    d_3, n_3, number_of_points_3, number_of_restricted_dims_3);

//    printf("after merge_using_gpu\n");

    gpuErrchk(hipPeekAtLastError());
    ScyTreeArray *merged_scy_tree = new ScyTreeArray(n_3, this->number_of_dims, this->number_of_restricted_dims,
                                                     number_of_points_3, this->number_of_cells,
                                                     d_cells_3, d_parents_3, d_counts_3,
                                                     d_dim_start_3, d_dims_3, d_restricted_dims_3,
                                                     d_points_3, d_points_placement_3);


    gpuErrchk(hipPeekAtLastError());

    return merged_scy_tree;
}

int ScyTreeArray::get_dims_idx() {
    int sum = 0;

    hipMemcpy(this->h_restricted_dims, this->d_restricted_dims, sizeof(int) * number_of_restricted_dims,
               hipMemcpyDeviceToHost);
    for (int i = 0; i < this->number_of_restricted_dims; i++) {
        int re_dim = this->h_restricted_dims[i];
        sum += 1 << re_dim;
    }
    return sum;
}

ScyTreeArray::ScyTreeArray(int number_of_nodes, int number_of_dims, int number_of_restricted_dims, int number_of_points,
                           int number_of_cells) {
    this->number_of_nodes = number_of_nodes;
    this->number_of_dims = number_of_dims;
    this->number_of_restricted_dims = number_of_restricted_dims;
    this->number_of_points = number_of_points;
    this->number_of_cells = number_of_cells;
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

//    printf("\nScyTreeArray - small - number_of_nodes:%d, number_of_dims:%d, number_of_restricted_dims:%d, number_of_points:%d, number_of_cells:%d\n", number_of_nodes, number_of_dims, number_of_restricted_dims, number_of_points,
//           number_of_cells);

    this->h_parents = new int[number_of_nodes];
    zero(this->h_parents, number_of_nodes);

    this->h_cells = new int[number_of_nodes];
    zero(this->h_cells, number_of_nodes);

    this->h_counts = new int[number_of_nodes];
    zero(this->h_counts, number_of_nodes);

    this->h_dim_start = new int[number_of_dims];
    zero(this->h_dim_start, number_of_dims);

    this->h_dims = new int[number_of_dims];
    zero(this->h_dims, number_of_dims);

    this->h_points = new int[number_of_points];
    zero(this->h_points, number_of_points);

    this->h_points_placement = new int[number_of_points];
    zero(this->h_points_placement, number_of_points);

    this->h_restricted_dims = new int[number_of_restricted_dims];
    zero(this->h_restricted_dims, number_of_restricted_dims);

    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    if (number_of_nodes > 0) {
        hipMalloc(&this->d_parents, number_of_nodes * sizeof(int));
        hipMemset(this->d_parents, 0, number_of_nodes * sizeof(int));

        hipMalloc(&this->d_cells, number_of_nodes * sizeof(int));
        hipMemset(this->d_cells, 0, number_of_nodes * sizeof(int));

        hipMalloc(&this->d_counts, number_of_nodes * sizeof(int));
        hipMemset(this->d_counts, 0, number_of_nodes * sizeof(int));

        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
    }

    if (number_of_dims > 0) {
        hipMalloc(&this->d_dim_start, number_of_dims * sizeof(int));
        hipMemset(this->d_dim_start, 0, number_of_dims * sizeof(int));

        hipMalloc(&this->d_dims, number_of_dims * sizeof(int));
        hipMemset(this->d_dims, 0, number_of_dims * sizeof(int));

        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
    }

    if (number_of_restricted_dims > 0) {
        hipMalloc(&this->d_restricted_dims, number_of_restricted_dims * sizeof(int));
        hipMemset(this->d_restricted_dims, 0, number_of_restricted_dims * sizeof(int));

        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
    }

    if (number_of_points > 0) {
        hipMalloc(&this->d_points, number_of_points * sizeof(int));
        gpuErrchk(hipPeekAtLastError());
        hipMemset(this->d_points, 0, number_of_points * sizeof(int));
        gpuErrchk(hipPeekAtLastError());

        hipMalloc(&this->d_points_placement, number_of_points * sizeof(int));
        gpuErrchk(hipPeekAtLastError());
        hipMemset(this->d_points_placement, 0, number_of_points * sizeof(int));

        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
    }
}

ScyTreeArray::ScyTreeArray(int
                           number_of_nodes, int
                           number_of_dims, int
                           number_of_restricted_dims, int
                           number_of_points,
                           int
                           number_of_cells, int *d_cells, int *d_parents, int *d_counts, int *d_dim_start,
                           int *d_dims, int *d_restricted_dims, int *d_points, int *d_points_placement) {

//    printf("\nScyTreeArray - large - number_of_nodes:%d, number_of_dims:%d, number_of_restricted_dims:%d, number_of_points:%d, number_of_cells:%d\n", number_of_nodes, number_of_dims, number_of_restricted_dims, number_of_points,
//           number_of_cells);

    this->number_of_nodes = number_of_nodes;
    this->number_of_dims = number_of_dims;
    this->number_of_restricted_dims = number_of_restricted_dims;
    this->number_of_points = number_of_points;
    this->number_of_cells = number_of_cells;

    this->h_parents = new int[number_of_nodes];
    zero(this->h_parents, number_of_nodes);

    this->h_cells = new int[number_of_nodes];
    zero(this->h_cells, number_of_nodes);

    this->h_counts = new int[number_of_nodes];
    zero(this->h_counts, number_of_nodes);

    this->h_dim_start = new int[number_of_dims];
    zero(this->h_dim_start, number_of_dims);

    this->h_dims = new int[number_of_dims];
    zero(this->h_dims, number_of_dims);

    this->h_points = new int[number_of_points];
    zero(this->h_points, number_of_points);

    this->h_points_placement = new int[number_of_points];
    zero(this->h_points_placement, number_of_points);

    this->h_restricted_dims = new int[number_of_restricted_dims];
    zero(this->h_restricted_dims, number_of_restricted_dims);


    this->d_parents = d_parents;

    this->d_cells = d_cells;

    this->d_counts = d_counts;

    this->d_dim_start = d_dim_start;

    this->d_dims = d_dims;

    this->d_restricted_dims = d_restricted_dims;

    this->d_points = d_points;

    this->d_points_placement = d_points_placement;
}

void ScyTreeArray::copy_to_host() {
    hipMemcpy(h_parents, d_parents, sizeof(int) * this->number_of_nodes, hipMemcpyDeviceToHost);
    hipMemcpy(h_cells, d_cells, sizeof(int) * this->number_of_nodes, hipMemcpyDeviceToHost);
    hipMemcpy(h_counts, d_counts, sizeof(int) * this->number_of_nodes, hipMemcpyDeviceToHost);
    hipMemcpy(h_dim_start, d_dim_start, sizeof(int) * this->number_of_dims, hipMemcpyDeviceToHost);
    hipMemcpy(h_dims, d_dims, sizeof(int) * this->number_of_dims, hipMemcpyDeviceToHost);
    hipMemcpy(h_points, d_points, sizeof(int) * this->number_of_points, hipMemcpyDeviceToHost);
    hipMemcpy(h_points_placement, d_points_placement, sizeof(int) * this->number_of_points,
               hipMemcpyDeviceToHost);
    hipMemcpy(h_restricted_dims, d_restricted_dims, sizeof(int) * this->number_of_restricted_dims,
               hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
}

void ScyTreeArray::copy_to_device() {
    hipMemcpy(d_parents, h_parents, sizeof(int) * this->number_of_nodes, hipMemcpyHostToDevice);
    hipMemcpy(d_cells, h_cells, sizeof(int) * this->number_of_nodes, hipMemcpyHostToDevice);
    hipMemcpy(d_counts, h_counts, sizeof(int) * this->number_of_nodes, hipMemcpyHostToDevice);
    hipMemcpy(d_dim_start, h_dim_start, sizeof(int) * this->number_of_dims, hipMemcpyHostToDevice);
    hipMemcpy(d_dims, h_dims, sizeof(int) * this->number_of_dims, hipMemcpyHostToDevice);
    hipMemcpy(d_points, h_points, sizeof(int) * this->number_of_points, hipMemcpyHostToDevice);
    hipMemcpy(d_points_placement, h_points_placement, sizeof(int) * this->number_of_points,
               hipMemcpyHostToDevice);
    hipMemcpy(d_restricted_dims, h_restricted_dims, sizeof(int) * this->number_of_restricted_dims,
               hipMemcpyHostToDevice);
    hipDeviceSynchronize();
}

void ScyTreeArray::print() {
    printf("\nnumber_of_nodes: %d, number_of_points: %d, number_of_dims: %d, number_of_restricted_dims: %d, number_of_cells: %d\n",
           this->number_of_nodes, this->number_of_points, this->number_of_dims, this->number_of_restricted_dims,
           this->number_of_cells);
    printf("d_parents:\n");
    print_array_gpu<<<1, 1>>>(this->d_parents, this->number_of_nodes);
    hipDeviceSynchronize();
    printf("h_parents:\n");
    print_array(this->h_parents, this->number_of_nodes);
    printf("h_cells:\n");
    print_array(this->h_cells, this->number_of_nodes);
    printf("h_counts:\n");
    print_array(this->h_counts, this->number_of_nodes);
    printf("h_dim_start:\n");
    print_array(this->h_dim_start, this->number_of_dims);
    printf("h_dims:\n");
    print_array(this->h_dims, this->number_of_dims);
    print_scy_tree(this->h_parents, this->h_cells, this->h_counts, this->h_dim_start, this->h_dims,
                   this->number_of_dims, this->number_of_nodes);

    printf("\n");
}


__device__
float dist_prune_gpu(int p_id, int q_id, float *X, int d, int *subspace, int subsapce_size) {
    float *p = &X[p_id * d];
    float *q = &X[q_id * d];
    float distance = 0;
    for (int i = 0; i < subsapce_size; i++) {
        int d_i = subspace[i];
        float diff = p[d_i] - q[d_i];
        distance += diff * diff;
    }
    return sqrt(distance);//todo squared can be removed by sqrt(x)<=y => x<=y*y if x>=0, y>=0
}


//__device__
//float phi_prune_gpu(int p_id, int *d_neighborhood, float neighborhood_size, int number_of_neighbors,
//                    float *X, int *d_points, int d) {
//    float sum = 0;
//    for (int j = 0; j < number_of_neighbors; j++) {
//        int q_id = d_neighborhood[j];//d_points[d_neighborhood[j]];
//        if (q_id >= 0) {
//            float distance = dist_prune_gpu(p_id, q_id, X, d) / neighborhood_size;
//            float sq = distance * distance;
//            sum += (1. - sq);
//        }
//    }
//    return sum;
//}

__device__
double gamma_prune_gpu(int n) {
    if (n == 2) {
        return 1.;
    } else if (n == 1) {
        return sqrt(PI);
    }
    return (n / 2. - 1.) * gamma_prune_gpu(n - 2);
}

__device__
float c_prune_gpu(int subspace_size) {
    float r = pow(PI, subspace_size / 2.);
    //r = r / gamma_gpu(subspace_size / 2. + 1.);
    r = r / gamma_prune_gpu(subspace_size + 2);
    return r;
}

__device__
float alpha_prune_gpu(int subspace_size, float neighborhood_size, int n) {
    float v = 1.;//todo v is missing?? what is it??
    float r = 2 * n * pow(neighborhood_size, subspace_size) * c_prune_gpu(subspace_size);
    r = r / (pow(v, subspace_size) * (subspace_size + 2));
    return r;
}

__device__
float omega_prune_gpu(int subspace_size) {
    return 2.0 / (subspace_size + 2.0);
}

//__global__
//void
//find_neighborhood_prune(int *d_neighborhoods, int *d_number_of_neighbors, float *X,
//                        int *d_points, int number_of_points, float neighborhood_size,
//                        int *subspace, int subspace_size, int n, int d) {
//    int i = blockIdx.x * blockDim.x + threadIdx.x;
//    if (i >= number_of_points) return;
//
//    int *d_neighborhood = &d_neighborhoods[i * n];//number_of_points];
//    int number_of_neighbors = 0;
//    int p_id = d_points[i];
//    for (int j = 0; j < n; j++) {//number_of_points; j++) {
//        int q_id = j;//d_points[j];
//        if (p_id != q_id) {
//            float distance = dist_prune_gpu(p_id, q_id, X, subspace, subspace_size, d);
//            if (neighborhood_size >= distance) {
//                d_neighborhood[number_of_neighbors] = j;//q_id;
//                number_of_neighbors++;
//            }
//        }
//    }
//    d_number_of_neighbors[i] = number_of_neighbors;
//}

//__global__
//void compute_is_weak_dense_prune(int *d_is_dense, int *d_points, int number_of_points,
//                                 int *d_neighborhoods, float neighborhood_size, int *d_number_of_neighbors,
//                                 float *X, int *subspace, int subspace_size, float F, int n, int num_obj, int d) {
//    int i = blockIdx.x * blockDim.x + threadIdx.x;
//    if (i < number_of_points) {
//        int *d_neighborhood = &d_neighborhoods[i * n];//number_of_points];
//
//        int p_id = d_points[i];
//        float p = phi_prune_gpu(p_id, d_neighborhood, neighborhood_size, d_number_of_neighbors[i], X, d_points,
//                                subspace, subspace_size, d);
//        float a = alpha_prune_gpu(d, neighborhood_size, n);
//        float w = omega_prune_gpu(d);
////        printf("GPU p_id: %d, p: %f, max: %f, n_size:%d\n", p_id, p, max(F * a, num_obj * w), d_number_of_neighbors[i]);
//        d_is_dense[i] = (p >= max(F * a, num_obj * w) ? 1 : 0);
//    }
//}

__global__
void
move_pruned_points(int *d_points, int *d_points_placement, int *d_new_position, int *d_is_dense,
                   int number_of_points,
                   int *d_counts, int *d_parents, int *d_dim_start, int number_of_nodes, int number_of_dims) {

    for (int i = threadIdx.x; i < number_of_nodes; i += blockDim.x) {
        if (d_counts[i] > 0) {
            d_counts[i] = 0;
        }
    }

    for (int i = 0; i < number_of_points; i += blockDim.x) {
        int j = i + threadIdx.x;//needed to get all threads into the barrier
        int new_pos = 0;
        int point = 0;
        int placement = 0;
        if (j < number_of_points) {
            new_pos = d_new_position[j] - 1;
            point = d_points[j];
            placement = d_points_placement[j];
        }
        __syncthreads();//this code looks strange, but we want all thread to reach this barrier
        if (j < number_of_points && d_is_dense[j]) {
            d_points[new_pos] = point;
            d_points_placement[new_pos] = placement;
            atomicAdd(&d_counts[placement], 1);
        }
    }

    __syncthreads();

    int leaf_start = number_of_dims > 0 ? d_dim_start[number_of_dims - 1] : 0;
    for (int i = threadIdx.x + leaf_start; i < number_of_nodes; i += blockDim.x) {
        int node = i;
        int parent = d_parents[node];
        int count = d_counts[node];
        while (count > 0 && node > 0) {
            atomicAdd(&d_counts[parent], count);
            node = parent;
            parent = d_parents[node];
        }
    }
}

bool ScyTreeArray::pruneRecursion_gpu(int min_size, float *d_X, int n, int d, float neighborhood_size, float F,
                                      int num_obj) {

//    if (this->number_of_points < min_size) {
//        return false;
//    }
//
//    int *d_neighborhoods; // number_of_points x number_of_points
//    int *d_number_of_neighbors; // number_of_points
//    int *d_is_dense; // number_of_points
//    int *d_new_position; // number_of_points
//    hipMalloc(&d_neighborhoods, sizeof(int) * number_of_points * n);//number_of_points);
//    hipMalloc(&d_number_of_neighbors, sizeof(int) * number_of_points);
//    hipMalloc(&d_is_dense, sizeof(int) * number_of_points);
//    hipMalloc(&d_new_position, sizeof(int) * number_of_points);
//
//    int number_of_blocks = number_of_points / BLOCK_SIZE;
//    if (number_of_points % BLOCK_SIZE) number_of_blocks++;
//    int number_of_threads = min(number_of_points, BLOCK_SIZE);
////    printf("before number_of_points: %d\n", number_of_points);
//
//    gpuErrchk(hipPeekAtLastError());
////    printf("<<<%d, %d>>>\n", number_of_blocks, number_of_threads);
//    find_neighborhood_prune << < number_of_blocks, number_of_threads >> >
//                                                   (d_neighborhoods, d_number_of_neighbors, d_X,
//                                                           this->d_points, number_of_points, neighborhood_size,
//                                                           this->d_restricted_dims, number_of_restricted_dims, n, d);
//
//
//    hipDeviceSynchronize();
//    gpuErrchk(hipPeekAtLastError());
//
//    compute_is_weak_dense_prune << < number_of_blocks, number_of_threads >> >
//                                                       (d_is_dense, this->d_points, number_of_points, d_neighborhoods,
//                                                               neighborhood_size, d_number_of_neighbors, d_X,
//                                                               this->d_restricted_dims, this->number_of_restricted_dims,
//                                                               F, n, num_obj, d);
//
////    hipDeviceSynchronize();
////    print_array_gpu<<<1,1>>>(d_is_dense, number_of_points);
////    hipDeviceSynchronize();
////    printf("\n");
//    hipDeviceSynchronize();
//    gpuErrchk(hipPeekAtLastError());
//
//    inclusive_scan(d_is_dense, d_new_position, number_of_points);
////    hipDeviceSynchronize();
////    print_array_gpu<<<1,1>>>(d_new_position, number_of_points);
////    hipDeviceSynchronize();
//    gpuErrchk(hipPeekAtLastError());
//
//
////    hipDeviceSynchronize();
////    printf("before d_counts:\n");
////    print_array_gpu<<<1, 1>>>(d_counts, number_of_nodes);
////    hipDeviceSynchronize();
////    printf("\n");
////
////    move_pruned_points<<<1, BLOCK_SIZE, number_of_points*sizeof(int)>>>(d_points, d_points_placement,
////                                                                        d_new_position, d_is_dense, number_of_points,
////                                                                        d_counts, d_parents, d_dim_start,
////                                                                        number_of_nodes, number_of_dims);
//
//
//    hipDeviceSynchronize();
////    printf("after d_counts:\n");
////    print_array_gpu<<<1, 1>>>(d_counts, number_of_nodes);
////    hipDeviceSynchronize();
////    printf("\n");
//
//    gpuErrchk(hipPeekAtLastError());
//    int *h_tmp = new int[1];
//    h_tmp[0] = 0;
//    hipMemcpy(h_tmp, d_new_position + number_of_points - 1, sizeof(int), hipMemcpyDeviceToHost);
//    int puned_number_of_points = h_tmp[0];
////    printf("after number_of_points: %d\n", number_of_points);
//
//    hipFree(d_neighborhoods);
//    hipFree(d_number_of_neighbors);
//    hipFree(d_is_dense);
//    hipFree(d_new_position);
//
//    return puned_number_of_points >= min_size;

    return this->number_of_points >= min_size;
}

bool ScyTreeArray::pruneRedundancy_gpu(float r, map <vector<int>, vector<int>, vec_cmp> result) {
    int max_min_size = 0;

    vector<int> subspace(this->h_restricted_dims, this->h_restricted_dims +
                                                  this->number_of_restricted_dims);
    vector<int> max_min_subspace;

    for (std::pair <vector<int>, vector<int>> subspace_clustering : result) {


        // find sizes of clusters
        vector<int> subspace_mark = subspace_clustering.first;
        if (subspace_of(subspace, subspace_mark)) {

            vector<int> clustering_mark = subspace_clustering.second;
            map<int, int> cluster_sizes;
            for (int cluster_id: clustering_mark) {
                if (cluster_id >= 0) {
                    if (cluster_sizes.count(cluster_id)) {
                        cluster_sizes[cluster_id]++;
                    } else {
                        cluster_sizes.insert(pair<int, int>(cluster_id, 1));
                    }
                }
            }


            // find the minimum size for each subspace
            int min_size = -1;
            for (std::pair<int, int> cluster_size : cluster_sizes) {
                int size = cluster_size.second;
                if (min_size == -1 ||
                    size < min_size) {//todo this min size should only be for clusters covering the region in question
                    min_size = size;
                }
            }

            // find the maximum minimum size for each subspace
            if (min_size > max_min_size) {
                max_min_size = min_size;
                max_min_subspace = subspace_mark;
            }
        }
    }

    if (max_min_size == 0) {
        return true;
    }

    return this->number_of_points * r > max_min_size * 1.;
}

ScyTreeArray::~ScyTreeArray() {
    if (number_of_nodes > 0) {
        hipFree(d_parents);
        hipFree(d_cells);
        hipFree(d_counts);
        delete[] h_parents;
        delete[] h_cells;
        delete[] h_counts;
    }
    if (number_of_dims > 0) {
        hipFree(d_dim_start);
        hipFree(d_dims);
        delete[] h_dim_start;
        delete[] h_dims;
    }
    if (number_of_restricted_dims > 0) {
        hipFree(d_restricted_dims);
        delete[] h_restricted_dims;
    }
    if (number_of_points > 0) {
        hipFree(d_points);
        hipFree(d_points_placement);
        delete[] h_points;
        delete[] h_points_placement;
    }
}

__global__
void compute_is_weak_dense_prune(int *d_is_dense, int *d_neighborhoods, int *d_neighborhood_end,
                                 int *d_points, int number_of_points,
                                 int *subspace, int subspace_size,
                                 float *X, int n, int d, float F, int num_obj,
                                 float neighborhood_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < number_of_points) {

        int p_id = d_points[i];

        float p = 0;
        int offset = p_id > 0 ? d_neighborhood_end[p_id - 1] : 0;
        for (int j = offset; j < d_neighborhood_end[p_id]; j++) {
            int q_id = d_neighborhoods[j];
            if (q_id >= 0) {
                float distance = dist_prune_gpu(p_id, q_id, X, d, subspace, subspace_size) / neighborhood_size;
                float sq = distance * distance;
                p += (1. - sq);
            }
        }
        float a = alpha_prune_gpu(d, neighborhood_size, n);
        float w = omega_prune_gpu(d);
        d_is_dense[i] = p >= max(F * a, num_obj * w) ? 1 : 0;
    }
}

__global__
void reset_counts_prune(int *d_counts, int number_of_nodes) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < number_of_nodes) {
        if (d_counts[i] > 0) {
            d_counts[i] = 0;
        }
    }
}

__global__
void remove_pruned_points_prune(int *d_is_dense, int *d_new_indices,
                                int *d_new_points, int *d_new_point_placement,
                                int *d_points, int *d_point_placement, int number_of_points,
                                int *d_counts, int *d_parents, int number_of_nodes) {
    for (int i = threadIdx.x; i < number_of_points; i += blockDim.x) {
        if (d_is_dense[i]) {
            int new_i = d_new_indices[i] - 1;
            d_new_points[new_i] = d_points[i];
            d_new_point_placement[new_i] = d_point_placement[i];
            int node = d_point_placement[i];
            atomicAdd(&d_counts[node], 1);
            int count = 0;
            while (d_parents[node] != node) {
                if (node < 0 || node >= number_of_nodes)
                    printf("out of range\n");
                if (node <= d_parents[node]) {
                    printf("remove_pruned_points_prune - hmmm... node:%d, parent:%d, number_of_nodes:%d, d_point_placement[i]:%d\n",
                           node, d_parents[node], number_of_nodes, d_point_placement[i]);
                    break;
                }
                count++;
                node = d_parents[node];
                atomicAdd(&d_counts[node], 1);
            }
        }
    }
}


bool ScyTreeArray::pruneRecursionAndRemove_gpu(int min_size, float *d_X, int n, int d, float neighborhood_size, float F,
                                               int num_obj, int *d_neighborhoods, int *d_neighborhood_end) {


    if (this->number_of_points < min_size) {
        return false;
    }
//    check_parents<<<1, 512>>>(this->d_points_placement, this->number_of_points, this->d_parents, this->number_of_nodes);

    int blocks_points = this->number_of_points / 512;
    if (this->number_of_points % 512) blocks_points++;
    int blocks_nodes = this->number_of_nodes / 512;
    if (this->number_of_nodes % 512) blocks_nodes++;

    int *d_is_dense;
    hipMalloc(&d_is_dense, sizeof(int) * this->number_of_points);
//    hipMemset(d_is_dense, 0, sizeof(int) * this->number_of_points);

    int *d_new_indices;
    hipMalloc(&d_new_indices, sizeof(int) * this->number_of_points);
    hipMemset(d_new_indices, 0, sizeof(int) * this->number_of_points);

    compute_is_weak_dense_prune <<< blocks_points, min(512, this->number_of_points) >>>(d_is_dense, d_neighborhoods,
                                                                                        d_neighborhood_end,
                                                                                        this->d_points,
                                                                                        this->number_of_points,
                                                                                        this->d_restricted_dims,
                                                                                        this->number_of_restricted_dims,
                                                                                        d_X, n, d,
                                                                                        F, num_obj, neighborhood_size);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    inclusive_scan(d_is_dense, d_new_indices, this->number_of_points);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    int new_number_of_points;
    hipMemcpy(&new_number_of_points, d_new_indices + this->number_of_points - 1, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    if (new_number_of_points == 0) {
        hipFree(d_is_dense);
        hipFree(d_new_indices);
        return false;
    }

    int *d_new_points;
    hipMalloc(&d_new_points, sizeof(int) * new_number_of_points);
//    hipMemset(d_new_points, 0, sizeof(int) * new_number_of_points);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    int *d_new_point_placement;
    hipMalloc(&d_new_point_placement, sizeof(int) * new_number_of_points);
//    hipMemset(d_new_point_placement, 0, sizeof(int) * new_number_of_points);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    reset_counts_prune<<<blocks_nodes, min(512, this->number_of_nodes)>>>(this->d_counts, this->number_of_nodes);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    remove_pruned_points_prune <<< 1, min(512, this->number_of_points) >>>(d_is_dense, d_new_indices,
                                                                           d_new_points, d_new_point_placement,
                                                                           this->d_points, this->d_points_placement,
                                                                           this->number_of_points,
                                                                           this->d_counts, this->d_parents,
                                                                           this->number_of_nodes);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());


//    printf("d_is_dense:\n");
//    print_array_gpu<< <1,1>>>(d_is_dense, this->number_of_points);
//    hipDeviceSynchronize();


    hipFree(this->d_points);
    hipFree(this->d_points_placement);
    hipFree(d_is_dense);
    hipFree(d_new_indices);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    this->d_points = d_new_points;
    this->d_points_placement = d_new_point_placement;
    this->number_of_points = new_number_of_points;

//    printf("d_points:\n");
//    print_array_gpu<< <1,1>>>(this->d_points, this->number_of_points);
//    hipDeviceSynchronize();
//    printf("d_points_placement:\n");
//    print_array_gpu<< <1,1>>>(this->d_points_placement, this->number_of_points);
//    hipDeviceSynchronize();




    return this->number_of_points >= min_size;
}

__global__
void compute_has_child_prune(int *d_has_child, int *d_parents, int *d_cells, int *d_counts, int number_of_nodes,
                             int number_of_cells) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < number_of_nodes) {
        if (d_counts[i] > 0) {
            int cell = d_cells[i];
            int parent = d_parents[i];
            if (parent != i) {
                d_has_child[parent * number_of_cells + cell] = 1;
            }
        }
    }
}

__global__
void compute_is_included_prune(int *d_is_included, int *d_has_child,
                               int *d_parents, int *d_cells, int *d_counts, int number_of_nodes, int number_of_cells) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < number_of_nodes) {
        int cell = d_cells[i];
        int parent = d_parents[i];
        if (parent == i || d_has_child[parent * number_of_cells + cell]) {
            d_is_included[i] = 1;
        }
    }
}

__global__
void update_point_placement(int *d_new_indices, int *d_points_placement, int number_of_points) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < number_of_points) {
        int placement = d_points_placement[i];
        d_points_placement[i] = d_new_indices[placement] - 1;
    }
}

__global__
void remove_nodes(int *d_new_indices, int *d_is_included, int *d_new_parents, int *d_new_cells, int *d_new_counts,
                  int *d_parents, int *d_cells, int *d_counts, int number_of_nodes) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < number_of_nodes) {
        if (d_is_included[i]) {
            int i_new = d_new_indices[i] - 1;
            int parent = d_parents[i];
            d_new_parents[i_new] = d_new_indices[parent] - 1;
            d_new_cells[i_new] = d_cells[i];
            d_new_counts[i_new] = d_counts[i];
        }
    }
}

__global__
void update_dim_start(int *d_new_indices, int *d_dim_start, int number_of_dims) {
    for (int i = threadIdx.x; i < number_of_dims; i += blockDim.x) {
        int idx = d_dim_start[i] - 1;
        d_dim_start[i] = idx >= 0 ? d_new_indices[idx] : 0;
    }
}


bool
ScyTreeArray::pruneRecursionAndRemove_gpu3(int min_size, float *d_X, int n, int d, float neighborhood_size, float F,
                                           int num_obj, int *d_neighborhoods, int *d_neighborhood_end) {


    if (this->number_of_points < min_size) {
        return false;
    }
    int blocks_points = this->number_of_points / 512;
    if (this->number_of_points % 512) blocks_points++;
    int blocks_nodes = this->number_of_nodes / 512;
    if (this->number_of_nodes % 512) blocks_nodes++;

    int *d_is_dense;
    hipMalloc(&d_is_dense, sizeof(int) * this->number_of_points);

    int *d_new_indices;
    hipMalloc(&d_new_indices, sizeof(int) * this->number_of_points);
    hipMemset(d_new_indices, 0, sizeof(int) * this->number_of_points);

    compute_is_weak_dense_prune <<< blocks_points, min(512, this->number_of_points) >>>(d_is_dense, d_neighborhoods,
                                                                                        d_neighborhood_end,
                                                                                        this->d_points,
                                                                                        this->number_of_points,
                                                                                        this->d_restricted_dims,
                                                                                        this->number_of_restricted_dims,
                                                                                        d_X, n, d,
                                                                                        F, num_obj, neighborhood_size);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    inclusive_scan(d_is_dense, d_new_indices, this->number_of_points);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    int new_number_of_points;
    hipMemcpy(&new_number_of_points, d_new_indices + this->number_of_points - 1, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    if (new_number_of_points == 0) {
        hipFree(d_is_dense);
        hipFree(d_new_indices);
        return false;
    }

    int *d_new_points;
    hipMalloc(&d_new_points, sizeof(int) * new_number_of_points);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    int *d_new_point_placement;
    hipMalloc(&d_new_point_placement, sizeof(int) * new_number_of_points);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    reset_counts_prune<<<blocks_nodes, min(512, this->number_of_nodes)>>>(this->d_counts, this->number_of_nodes);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    remove_pruned_points_prune <<< 1, min(512, this->number_of_points) >>>(d_is_dense, d_new_indices,
                                                                           d_new_points, d_new_point_placement,
                                                                           this->d_points, this->d_points_placement,
                                                                           this->number_of_points,
                                                                           this->d_counts, this->d_parents,
                                                                           this->number_of_nodes);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());


    hipFree(this->d_points);
    hipFree(this->d_points_placement);
    hipFree(d_is_dense);
    hipFree(d_new_indices);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    this->d_points = d_new_points;
    this->d_points_placement = d_new_point_placement;
    this->number_of_points = new_number_of_points;

    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());


    int *d_is_included;
    hipMalloc(&d_is_included, sizeof(int) * this->number_of_nodes);
    gpuErrchk(hipPeekAtLastError());
    hipMemset(d_is_included, 0, sizeof(int) * this->number_of_nodes);
    gpuErrchk(hipPeekAtLastError());

//    int *d_new_indices;
    hipMalloc(&d_new_indices, sizeof(int) * this->number_of_nodes);
    gpuErrchk(hipPeekAtLastError());
    hipMemset(d_new_indices, 0, sizeof(int) * this->number_of_nodes);
    gpuErrchk(hipPeekAtLastError());

    int *d_has_child;
    hipMalloc(&d_has_child, sizeof(int) * this->number_of_nodes * this->number_of_cells);
    gpuErrchk(hipPeekAtLastError());
    hipMemset(d_has_child, 0, sizeof(int) * this->number_of_nodes * this->number_of_cells);
    gpuErrchk(hipPeekAtLastError());

    compute_has_child_prune << < blocks_nodes, min(512, this->number_of_nodes) >> > (d_has_child,
            this->d_parents, this->d_cells, this->d_counts, this->number_of_nodes, this->number_of_cells);

    gpuErrchk(hipPeekAtLastError());

    compute_is_included_prune << < blocks_nodes, min(512, this->number_of_nodes) >> > (d_is_included, d_has_child,
            this->d_parents, this->d_cells, this->d_counts, this->number_of_nodes, this->number_of_cells);

    gpuErrchk(hipPeekAtLastError());

    inclusive_scan(d_is_included, d_new_indices, this->number_of_nodes);

    gpuErrchk(hipPeekAtLastError());

    int new_number_of_nodes;
    hipMemcpy(&new_number_of_nodes, d_new_indices + this->number_of_nodes - 1, sizeof(int), hipMemcpyDeviceToHost);

    gpuErrchk(hipPeekAtLastError());
    if (new_number_of_nodes <= 0) {
        hipFree(d_is_included);
        hipFree(d_new_indices);
        return false;
    }

    int *d_new_parents;
    int *d_new_cells;
    int *d_new_counts;
    hipMalloc(&d_new_parents, sizeof(int) * new_number_of_nodes);
    hipMalloc(&d_new_cells, sizeof(int) * new_number_of_nodes);
    hipMalloc(&d_new_counts, sizeof(int) * new_number_of_nodes);
    gpuErrchk(hipPeekAtLastError());

    blocks_points = this->number_of_points / 512;
    if (this->number_of_points % 512) blocks_points++;
    update_point_placement << < blocks_points, min(512, this->number_of_points) >> >
                                               (d_new_indices, this->d_points_placement, this->number_of_points);

    gpuErrchk(hipPeekAtLastError());

    remove_nodes << < blocks_nodes, min(512, this->number_of_nodes) >> >
                                    (d_new_indices, d_is_included, d_new_parents, d_new_cells, d_new_counts,
                                            this->d_parents, this->d_cells, this->d_counts, this->number_of_nodes);

    gpuErrchk(hipPeekAtLastError());
    hipFree(this->d_parents);
    hipFree(this->d_cells);
    hipFree(this->d_counts);
    gpuErrchk(hipPeekAtLastError());

//    printf("d_new_parents:\n");
//    print_array_gpu << < 1, 1 >> > (d_new_parents, new_number_of_nodes);
//    hipDeviceSynchronize();
//
//    printf("d_new_indices:\n");
//    print_array_gpu << < 1, 1 >> > (d_new_indices, this->number_of_nodes);
//    hipDeviceSynchronize();
//
//    printf("d_dim_start:\n");
//    print_array_gpu << < 1, 1 >> > (this->d_dim_start, this->number_of_dims);
//    hipDeviceSynchronize();

    this->d_parents = d_new_parents;
    this->d_cells = d_new_cells;
    this->d_counts = d_new_counts;
    this->number_of_nodes = new_number_of_nodes;

    if (this->number_of_dims > 0) {

        update_dim_start << < 1, min(512, this->number_of_dims) >> >
                                 (d_new_indices, this->d_dim_start, this->number_of_dims);

        gpuErrchk(hipPeekAtLastError());
    }

//    printf("d_dim_start:\n");
//    print_array_gpu << < 1, 1 >> > (this->d_dim_start, this->number_of_dims);
//    hipDeviceSynchronize();

    hipFree(d_is_included);
    hipFree(d_new_indices);

    return this->number_of_points >= min_size;
}
