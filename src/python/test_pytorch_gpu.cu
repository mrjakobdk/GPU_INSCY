#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


__global__
void add_kernel(float *A, float *B, float *C) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    C[i] = A[i] + B[i];
}

void add_cuda(at::Tensor A, at::Tensor B, at::Tensor C) {

    int d = A.size(0);
    float *h_A = A.data_ptr<float>();
    float *h_B = B.data_ptr<float>();
    float *h_C = C.data_ptr<float>();

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, d * sizeof(float));
    hipMalloc(&d_B, d * sizeof(float));
    hipMalloc(&d_C, d * sizeof(float));

    hipMemcpy(d_A, h_A, d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, d * sizeof(float), hipMemcpyHostToDevice);

    add_kernel << < 1, d >> > (d_A, d_B, d_C);

    hipMemcpy(h_C, d_C, d * sizeof(float), hipMemcpyDeviceToHost);
}