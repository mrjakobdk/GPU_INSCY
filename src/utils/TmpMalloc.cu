//
// Created by mrjakobdk on 6/8/20.
//

#include "TmpMalloc.cuh"

TmpMalloc::TmpMalloc(int number_of_nodes, int number_of_points, int number_of_dims, int number_of_cells) {
    //temps for merge
    int n_total = 2 * number_of_nodes;
    hipMalloc(&d_map_to_new, n_total * sizeof(int));
    hipMalloc(&d_map_to_old, n_total * sizeof(int));
    hipMalloc(&d_is_included_merge, n_total * sizeof(int));
    hipMalloc(&d_new_indecies_merge, n_total * sizeof(int));

    int step = 4; //todo find better
    int n_pivots = (n_total / step + (n_total % step ? 1 : 0));
    hipMalloc(&pivots_1, n_pivots * sizeof(int));
    hipMalloc(&pivots_2, n_pivots * sizeof(int));

    //temps for restrict
    int number_of_restrictions = number_of_dims * number_of_cells;
    hipMalloc(&d_new_indecies, number_of_nodes * number_of_restrictions * sizeof(int));
    hipMalloc(&d_new_counts, number_of_nodes * number_of_restrictions * sizeof(int));
    hipMalloc(&d_is_included, number_of_nodes * number_of_restrictions * sizeof(int));

    hipMalloc(&d_is_point_included, number_of_points * number_of_restrictions * sizeof(int));
    hipMalloc(&d_point_new_indecies, number_of_points * number_of_restrictions * sizeof(int));

    hipMalloc(&d_is_s_connected, number_of_restrictions * sizeof(int));

    hipMalloc(&d_dim_i, number_of_dims * sizeof(int));

    //temps for clustering
    hipMalloc(&d_neighborhoods, sizeof(int) * number_of_points * number_of_points);
    hipMalloc(&d_number_of_neighbors, sizeof(int) * number_of_points);
    hipMalloc(&d_is_dense, sizeof(bool) * number_of_points);
    hipMalloc(&d_disjoint_set, sizeof(int) * number_of_points);

    hipMalloc(&d_clustering, sizeof(int) * number_of_points);
}

TmpMalloc::~TmpMalloc() {
    //temps for merge
    hipFree(d_map_to_old);
    hipFree(d_map_to_new);
    hipFree(d_is_included_merge);
    hipFree(d_new_indecies_merge);
    hipFree(pivots_1);
    hipFree(pivots_2);

    //temps for restrict
    hipFree(d_new_indecies);
    hipFree(d_new_counts);
    hipFree(d_is_included);

    hipFree(d_is_point_included);
    hipFree(d_point_new_indecies);

    hipFree(d_is_s_connected);

    hipFree(d_dim_i);

    //temps for clustering
    hipFree(d_neighborhoods);
    hipFree(d_number_of_neighbors);
    hipFree(d_is_dense);
    hipFree(d_disjoint_set);

    hipFree(d_clustering);
}
