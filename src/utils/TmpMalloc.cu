//
// Created by mrjakobdk on 6/8/20.
//

#include "TmpMalloc.cuh"
#include "../../../../../../usr/include/c++/7/utility"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//#include <map>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

//using namespace std;

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

TmpMalloc::TmpMalloc(int number_of_nodes, int number_of_points, int number_of_dims, int number_of_cells, bool multi) {

    //temps for merge
    int n_total = 2 * number_of_nodes;
    hipMalloc(&d_map_to_new, n_total * sizeof(int));
    hipMalloc(&d_map_to_old, n_total * sizeof(int));
    hipMalloc(&d_is_included_merge, n_total * sizeof(int));
    hipMalloc(&d_new_indecies_merge, n_total * sizeof(int));
    gpuErrchk(hipPeekAtLastError());

    int step = 4; //todo find better - not needed any more
    int n_pivots = (n_total / step + (n_total % step ? 1 : 0));
    hipMalloc(&pivots_1, n_pivots * sizeof(int));
    hipMalloc(&pivots_2, n_pivots * sizeof(int));
    gpuErrchk(hipPeekAtLastError());

    //temps for restrict
    int number_of_restrictions = number_of_dims * number_of_cells;
    hipMalloc(&d_new_indecies, number_of_nodes * number_of_restrictions * sizeof(int));
    hipMalloc(&d_new_counts, number_of_nodes * number_of_restrictions * sizeof(int));
    hipMalloc(&d_is_included, number_of_nodes * number_of_restrictions * sizeof(int));

    hipMalloc(&d_is_point_included, number_of_points * number_of_restrictions * sizeof(int));
    hipMalloc(&d_point_new_indecies, number_of_points * number_of_restrictions * sizeof(int));

    hipMalloc(&d_is_s_connected, number_of_restrictions * sizeof(int));

    hipMalloc(&d_dim_i, number_of_dims * sizeof(int));
    gpuErrchk(hipPeekAtLastError());

    //temps for clustering //number_of_dims and number_of_cells is needed for multi clustering

    int multi_scalar = multi ? number_of_dims * number_of_cells : 1;

//    hipMalloc(&d_neighborhoods, sizeof(int) * number_of_points * number_of_points * multi_scalar);
//    printf("space for d_neighborhoods:%d\n", sizeof(int) * number_of_points * number_of_points * multi_scalar);
//    hipMalloc(&d_distance_matrix, sizeof(float) * number_of_points * number_of_points * multi_scalar);
//    printf("space for d_distance_matrix:%d\n", sizeof(float) * number_of_points * number_of_points * multi_scalar);
    hipMalloc(&d_number_of_neighbors, sizeof(int) * number_of_points * multi_scalar);
    hipMalloc(&d_is_dense, sizeof(bool) * number_of_points * multi_scalar);
    hipMalloc(&d_disjoint_set, sizeof(int) * number_of_points * multi_scalar);

    hipMalloc(&d_clustering, sizeof(int) * number_of_points * number_of_dims);
    gpuErrchk(hipPeekAtLastError());
}

TmpMalloc::~TmpMalloc() {
    //temps smart
//    for(pair<char *, int *> p: int_arrays){
//        hipFree(p.second);
//    }

    //temps for merge
    hipFree(d_map_to_old);
    hipFree(d_map_to_new);
    hipFree(d_is_included_merge);
    hipFree(d_new_indecies_merge);
    hipFree(pivots_1);
    hipFree(pivots_2);

    //temps for restrict
    hipFree(d_new_indecies);
    hipFree(d_new_counts);
    hipFree(d_is_included);

    hipFree(d_is_point_included);
    hipFree(d_point_new_indecies);

    hipFree(d_is_s_connected);

    hipFree(d_dim_i);

    //temps for clustering
//    hipFree(d_neighborhoods);
//    hipFree(d_distance_matrix);
    hipFree(d_number_of_neighbors);
    hipFree(d_is_dense);
    hipFree(d_disjoint_set);

    hipFree(d_clustering);
}
//
//int *TmpMalloc::get_int_array(char *name, int size) {
//    int *tmp;
//    map<char *, int *>::iterator it = int_arrays.find(name);
//    if (it != int_arrays.end()) {
//        tmp = int_arrays[name];
//        int tmp_size = int_array_sizes[name];
//        if (size > tmp_size) {
//            hipFree(tmp);
//            hipMalloc(&tmp, size * sizeof(int));
//            int_arrays[name] = tmp;
//            int_array_sizes[name] = size;
//        }
//    } else {
//        hipMalloc(&tmp, size * sizeof(int));
//        int_arrays.insert(pair<char *, int *>(name, tmp));
//        int_array_sizes.insert(pair<char *, int>(name, size));
//    }
//    return tmp;
//}
