//
// Created by mrjakobdk on 6/8/20.
//

#include "TmpMalloc.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

TmpMalloc::TmpMalloc(int number_of_nodes, int number_of_points, int number_of_dims, int number_of_cells) {
    //temps for merge
    int n_total = 2 * number_of_nodes;
    hipMalloc(&d_map_to_new, n_total * sizeof(int));
    hipMalloc(&d_map_to_old, n_total * sizeof(int));
    hipMalloc(&d_is_included_merge, n_total * sizeof(int));
    hipMalloc(&d_new_indecies_merge, n_total * sizeof(int));
    gpuErrchk(hipPeekAtLastError());

    int step = 4; //todo find better - not needed any more
    int n_pivots = (n_total / step + (n_total % step ? 1 : 0));
    hipMalloc(&pivots_1, n_pivots * sizeof(int));
    hipMalloc(&pivots_2, n_pivots * sizeof(int));
    gpuErrchk(hipPeekAtLastError());

    //temps for restrict
    int number_of_restrictions = number_of_dims * number_of_cells;
    hipMalloc(&d_new_indecies, number_of_nodes * number_of_restrictions * sizeof(int));
    hipMalloc(&d_new_counts, number_of_nodes * number_of_restrictions * sizeof(int));
    hipMalloc(&d_is_included, number_of_nodes * number_of_restrictions * sizeof(int));

    hipMalloc(&d_is_point_included, number_of_points * number_of_restrictions * sizeof(int));
    hipMalloc(&d_point_new_indecies, number_of_points * number_of_restrictions * sizeof(int));

    hipMalloc(&d_is_s_connected, number_of_restrictions * sizeof(int));

    hipMalloc(&d_dim_i, number_of_dims * sizeof(int));
    gpuErrchk(hipPeekAtLastError());

    //temps for clustering //number_of_dims and number_of_cells is needed for multi clustering
    hipMallocManaged(&d_neighborhoods, sizeof(int) * number_of_points * number_of_points * number_of_dims * number_of_cells);
    printf("space for d_neighborhoods:%d\n", sizeof(int) * number_of_points * number_of_points * number_of_dims * number_of_cells);
    hipMallocManaged(&d_distance_matrix, sizeof(float) * number_of_points * number_of_points * number_of_dims * number_of_cells);
    printf("space for d_distance_matrix:%d\n", sizeof(float) * number_of_points * number_of_points * number_of_dims * number_of_cells);
    hipMalloc(&d_number_of_neighbors, sizeof(int) * number_of_points * number_of_dims * number_of_cells);
    hipMalloc(&d_is_dense, sizeof(bool) * number_of_points * number_of_dims * number_of_cells);
    hipMalloc(&d_disjoint_set, sizeof(int) * number_of_points * number_of_dims * number_of_cells);

    hipMalloc(&d_clustering, sizeof(int) * number_of_points * number_of_dims);
    gpuErrchk(hipPeekAtLastError());
}

TmpMalloc::~TmpMalloc() {
    //temps for merge
    hipFree(d_map_to_old);
    hipFree(d_map_to_new);
    hipFree(d_is_included_merge);
    hipFree(d_new_indecies_merge);
    hipFree(pivots_1);
    hipFree(pivots_2);

    //temps for restrict
    hipFree(d_new_indecies);
    hipFree(d_new_counts);
    hipFree(d_is_included);

    hipFree(d_is_point_included);
    hipFree(d_point_new_indecies);

    hipFree(d_is_s_connected);

    hipFree(d_dim_i);

    //temps for clustering
    hipFree(d_neighborhoods);
    hipFree(d_distance_matrix);
    hipFree(d_number_of_neighbors);
    hipFree(d_is_dense);
    hipFree(d_disjoint_set);

    hipFree(d_clustering);
}
