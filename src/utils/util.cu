#include "hip/hip_runtime.h"
#include "util.h"
#include <stdio.h>
#include <stdlib.h>
#include <thrust/device_vector.h>
#include <numeric>

#include <ATen/ATen.h>
#include <torch/extension.h>


#define SECTION_SIZE 64
#define SECTION_SIZE_LARGE 512
#define BLOCK_WIDTH 64

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


using namespace std;

float *copy_to_device(vector <vector<float>> X, int number_of_points, int number_of_dims) {
    float *d_X;
    hipMalloc(&d_X, sizeof(float) * number_of_points * number_of_dims);
    for (int i = 0; i < number_of_points; i++) {
        float *h_x_i = X[i].data();
        hipMemcpy(&d_X[i * number_of_dims], h_x_i, sizeof(float) * number_of_dims, hipMemcpyHostToDevice);
    }
    return d_X;
}


float *copy_to_device(at::Tensor X, int number_of_points, int number_of_dims) {
    float *d_X;
    hipMalloc(&d_X, sizeof(float) * number_of_points * number_of_dims);
    hipMemcpy(d_X, X.data_ptr<float>(), sizeof(float) * number_of_points * number_of_dims, hipMemcpyHostToDevice);
    return d_X;
}

__global__
void print_array_gpu(int *x, int n) {
    for (int i = 0; i < n; i++) {
        if (x[i] < 10 && x[i] > -1)
            printf(" ");
        if (x[i] < 100 && x[i] > -10)
            printf(" ");
        printf("%d ", x[i]);
    }
    printf("\n");
}

__global__
void print_array_gpu(float *x, int n) {
    for (int i = 0; i < n; i++) {
        printf("%f ", x[i]);
    }
    printf("\n");
}

__global__
void print_array_gpu(bool *x, int n) {
    for (int i = 0; i < n; i++) {
        printf("%d ", (int) x[i]);
    }
    printf("\n");
}

__global__
void scan_kernel_eff(int *x, int *y, int n) {
/**
 * from the cuda book
 */
    __shared__ int XY[SECTION_SIZE];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        XY[threadIdx.x] = x[i];
    }

    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
        __syncthreads();
        int index = (threadIdx.x + 1) * 2 * stride - 1;
        if (index < blockDim.x) {
            XY[index] += XY[index - stride];
        }
    }

    for (int stride = SECTION_SIZE; stride > 0; stride /= 2) {
        __syncthreads();
        int index = (threadIdx.x + 1) * stride * 2 - 1;
        if (index + stride < SECTION_SIZE) {
            XY[index + stride] += XY[index];
        }
    }

    __syncthreads();

    if (i < n) {
        y[i] = XY[threadIdx.x];
    }
}


__global__
void scan_kernel_eff_large1(int *x, int *y, int *S, int n) {
/**
 * from the cuda book
 */
    __shared__ int XY[SECTION_SIZE];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        XY[threadIdx.x] = x[i];
    }

    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
        __syncthreads();
        int index = (threadIdx.x + 1) * 2 * stride - 1;
        if (index < blockDim.x) {
            XY[index] += XY[index - stride];
        }
    }

    for (int stride = SECTION_SIZE; stride > 0; stride /= 2) {
        __syncthreads();
        int index = (threadIdx.x + 1) * stride * 2 - 1;
        if (index + stride < SECTION_SIZE) {
            XY[index + stride] += XY[index];
        }
    }

    __syncthreads();

    if (i < n) {
        y[i] = XY[threadIdx.x];
    }

    if (threadIdx.x == 0) {
        S[blockIdx.x] = XY[SECTION_SIZE - 1];
    }

}

__global__
void scan_kernel_eff_large3(int *y, int *S, int n) {
/**
 * from the cuda book
 */
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (blockIdx.x > 0 && i < n) {
        y[i] += S[blockIdx.x - 1];
    }
}

void inclusive_scan(int *source, int *result, int n) {
    int numBlocks = n / SECTION_SIZE;
    if (n % SECTION_SIZE) numBlocks++;

    if (n > SECTION_SIZE) {
        int *S;
        hipMalloc((void **) &S, numBlocks * sizeof(int));
        gpuErrchk(hipPeekAtLastError());
        scan_kernel_eff_large1 << < numBlocks, SECTION_SIZE >> > (source, result, S, n);
        gpuErrchk(hipPeekAtLastError());
        inclusive_scan(S, S, numBlocks);
        gpuErrchk(hipPeekAtLastError());
        scan_kernel_eff_large3 << < numBlocks, SECTION_SIZE >> > (result, S, n);
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
        hipFree(S);//todo fix
        gpuErrchk(hipPeekAtLastError());
    } else {
        gpuErrchk(hipPeekAtLastError());
        scan_kernel_eff << < numBlocks, SECTION_SIZE >> > (source, result, n);
        gpuErrchk(hipPeekAtLastError());
    }
}

void inclusive_scan_cpu(int *d_x, int *d_y, int n) {
    int *h_x = new int[n];
    int *h_y = new int[n];
    hipMemcpy(h_y, d_y, n * sizeof(int), hipMemcpyDeviceToHost);

    int tmp = 0;
    for (int i = 0; i < n; i++) {
        tmp += h_y[i];
        h_x[i] = tmp;
    }

    hipMemcpy(d_x, h_x, n * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    delete h_x;
    delete h_y;
}

__global__ void prescan(int *g_odata, int *g_idata, int n) {
    //https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda
    extern __shared__ float temp[];// allocated on invocation
    int thid = threadIdx.x;
    int offset = 1;
    temp[2 * thid] = g_idata[2 * thid]; // load input into shared memory
    temp[2 * thid + 1] = g_idata[2 * thid + 1];
    for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
    {
        __syncthreads();
        if (thid < d) {
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }
    if (thid == 0) { temp[n - 1] = 0; } // clear the last element
    for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
    {
        offset >>= 1;
        __syncthreads();
        if (thid < d) {
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;
            float t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();
    g_odata[2 * thid] = temp[2 * thid]; // write results to device memory
    g_odata[2 * thid + 1] = temp[2 * thid + 1];
}

void inclusive_scan_v2(int *x, int *y, int n) {

}


void inclusive_scan_async(int *x, int *y, int n, hipStream_t stream) {
    int numBlocks = n / BLOCK_WIDTH;
    if (n % BLOCK_WIDTH) numBlocks++;

    if (n > SECTION_SIZE) {
        int *S;
        hipMalloc((void **) &S, (n / SECTION_SIZE) *
                                 sizeof(int));//todo should be async, but that is not possible - maybe allocate for all earlier on
        scan_kernel_eff_large1 << < numBlocks, BLOCK_WIDTH, 0, stream >> > (x, y, S, n);
        inclusive_scan_async(S, S, n / SECTION_SIZE, stream);
        scan_kernel_eff_large3 << < numBlocks, BLOCK_WIDTH, 0, stream >> > (y, S, n);
        hipFree(S);
    } else {
        scan_kernel_eff << < numBlocks, BLOCK_WIDTH, 0, stream >> > (x, y, n);
    }
}

void populate(int *parents, int *cells, int *counts, int *dim_start, int *dims, int c, int d, int n) {
    int lvl_size = c - c * 1 / 3;
    int prev_lvl_size = 0;
    int prev_count = 0;

    for (int i = 0; i < d; i++) {
        dims[i] = d - i;
        dim_start[i] = prev_count;
        int p = -1;
        for (int j = 0; j < lvl_size; j++) {
            p += j % 3 == 2 ? 0 : 1;

            if (i == 0) {
                parents[j + prev_count] = -1;
            } else {
                parents[j + prev_count] = prev_count - prev_lvl_size + p;
            }
        }
        prev_count += lvl_size;
        prev_lvl_size = lvl_size;
        lvl_size *= 1.5;
    }

    for (int i = 0; i < d; i++) {
        int r_count = 0;
        int c_no = 0;
        for (int j = 0; j < ((i < d - 1 ? dim_start[i + 1] : n) - dim_start[i]); j++) {
            int m = (i == 0 ? c * 1 / 3 : c - 2);
            if (i != 0 && j % 3 != 2) {
                r_count = 0;
                c_no = 0;
            }
            while (r_count < m && rand() % c < m) {
                r_count++;
                c_no++;
            }

            cells[dim_start[i] + j] = c_no + 1;

            c_no++;
        }
    }

    for (int j = 0; j < dim_start[d - 1]; j++) {
        counts[j] = 0;
    }

    for (int j = dim_start[d - 1]; j < n; j++) {
        int count = rand() % 10 * rand() % 10 + 1;
        counts[j] = count;
        int p = parents[j];
        while (p != -1) {
            counts[p] += count;
            p = parents[p];
        }
    }
}

void print_scy_tree(int *parents, int *cells, int *counts, int *dim_start, int *dims, int d, int n) {

    printf("r:  %d/%d\n", cells[0], counts[0]);
    if (d == 0)
        return;

    int *leaf_count = new int[n];

    for (int i = 0; i < n; i++)
        leaf_count[i] = 0;

    for (int i = dim_start[d - 1]; i < n; i++) {
        leaf_count[i] = 0;
        int p = i;
        while (p > 0) {
            leaf_count[p]++;
            p = parents[p];
        }
    }
    for (int i = 0; i < d; i++) {
        printf("%d: ", dims[i]);
        for (int j = dim_start[i]; j < ((i < (d - 1)) ? dim_start[i + 1] : n); j++) {

            if (cells[j] < 100) printf(" ");
            if (cells[j] < 10) printf(" ");
            printf("%d/%d ", cells[j], counts[j]);
            if (counts[j] < 100 && counts[j] > -10) printf(" ");
            if (counts[j] < 10 && counts[j] > -1) printf(" ");

            for (int k = 0; k < leaf_count[j] - 1; k++) {
                printf("        ", cells[j], counts[j]);
            }
        }
        printf("\n");
    }
}

int get_size(int c, int d) {
    int lvl_size = c - c * 1 / 3;
    int prev_count = 0;

    for (int i = 0; i < d; i++) {
        prev_count += lvl_size;
        lvl_size *= 1.5;
    }
    return prev_count;
}

void print_array_range(int *x, int start, int end) {
    for (int i = start; i < end; i++) {
        printf("%d ", (int) x[i]);
    }
    printf("\n\n");
}

void print_array(int *x, int n) {
    int left = 400;
    int right = 400;

    if (n <= left + right) {
        for (int i = 0; i < n; i++) {
            if (x[i] < 10 && x[i] > -1)
                printf(" ");
            if (x[i] < 100 && x[i] > -10)
                printf(" ");
            printf("%d ", (int) x[i]);
        }
    } else {
        for (int i = 0; i < left; i++) {
            printf("%d ", (int) x[i]);
        }
        printf(" ... ");
        for (int i = n - right; i < n; i++) {
            printf("%d ", (int) x[i]);
        }
    }
    printf("\n\n");
}

void print_array(vector<int> x, int n) {
    int left = 400;
    int right = 400;

    if (n <= left + right) {
        for (int i = 0; i < n; i++) {
            printf("%d ", (int) x[i]);
        }
    } else {
        for (int i = 0; i < left; i++) {
            printf("%d ", (int) x[i]);
        }
        printf(" ... ");
        for (int i = n - right; i < n; i++) {
            printf("%d ", (int) x[i]);
        }
    }
    printf("\n\n");
}

void print_array(float *x, int n) {
    int left = 30;
    int right = 10;

    if (n <= left + right) {
        for (int i = 0; i < n; i++) {
            printf("%f ", (float) x[i]);
        }
    } else {
        for (int i = 0; i < left; i++) {
            printf("%f ", (float) x[i]);
        }
        printf(" ... ");
        for (int i = n - right; i < n; i++) {
            printf("%f ", (float) x[i]);
        }
    }
    printf("\n\n");
}

void print_array(thrust::device_vector<int> x, int n) {
    int left = 30;
    int right = 10;

    if (n <= left + right) {
        for (int i = 0; i < n; i++) {
            printf("%d ", (int) x[i]);
        }
    } else {
        for (int i = 0; i < left; i++) {
            printf("%d ", x[i]);
        }
        printf(" ... ");
        for (int i = n - right; i < n; i++) {
            printf("%d ", x[i]);
        }
    }
    printf("\n\n");
}


int get_incorrect(int *array_1, int *array_2, int n) {
    int count = 0;
    for (int i = 0; i < n; i++) {
        if (array_1[i] != array_2[i]) {
            count++;
        }
    }
    return count;
}


float v_mean(std::vector<float> v) {
    //https://stackoverflow.com/questions/28574346/find-average-of-input-to-vector-c
    return accumulate(v.begin(), v.end(), 0.0) / v.size();
}


vector<float> m_get_col(vector <vector<float>> m, int i) {
    vector<float> col;
    for (int j = 0; j < m.size(); j++) {
        col.push_back(m[j][i]);
    }
    return col;
}

float v_min(std::vector<float> v) {
    float min = 100000.;//todo not good
    for (int i = 0; i < v.size(); i++) {
        if (v[i] < min) {
            min = v[i];
        }
    }
    return min;
}

float v_max(std::vector<float> v) {
    float max = -100000.;//todo not good
    for (int i = 0; i < v.size(); i++) {
        if (v[i] > max) {
            max = v[i];
        }
    }
    return max;
}

int v_max(std::vector<int> v) {
    int max = -100000.;//todo not good
    for (int i = 0; i < v.size(); i++) {
        if (v[i] > max) {
            max = v[i];
        }
    }
    return max;
}


void m_normalize(std::vector <std::vector<float>> &m) {

    float *min = new float[m[0].size()];
    float *max = new float[m[0].size()];

    for (int j = 0; j < m[0].size(); j++) {
        min[j] = 100000.;//todo not good
        max[j] = -100000.;//todo not good
    }

    for (int i = 0; i < m.size(); i++) {
        for (int j = 0; j < m[0].size(); j++) {
            min[j] = min[j] < m[i][j] ? min[j] : m[i][j];
            max[j] = max[j] > m[i][j] ? max[j] : m[i][j];
        }
        printf("finding min/max: %d%%\r", int(((i + 1) * 100) / m.size()));
    }
    printf("finding min/max: 100%%\n");

    for (int i = 0; i < m.size(); i++) {
        for (int j = 0; j < m[0].size(); j++) {
            m[i][j] = max[j] != min[j] ? (m[i][j] - min[j]) / (max[j] - min[j]) : 0;
        }
        printf("normalizing: %d%%\r", int(((i + 1) * 100) / m.size()));
    }
    printf("normalizing: 100%%\n");
}

template<class T>
vector <T> clone(vector <T> v_old) {
    vector <T> v_clone(v_old);
    return v_clone;
}

void zero(int *array, int n) {
    for (int i = 0; i < n; i++)
        array[i] = 0;
}


bool subspace_of(vector<int> subspace, vector<int> subspace_mark) {
    int i = 0;
    int j = 0;
    while (j < subspace_mark.size() && i < subspace.size()) {
        if (subspace[i] == subspace_mark[j]) {
            i++;
            j++;
        } else {
            j++;
        }
    }
    return i == subspace.size();
}

bool vec_cmp::operator()(const vector<int> &a, const vector<int> &b) const {
//    int i = 0;
//    while (a[i] == b[i]) {
//        i++;
//        if (i >= min(a.size(), b.size())) {
//            return a.size() < b.size();
//        }
//    }
//    return a[i] > b[i];
    int i = a.size() - 1;
    int j = b.size() - 1;
    while (a[i] == b[j]) {
        i--;
        j--;
        if (i < 0 || j < 0) {
            return i < j;
        }
    }

    return a[i] < b[j];
}

void join(map <vector<int>, vector<int>, vec_cmp> &result, vector<int> &clustering, vector<int> subspace, int min_size,
          float r) {

    int clustering_max = v_max(clustering);
    if (clustering_max < 0) {
        return;
    }

    int n = clustering.size();

    map<int, int> sizes;

    for (int i = 0; i < n; i++) {
        int cluster = clustering[i];
        if (cluster >= 0) {
            if (sizes.count(cluster)) {
                sizes[cluster]++;
            } else {
                sizes.insert(pair<int, int>(cluster, 1));
            }
        }
    }

    for (int i = 0; i < n; i++) {
        int cluster = clustering[i];
        if (cluster >= 0 && sizes[cluster] < min_size) {
            clustering[i] = -1;
        }
    }

//    vector<int> subspace_R(scy_tree->restricted_dims, scy_tree->restricted_dims +
//                                                      scy_tree->number_of_restricted_dims);

    for (pair <vector<int>, vector<int>> subspace_clustering : result) {

        vector<int> subspace_H = subspace_clustering.first;
        vector<int> clustering_H = subspace_clustering.second;

        if (subspace_of(subspace, subspace_H)) {

            map<int, int> sizes_H;
            set<int> to_be_removed;
            for (int cluster_id: clustering_H) {//todo this seems a bit expensive?
                if (cluster_id >= 0) {
                    if (sizes_H.count(cluster_id)) {
                        sizes_H[cluster_id]++;
                    } else {
                        sizes_H.insert(pair<int, int>(cluster_id, 1));
                    }
                }
            }

            for (int i = 0; i < n; i++) {
                int cluster = clustering[i];
                int cluster_H = clustering_H[i];
                if (cluster >= 0 && cluster_H >= 0 && sizes[cluster] * r < sizes_H[cluster_H]) {
                    //subspace_clustering[i] = -1;//todo this could course problems - all points should be remove it a part of the cluster is covered by a large enough cluster.
                    to_be_removed.insert(cluster);
                }
            }

            for (int i = 0; i < n; i++) {
                int cluster = clustering[i];
                if (cluster >= 0 &&
                    to_be_removed.find(cluster) != to_be_removed.end()) {//todo this seems a bit expensive to compute
                    clustering[i] = -1;
                }
            }
        }
    }

    clustering_max = v_max(clustering);
    if (clustering_max >= 0) {
        if (result.count(subspace)) {
            vector<int> clustering_old = result[subspace];
            int m = v_max(clustering_old);
            for (int i = 0; i < n; i++) {
                if (clustering[i] == -2) {
                    clustering_old[i] = clustering[i];
                } else if (clustering[i] >= 0) {
                    clustering_old[i] = m + 1 + clustering[i];
                }
            }
            result[subspace] = clustering_old;
        } else {
            result.insert(pair < vector < int > , vector < int >> (subspace, clustering));
        }
    }
}